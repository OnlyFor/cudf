#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <benchmarks/common/generate_input.hpp>
#include <benchmarks/fixture/benchmark_fixture.hpp>
#include <benchmarks/synchronization/synchronization.hpp>

#include <cuda/functional>

#include <cudf/detail/tdigest/tdigest.hpp>

#include <cudf_test/column_wrapper.hpp>

#include <cudf/utilities/default_stream.hpp>

#include <rmm/exec_policy.hpp>

#include <thrust/copy.h>
#include <thrust/execution_policy.h>

class TDigest : public cudf::benchmark {};

static void BM_tdigest_merge(benchmark::State& state)
{
  cudf::size_type const num_tdigests{(cudf::size_type)state.range(0)};
  cudf::size_type const tdigest_size{(cudf::size_type)state.range(1)};
  cudf::size_type const tdigests_per_group{(cudf::size_type)state.range(2)};
  cudf::size_type const max_centroids{(cudf::size_type)state.range(3)};
  auto const num_groups = num_tdigests / tdigests_per_group;
  auto const total_centroids = num_tdigests * tdigest_size;

  auto stream = cudf::get_default_stream();
  auto mr = rmm::mr::get_current_device_resource();

  constexpr int base_value = 5;
  
  // construct inner means/weights
  auto val_iter = cudf::detail::make_counting_transform_iterator(0, cuda::proclaim_return_type<double>([tdigest_size] (cudf::size_type i) {
    return static_cast<double>(base_value + i % tdigest_size);
  }));
  auto one_iter = thrust::make_constant_iterator(1);
  cudf::test::fixed_width_column_wrapper<double> means(val_iter, val_iter + total_centroids);
  cudf::test::fixed_width_column_wrapper<double> weights(one_iter, one_iter + total_centroids);
  std::vector<std::unique_ptr<cudf::column>> inner_struct_children;
  inner_struct_children.push_back(means.release());
  inner_struct_children.push_back(weights.release());
  cudf::test::structs_column_wrapper inner_struct(std::move(inner_struct_children));

  // construct the tdigest lists themselves
  auto offset_iter = cudf::detail::make_counting_transform_iterator(0, cuda::proclaim_return_type<cudf::size_type>([tdigest_size] (cudf::size_type i){
    return i * tdigest_size;
  }));  
  cudf::test::fixed_width_column_wrapper<int> offsets(offset_iter, offset_iter + num_tdigests + 1);
  auto list_col = cudf::make_lists_column(num_tdigests,
                                          offsets.release(),
                                          inner_struct.release(),
                                          0,
                                          {},
                                          stream,
                                          mr);

  // min and max columns
  auto min_iter = thrust::make_constant_iterator(base_value);
  auto max_iter = thrust::make_constant_iterator(base_value + (tdigest_size - 1));
  cudf::test::fixed_width_column_wrapper<double> mins(min_iter, min_iter + num_tdigests);
  cudf::test::fixed_width_column_wrapper<double> maxes(max_iter, max_iter + num_tdigests);

  // assemble the whole thing
  std::vector<std::unique_ptr<cudf::column>> tdigest_children;
  tdigest_children.push_back(std::move(list_col));
  tdigest_children.push_back(mins.release());
  tdigest_children.push_back(maxes.release());
  cudf::test::structs_column_wrapper tdigest(std::move(tdigest_children));

  rmm::device_uvector<cudf::size_type> group_offsets(num_groups+1, stream, mr);
  rmm::device_uvector<cudf::size_type> group_labels(num_tdigests, stream, mr);
  auto group_offset_iter = cudf::detail::make_counting_transform_iterator(0, cuda::proclaim_return_type<cudf::size_type>([tdigests_per_group] __device__ (cudf::size_type i){
    return i * tdigests_per_group;
  }));
  thrust::copy(rmm::exec_policy_nosync(stream, mr),
               group_offset_iter,
               group_offset_iter + num_groups + 1,
               group_offsets.begin());
  auto group_label_iter = cudf::detail::make_counting_transform_iterator(0, cuda::proclaim_return_type<cudf::size_type>([tdigests_per_group] __device__ (cudf::size_type i){
    return i / tdigests_per_group;
  }));
  thrust::copy(rmm::exec_policy_nosync(stream, mr),
               group_label_iter,
               group_label_iter + num_tdigests,
               group_labels.begin());

  for (auto _ : state) {
    cuda_event_timer raii(state, true, stream);

    auto result = cudf::tdigest::detail::group_merge_tdigest(tdigest,
                                                             group_offsets,
                                                             group_labels,
                                                             num_groups,
                                                             max_centroids,
                                                             stream,
                                                             mr);
  }
}

#define TDIGEST_BENCHMARK_DEFINE(name, num_tdigests, tdigest_size, tdigests_per_group, max_centroids)   \
  BENCHMARK_DEFINE_F(TDigest, name)                                                                     \
  (::benchmark::State & st) { BM_tdigest_merge(st); }                                                   \
  BENCHMARK_REGISTER_F(TDigest, name)                                                                   \
    ->Args({num_tdigests, tdigest_size, tdigests_per_group, max_centroids})                             \
    ->Unit(benchmark::kMillisecond)                                                                     \
    ->UseManualTime()                                                                                   \
    ->Iterations(8)

TDIGEST_BENCHMARK_DEFINE(many_tiny_groups, 1'000'000, 1, 1, 10000);
TDIGEST_BENCHMARK_DEFINE(many_tiny_groups2, 1'000'000, 1, 1, 1000);

TDIGEST_BENCHMARK_DEFINE(many_small_groups, 3'000'000, 3, 3, 10000);
TDIGEST_BENCHMARK_DEFINE(many_small_groups2, 3'000'000, 3, 3, 1000);