/*
 * Copyright (c) 2022-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <io/fst/lookup_tables.cuh>
#include <io/utilities/hostdevice_vector.hpp>

#include <cudf_test/base_fixture.hpp>
#include <cudf_test/cudf_gtest.hpp>

#include <cudf/scalar/scalar_factories.hpp>
#include <cudf/strings/repeat_strings.hpp>
#include <cudf/types.hpp>

#include <rmm/hip_stream.hpp>
#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_buffer.hpp>
#include <rmm/device_uvector.hpp>

#include <cstdlib>
#include <string>
#include <vector>

namespace {

//------------------------------------------------------------------------------
// CPU-BASED IMPLEMENTATIONS FOR VERIFICATION
//------------------------------------------------------------------------------
/**
 * @brief CPU-based implementation of a finite-state transducer (FST).
 *
 * @tparam InputItT Forward input iterator type to symbols fed into the FST
 * @tparam StateT Type representing states of the finite-state machine
 * @tparam SymbolGroupLutT Sequence container of symbol groups. Each symbol group is a sequence
 * container to symbols within that group.
 * @tparam TransitionTableT Two-dimensional container type
 * @tparam TransducerTableT Two-dimensional container type
 * @tparam OutputItT Forward output iterator type
 * @tparam IndexOutputItT Forward output iterator type
 * @param[in] begin Forward iterator to the beginning of the symbol sequence
 * @param[in] end Forward iterator to one past the last element of the symbol sequence
 * @param[in] init_state The starting state of the finite-state machine
 * @param[in] symbol_group_lut Sequence container of symbol groups. Each symbol group is a sequence
 * container to symbols within that group. The index of the symbol group containing a symbol being
 * read will be used as symbol_gid of the transition and translation tables.
 * @param[in] transition_table The two-dimensional transition table, i.e.,
 * transition_table[state][symbol_gid] -> new_state
 * @param[in] translation_table The two-dimensional transducer table, i.e.,
 * translation_table[state][symbol_gid] -> range_of_output_symbols
 * @param[out] out_tape A forward output iterator to which the transduced input will be written
 * @param[out] out_index_tape A forward output iterator to which indexes of the symbols that
 * actually caused some output are written to
 * @return A pair of iterators to one past the last element of (1) the transduced output symbol
 * sequence and (2) the indexes of
 */
template <typename InputItT,
          typename StateT,
          typename SymbolGroupLutT,
          typename TransitionTableT,
          typename TransducerTableT,
          typename OutputItT,
          typename IndexOutputItT>
static std::pair<OutputItT, IndexOutputItT> fst_baseline(InputItT begin,
                                                         InputItT end,
                                                         StateT const& init_state,
                                                         SymbolGroupLutT symbol_group_lut,
                                                         TransitionTableT transition_table,
                                                         TransducerTableT translation_table,
                                                         OutputItT out_tape,
                                                         IndexOutputItT out_index_tape)
{
  // Initialize "FSM" with starting state
  StateT state = init_state;

  // To track the symbol offset within the input that caused the FST to output
  std::size_t in_offset = 0;
  for (auto it = begin; it < end; it++) {
    // The symbol currently being read
    auto const& symbol = *it;

    // Iterate over symbol groups and search for the first symbol group containing the current
    // symbol, if no match is found we use cend(symbol_group_lut) as the "catch-all" symbol group
    auto symbol_group_it =
      std::find_if(std::cbegin(symbol_group_lut), std::cend(symbol_group_lut), [symbol](auto& sg) {
        return std::find(std::cbegin(sg), std::cend(sg), symbol) != std::cend(sg);
      });
    auto symbol_group = std::distance(std::cbegin(symbol_group_lut), symbol_group_it);

    // Output the translated symbols to the output tape
    out_tape = std::copy(std::cbegin(translation_table[state][symbol_group]),
                         std::cend(translation_table[state][symbol_group]),
                         out_tape);

    auto out_size = std::distance(std::cbegin(translation_table[state][symbol_group]),
                                  std::cend(translation_table[state][symbol_group]));

    out_index_tape = std::fill_n(out_index_tape, out_size, in_offset);

    // Transition the state of the finite-state machine
    state = static_cast<char>(transition_table[state][symbol_group]);

    // Continue with next symbol from input tape
    in_offset++;
  }
  return {out_tape, out_index_tape};
}
}  // namespace

// Base test fixture for tests
struct FstTest : public cudf::test::BaseFixture {};

void run_test(std::string& input)
{
  // Type used to represent the atomic symbol type used within the finite-state machine
  using SymbolT = char;

  // Type sufficiently large to index symbols within the input and output (may be unsigned)
  using SymbolOffsetT = uint32_t;

  // Prepare cuda stream for data transfers & kernels
  rmm::hip_stream stream{};
  rmm::cuda_stream_view stream_view(stream);

  // Run algorithm
  enum class dfa_states : char { TT_OOS = 0U, TT_DQS, TT_SQS, TT_DEC, TT_SEC, TT_NUM_STATES };

  enum class dfa_symbol_group_id : uint32_t {
    OPENING_BRACE,      ///< Opening brace SG: {
    OPENING_BRACKET,    ///< Opening bracket SG: [
    CLOSING_BRACE,      ///< Closing brace SG: }
    CLOSING_BRACKET,    ///< Closing bracket SG: ]
    DOUBLE_QUOTE_CHAR,  ///< Quote character SG: "
    SINGLE_QUOTE_CHAR,  ///< Quote character SG: '
    ESCAPE_CHAR,        ///< Escape character SG: '\'
    OTHER_SYMBOLS,      ///< SG implicitly matching all other characters
    NUM_SYMBOL_GROUPS   ///< Total number of symbol groups
  };

  // Aliases for readability of the transition table
  constexpr auto TT_OOS = dfa_states::TT_OOS;
  constexpr auto TT_DQS = dfa_states::TT_DQS;
  constexpr auto TT_SQS = dfa_states::TT_SQS;
  constexpr auto TT_DEC = dfa_states::TT_DEC;
  constexpr auto TT_SEC = dfa_states::TT_SEC;

  constexpr auto TT_NUM_STATES     = static_cast<char>(dfa_states::TT_NUM_STATES);
  constexpr auto NUM_SYMBOL_GROUPS = static_cast<uint32_t>(dfa_symbol_group_id::NUM_SYMBOL_GROUPS);

  // The i-th string representing all the characters of a symbol group
  std::array<std::string, NUM_SYMBOL_GROUPS - 1> const qna_sgs{"{", "[", "}", "]", "\"", "'", "\\"};

  // Transition table
  // Does not support JSON lines
  std::array<std::array<dfa_states, NUM_SYMBOL_GROUPS>, TT_NUM_STATES> const qna_state_tt{{
    /* IN_STATE    {       [       }       ]       "       '       \       OTHER */
    /* TT_OOS */ {{TT_OOS, TT_OOS, TT_OOS, TT_OOS, TT_DQS, TT_SQS, TT_OOS, TT_OOS}},
    /* TT_DQS */ {{TT_DQS, TT_DQS, TT_DQS, TT_DQS, TT_OOS, TT_DQS, TT_DEC, TT_DQS}},
    /* TT_SQS */ {{TT_SQS, TT_SQS, TT_SQS, TT_SQS, TT_SQS, TT_OOS, TT_SEC, TT_SQS}},
    /* TT_DEC */ {{TT_DQS, TT_DQS, TT_DQS, TT_DQS, TT_DQS, TT_DQS, TT_DQS, TT_DQS}},
    /* TT_SEC */ {{TT_SQS, TT_SQS, TT_SQS, TT_SQS, TT_SQS, TT_SQS, TT_SQS, TT_SQS}},
  }};

  // Translation table (i.e., for each transition, what are the symbols that we output)
  std::array<std::array<std::vector<char>, NUM_SYMBOL_GROUPS>, TT_NUM_STATES> const qna_out_tt{
    {/* IN_STATE         {      [      }      ]      "              '      \    OTHER */
     /* TT_OOS    */ {{{'{'}, {'['}, {'}'}, {']'}, {'"'}, {'"'}, {'\\'}, {'x'}}},
     /* TT_DQS    */ {{{'{'}, {'['}, {'}'}, {']'}, {'"'}, {'\''}, {'\\'}, {'x'}}},
     /* TT_SQS    */ {{{'{'}, {'['}, {'}'}, {']'}, {'\\', '"'}, {'"'}, {'\\'}, {'x'}}},
     /* TT_DEC    */ {{{'{'}, {'['}, {'}'}, {']'}, {'"'}, {'\''}, {'\\'}, {'x'}}},
     /* TT_SEC    */ {{{'{'}, {'['}, {'}'}, {']'}, {'"'}, {'\''}, {'\\'}, {'x'}}}}};

  // The DFA's starting state
  constexpr char start_state = static_cast<char>(TT_OOS);

  auto parser = cudf::io::fst::detail::make_fst(
    cudf::io::fst::detail::make_symbol_group_lut(qna_sgs),
    cudf::io::fst::detail::make_transition_table(qna_state_tt),
    cudf::io::fst::detail::make_translation_table<TT_NUM_STATES * NUM_SYMBOL_GROUPS>(qna_out_tt),
    stream);

  auto d_input_scalar = cudf::make_string_scalar(input);
  auto& d_input       = static_cast<cudf::scalar_type_t<std::string>&>(*d_input_scalar);

  // Prepare input & output buffers
  constexpr std::size_t single_item = 1;
  cudf::detail::hostdevice_vector<SymbolT> output_gpu(input.size() * 2, stream_view);
  cudf::detail::hostdevice_vector<SymbolOffsetT> output_gpu_size(single_item, stream_view);
  cudf::detail::hostdevice_vector<SymbolOffsetT> out_indexes_gpu(input.size(), stream_view);

  // Allocate device-side temporary storage & run algorithm
  parser.Transduce(d_input.data(),
                   static_cast<SymbolOffsetT>(d_input.size()),
                   output_gpu.device_ptr(),
                   out_indexes_gpu.device_ptr(),
                   output_gpu_size.device_ptr(),
                   start_state,
                   stream.value());

  // Async copy results from device to host
  output_gpu.device_to_host_async(stream.view());
  out_indexes_gpu.device_to_host_async(stream.view());
  output_gpu_size.device_to_host_async(stream.view());

  // Prepare CPU-side results for verification
  std::string output_cpu{};
  std::vector<SymbolOffsetT> out_index_cpu{};
  output_cpu.reserve(input.size());
  out_index_cpu.reserve(input.size());

  // Run CPU-side algorithm
  fst_baseline(std::begin(input),
               std::end(input),
               start_state,
               qna_sgs,
               qna_state_tt,
               qna_out_tt,
               std::back_inserter(output_cpu),
               std::back_inserter(out_index_cpu));

  // Make sure results have been copied back to host
  stream.synchronize();

  // Verify results
  ASSERT_EQ(output_gpu_size[0], output_cpu.size());
  std::cout << output_cpu << std::endl;
  CUDF_TEST_EXPECT_VECTOR_EQUAL(output_gpu, output_cpu, output_cpu.size());
  // TODO: indexing for multicharacter translations
  // CUDF_TEST_EXPECT_VECTOR_EQUAL(out_indexes_gpu, out_index_cpu, output_cpu.size());
}

TEST_F(FstTest, GroundTruth_QuoteNormalizationSimple1)
{
  std::string input = R"({"A":'TEST"'})";
  run_test(input);
}
TEST_F(FstTest, GroundTruth_QuoteNormalizationSimple2)
{
  std::string input = R"({'A':"TEST'"} ['OTHER STUFF'])";
  run_test(input);
}
TEST_F(FstTest, GroundTruth_QuoteNormalizationSimple3)
{
  std::string input = R"(['{"A": "B"}',"{'A': 'B'}"])";
  run_test(input);
}
TEST_F(FstTest, GroundTruth_QuoteNormalizationSimple4)
{
  std::string input = R"({"ain't ain't a word and you ain't supposed to say it":'"""""""""""'})";
  run_test(input);
}
TEST_F(FstTest, GroundTruth_QuoteNormalizationSimple5)
{
  std::string input = R"({"\"'\"'\"'\"'":'"\'"\'"\'"\'"'})";
  run_test(input);
}
TEST_F(FstTest, GroundTruth_QuoteNormalizationSimple6)
{
  std::string input = R"([{"ABC':'CBA":'XYZ":"ZXY'}])";
  run_test(input);
}
TEST_F(FstTest, GroundTruth_QuoteNormalizationSimple7)
{
  std::string input = R"(["\t","\\t","\\","\\\'\"\\\\","\n","\b"])";
  run_test(input);
}
CUDF_TEST_PROGRAM_MAIN()
