#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/copy.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/strings/detail/utilities.hpp>
#include <cudf/strings/find.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/utilities/default_stream.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/memory_resource.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

#include <hipcub/hipcub.hpp>
#include <cuda/functional>
#include <thrust/binary_search.h>
#include <thrust/fill.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/sequence.h>
#include <thrust/unique.h>

#include <algorithm>

namespace cudf {
namespace strings {
namespace detail {
namespace {

/**
 * @brief Threshold to decide on using string or warp parallel functions.
 *
 * If the average byte length of a string in a column exceeds this value then
 * a warp-parallel function is used.
 */
constexpr size_type AVG_CHAR_BYTES_THRESHOLD = 64;

CUDF_KERNEL void multi_contains_warp_parallel(column_device_view const d_strings,
                                              column_device_view const d_targets,
                                              u_char const* d_first_bytes,
                                              size_type const* d_indices,
                                              size_type const* d_offsets,
                                              size_type unique_count,
                                              cudf::device_span<bool*> d_results)
{
  auto const num_targets = d_targets.size();
  auto const idx         = cudf::detail::grid_1d::global_thread_id();
  auto const str_idx     = idx / cudf::detail::warp_size;
  if (str_idx >= d_strings.size()) { return; }
  if (d_strings.is_null(str_idx)) { return; }
  // get the string for this warp
  auto const d_str = d_strings.element<string_view>(str_idx);

  // size of shared_bools = targets_size * block_size
  // each thread uses targets_size bools
  extern __shared__ bool shared_bools[];
  auto const lane_idx = idx % cudf::detail::warp_size;

  // initialize result: set true if target is empty, false otherwise
  for (int target_idx = 0; target_idx < num_targets; target_idx++) {
    auto const d_target = d_targets.element<string_view>(target_idx);
    shared_bools[threadIdx.x * num_targets + target_idx] = d_target.empty();
  }

  auto const last_ptr = d_first_bytes + unique_count;
  for (size_type str_byte_idx = lane_idx; str_byte_idx < d_str.size_bytes();
       str_byte_idx += cudf::detail::warp_size) {
    // search for byte in first_bytes array
    auto const chr      = static_cast<u_char>(*(d_str.data() + str_byte_idx));
    auto const byte_ptr = thrust::lower_bound(thrust::seq, d_first_bytes, last_ptr, chr);
    // if not found, continue to next byte
    if ((byte_ptr == last_ptr) || (*byte_ptr != chr)) { continue; }
    // compute index of matched byte
    auto offset_idx     = static_cast<size_type>(thrust::distance(d_first_bytes, byte_ptr));
    auto map_idx        = d_offsets[offset_idx];
    auto const last_idx = (offset_idx + 1) < unique_count ? d_offsets[offset_idx + 1] : num_targets;
    // check for targets that begin with chr
    while (map_idx < last_idx) {
      auto const target_idx      = d_indices[map_idx++];
      auto const temp_result_idx = (threadIdx.x * num_targets) + target_idx;
      if (!shared_bools[temp_result_idx]) {  // not found before
        auto const d_target = d_targets.element<string_view>(target_idx);
        if ((d_str.size_bytes() - str_byte_idx) >= d_target.size_bytes()) {
          // first char already checked, only need to check the [2nd, end) chars if has.
          bool found = true;
          for (auto i = 1; i < d_target.size_bytes() && found; i++) {
            if (*(d_str.data() + str_byte_idx + i) != *(d_target.data() + i)) { found = false; }
          }
          if (found) { shared_bools[temp_result_idx] = true; }
        }
      }
    }
  }

  // wait all lanes are done in a warp
  __syncwarp();

  if (lane_idx == 0) {
    for (int target_idx = 0; target_idx < num_targets; target_idx++) {
      bool found = false;
      // use thrust::any() algorithm with strided iterator?
      for (size_type lidx = 0; lidx < cudf::detail::warp_size && !found; lidx++) {
        auto const temp_idx = ((threadIdx.x + lidx) * num_targets) + target_idx;
        if (shared_bools[temp_idx]) { found = true; }
      }
      d_results[target_idx][str_idx] = found;
    }
  }
}

CUDF_KERNEL void multi_contains_row_parallel(column_device_view const d_strings,
                                             column_device_view const d_targets,
                                             u_char const* d_first_bytes,
                                             size_type const* d_indices,
                                             size_type const* d_offsets,
                                             size_type unique_count,
                                             cudf::device_span<bool*> d_results)
{
  auto const str_idx     = static_cast<size_type>(cudf::detail::grid_1d::global_thread_id());
  auto const num_targets = d_targets.size();
  if (str_idx >= d_strings.size()) { return; }
  if (d_strings.is_null(str_idx)) { return; }
  auto const d_str = d_strings.element<string_view>(str_idx);

  // initialize output; the result of searching empty target is true
  for (auto target_idx = 0; target_idx < num_targets; ++target_idx) {
    auto const d_target            = d_targets.element<string_view>(target_idx);
    d_results[target_idx][str_idx] = d_target.empty();
  }

  // process each byte of the current string
  auto const last_ptr = d_first_bytes + unique_count;
  for (auto str_byte_idx = 0; str_byte_idx < d_str.size_bytes(); ++str_byte_idx) {
    // search for byte in first_bytes array
    auto const chr      = static_cast<u_char>(*(d_str.data() + str_byte_idx));
    auto const byte_ptr = thrust::lower_bound(thrust::seq, d_first_bytes, last_ptr, chr);
    // if not found, continue to next byte
    if ((byte_ptr == last_ptr) || (*byte_ptr != chr)) { continue; }
    // compute index of matched byte
    auto offset_idx     = static_cast<size_type>(thrust::distance(d_first_bytes, byte_ptr));
    auto map_idx        = d_offsets[offset_idx];
    auto const last_idx = (offset_idx + 1) < unique_count ? d_offsets[offset_idx + 1] : num_targets;
    // check for targets that begin with chr
    while (map_idx < last_idx) {
      auto const target_idx = d_indices[map_idx++];
      if (!d_results[target_idx][str_idx]) {  // not found before
        auto const d_target = d_targets.element<string_view>(target_idx);
        if ((d_str.size_bytes() - str_byte_idx) >= d_target.size_bytes()) {
          // first char already checked, only need to check the [2nd, end) chars
          bool found = true;
          for (auto i = 1; i < d_target.size_bytes() && found; i++) {
            if (*(d_str.data() + str_byte_idx + i) != *(d_target.data() + i)) { found = false; }
          }
          if (found) { d_results[target_idx][str_idx] = true; }
        }
      }
    }
  }
}

std::vector<std::unique_ptr<column>> multi_contains(bool warp_parallel,
                                                    strings_column_view const& input,
                                                    strings_column_view const& targets,
                                                    rmm::cuda_stream_view stream,
                                                    rmm::mr::device_memory_resource* mr)
{
  auto const num_targets = static_cast<size_type>(targets.size());

  auto const d_strings = column_device_view::create(input.parent(), stream);
  auto const d_targets = column_device_view::create(targets.parent(), stream);

  // copy the first byte of each target and sort the first bytes
  auto first_bytes = rmm::device_uvector<u_char>(targets.size(), stream);
  auto indices     = rmm::device_uvector<size_type>(targets.size(), stream);
  {
    auto tgt_itr = thrust::make_transform_iterator(
      d_targets->begin<string_view>(),
      cuda::proclaim_return_type<u_char>([] __device__(auto const& d_tgt) -> u_char {
        return d_tgt.empty() ? u_char{0} : static_cast<u_char>(d_tgt.data()[0]);
      }));
    auto count_itr = thrust::make_counting_iterator<size_type>(0);
    auto keys_out  = first_bytes.begin();
    auto vals_out  = indices.begin();
    auto cmp_op    = thrust::less();
    auto sv        = stream.value();

    std::size_t tmp_bytes = 0;
    hipcub::DeviceMergeSort::SortPairsCopy(
      nullptr, tmp_bytes, tgt_itr, count_itr, keys_out, vals_out, num_targets, cmp_op, sv);
    auto tmp_stg = rmm::device_buffer(tmp_bytes, stream);
    hipcub::DeviceMergeSort::SortPairsCopy(
      tmp_stg.data(), tmp_bytes, tgt_itr, count_itr, keys_out, vals_out, num_targets, cmp_op, sv);
  }

  // remove duplicates to speed up lower_bound
  auto offsets = rmm::device_uvector<size_type>(targets.size(), stream);
  thrust::sequence(rmm::exec_policy_nosync(stream), offsets.begin(), offsets.end());
  auto end = thrust::unique_by_key(
    rmm::exec_policy_nosync(stream), first_bytes.begin(), first_bytes.end(), offsets.begin());
  auto ucount = static_cast<size_type>(thrust::distance(first_bytes.begin(), end.first));

  // create output columns
  auto const results_iter = cudf::detail::make_counting_transform_iterator(0, [&](int i) {
    return make_numeric_column(data_type{type_id::BOOL8},
                               input.size(),
                               cudf::detail::copy_bitmask(input.parent(), stream, mr),
                               input.null_count(),
                               stream,
                               mr);
  });
  auto results_list =
    std::vector<std::unique_ptr<column>>(results_iter, results_iter + targets.size());
  auto device_results_list = [&] {
    auto host_results_pointer_iter =
      thrust::make_transform_iterator(results_list.begin(), [](auto const& results_column) {
        return results_column->mutable_view().template data<bool>();
      });
    auto host_results_pointers = std::vector<bool*>(
      host_results_pointer_iter, host_results_pointer_iter + results_list.size());
    return cudf::detail::make_device_uvector_async(host_results_pointers, stream, mr);
  }();

  constexpr cudf::thread_index_type block_size = 256;

  auto d_first_bytes = first_bytes.data();
  auto d_indices     = indices.data();
  auto d_offsets     = offsets.data();

  if (warp_parallel) {
    cudf::detail::grid_1d grid{
      static_cast<cudf::thread_index_type>(input.size()) * cudf::detail::warp_size, block_size};
    int shared_mem_size = block_size * targets.size();
    multi_contains_warp_parallel<<<grid.num_blocks,
                                   grid.num_threads_per_block,
                                   shared_mem_size,
                                   stream.value()>>>(
      *d_strings, *d_targets, d_first_bytes, d_indices, d_offsets, ucount, device_results_list);
  } else {
    cudf::detail::grid_1d grid{static_cast<cudf::thread_index_type>(input.size()), block_size};
    multi_contains_row_parallel<<<grid.num_blocks, grid.num_threads_per_block, 0, stream.value()>>>(
      *d_strings, *d_targets, d_first_bytes, d_indices, d_offsets, ucount, device_results_list);
  }

  return results_list;
}

}  // namespace

std::unique_ptr<table> contains_multiple(strings_column_view const& input,
                                         strings_column_view const& targets,
                                         rmm::cuda_stream_view stream,
                                         rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(not targets.is_empty(), "Must specify at least one target string.");
  CUDF_EXPECTS(not targets.has_nulls(), "Target strings cannot be null");

  if ((input.null_count() == input.size()) ||
      ((input.chars_size(stream) / (input.size() - input.null_count())) <=
       AVG_CHAR_BYTES_THRESHOLD)) {
    // Small strings. Searching for multiple targets in one thread seems to work fastest.
    return std::make_unique<table>(
      multi_contains(/**warp parallel**/ false, input, targets, stream, mr));
  }

  // Long strings
  // Use warp parallel when the average string width is greater than the threshold
  static constexpr size_type target_group_size = 16;  // perhaps can be calculated
  if (targets.size() <= target_group_size) {
    return std::make_unique<table>(
      multi_contains(/**warp parallel**/ true, input, targets, stream, mr));
  }

  // Too many targets will consume more shared memory, so split targets
  // TODO: test with large working memory (instead of shared-memory)
  std::vector<std::unique_ptr<column>> ret_columns;
  auto const num_groups = cudf::util::div_rounding_up_safe(targets.size(), target_group_size);
  for (size_type group_idx = 0; group_idx < num_groups; group_idx++) {
    auto const start_target = group_idx * target_group_size;
    auto const end_target   = std::min(start_target + target_group_size, targets.size());

    auto target_group = cudf::detail::slice(targets.parent(), start_target, end_target, stream);
    auto bool_columns = multi_contains(
      /**warp parallel**/ true, input, strings_column_view(target_group), stream, mr);
    for (auto& c : bool_columns) {
      ret_columns.push_back(std::move(c));  // transfer ownership
    }
  }
  return std::make_unique<table>(std::move(ret_columns));
}

}  // namespace detail

std::unique_ptr<table> contains_multiple(strings_column_view const& strings,
                                         strings_column_view const& targets,
                                         rmm::cuda_stream_view stream,
                                         rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::contains_multiple(strings, targets, stream, mr);
}

}  // namespace strings
}  // namespace cudf
