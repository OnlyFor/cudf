#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/copy.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/strings/detail/utilities.hpp>
#include <cudf/strings/find.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/utilities/default_stream.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/memory_resource.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

#include <hipcub/hipcub.hpp>
#include <cuda/functional>
#include <thrust/binary_search.h>
#include <thrust/fill.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/logical.h>
#include <thrust/sequence.h>
#include <thrust/unique.h>

#include <algorithm>

namespace cudf {
namespace strings {
namespace detail {
namespace {

/**
 * @brief Threshold to decide on using string or warp parallel functions.
 *
 * If the average byte length of a string in a column exceeds this value then
 * a warp-parallel function is used.
 */
constexpr size_type AVG_CHAR_BYTES_THRESHOLD = 64;

CUDF_KERNEL void multi_contains_warp_parallel(column_device_view const d_strings,
                                              column_device_view const d_targets,
                                              u_char const* d_first_bytes,
                                              size_type const* d_indices,
                                              size_type const* d_offsets,
                                              size_type unique_count,
                                              bool* working_memory,
                                              cudf::device_span<bool*> d_results)
{
  auto const num_targets = d_targets.size();
  auto const idx         = cudf::detail::grid_1d::global_thread_id();
  auto const str_idx     = idx / cudf::detail::warp_size;
  if (str_idx >= d_strings.size()) { return; }
  if (d_strings.is_null(str_idx)) { return; }
  // get the string for this warp
  auto const d_str = d_strings.element<string_view>(str_idx);

  auto const lane_idx = idx % cudf::detail::warp_size;

  // size of shared_bools = num_targets * block_size
  // each thread uses num_targets bools
  extern __shared__ bool shared_bools[];
  auto const warp_idx = threadIdx.x / cudf::detail::warp_size;
  // bools for the current string
  auto bools = working_memory == nullptr
                 ? (shared_bools + (warp_idx * cudf::detail::warp_size * num_targets))
                 : (working_memory + (str_idx * cudf::detail::warp_size * num_targets));

  // initialize result: set true if target is empty, false otherwise
  for (auto target_idx = lane_idx; target_idx < num_targets;
       target_idx += cudf::detail::warp_size) {
    auto const d_target = d_targets.element<string_view>(target_idx);
    auto const begin    = bools + (target_idx * cudf::detail::warp_size);
    thrust::uninitialized_fill(
      thrust::seq, begin, begin + cudf::detail::warp_size, d_target.empty());
  }

  auto const last_ptr = d_first_bytes + unique_count;
  for (size_type str_byte_idx = lane_idx; str_byte_idx < d_str.size_bytes();
       str_byte_idx += cudf::detail::warp_size) {
    // search for byte in first_bytes array
    auto const chr      = static_cast<u_char>(*(d_str.data() + str_byte_idx));
    auto const byte_ptr = thrust::lower_bound(thrust::seq, d_first_bytes, last_ptr, chr);
    // if not found, continue to next byte
    if ((byte_ptr == last_ptr) || (*byte_ptr != chr)) { continue; }
    // compute index of matched byte
    auto const offset_idx = static_cast<size_type>(thrust::distance(d_first_bytes, byte_ptr));
    auto map_idx          = d_offsets[offset_idx];
    auto const last_idx = (offset_idx + 1) < unique_count ? d_offsets[offset_idx + 1] : num_targets;
    // check for targets that begin with chr
    while (map_idx < last_idx) {
      auto const target_idx = d_indices[map_idx++];
      auto const bool_idx   = (target_idx * cudf::detail::warp_size) + lane_idx;
      if (!bools[bool_idx]) {  // not found before
        auto const d_target = d_targets.element<string_view>(target_idx);
        if ((d_str.size_bytes() - str_byte_idx) >= d_target.size_bytes()) {
          // first char already checked, only need to check the [2nd, end) chars if has.
          bool found = true;
          for (auto i = 1; i < d_target.size_bytes() && found; i++) {
            if (*(d_str.data() + str_byte_idx + i) != *(d_target.data() + i)) { found = false; }
          }
          if (found) { bools[bool_idx] = true; }
        }
      }
    }
  }

  // wait all lanes are done in a warp
  __syncwarp();

  // reduce the bools for each target to store in the result
  for (auto target_idx = lane_idx; target_idx < num_targets;
       target_idx += cudf::detail::warp_size) {
    auto begin = bools + (target_idx * cudf::detail::warp_size);
    auto found =
      thrust::any_of(thrust::seq, begin, begin + cudf::detail::warp_size, thrust::identity<bool>{});
    d_results[target_idx][str_idx] = found;
  }
}

CUDF_KERNEL void multi_contains_row_parallel(column_device_view const d_strings,
                                             column_device_view const d_targets,
                                             u_char const* d_first_bytes,
                                             size_type const* d_indices,
                                             size_type const* d_offsets,
                                             size_type unique_count,
                                             cudf::device_span<bool*> d_results)
{
  auto const str_idx     = static_cast<size_type>(cudf::detail::grid_1d::global_thread_id());
  auto const num_targets = d_targets.size();
  if (str_idx >= d_strings.size()) { return; }
  if (d_strings.is_null(str_idx)) { return; }
  auto const d_str = d_strings.element<string_view>(str_idx);

  // initialize output; the result of searching empty target is true
  for (auto target_idx = 0; target_idx < num_targets; ++target_idx) {
    auto const d_target            = d_targets.element<string_view>(target_idx);
    d_results[target_idx][str_idx] = d_target.empty();
  }

  // process each byte of the current string
  auto const last_ptr = d_first_bytes + unique_count;
  for (auto str_byte_idx = 0; str_byte_idx < d_str.size_bytes(); ++str_byte_idx) {
    // search for byte in first_bytes array
    auto const chr      = static_cast<u_char>(*(d_str.data() + str_byte_idx));
    auto const byte_ptr = thrust::lower_bound(thrust::seq, d_first_bytes, last_ptr, chr);
    // if not found, continue to next byte
    if ((byte_ptr == last_ptr) || (*byte_ptr != chr)) { continue; }
    // compute index of matched byte
    auto const offset_idx = static_cast<size_type>(thrust::distance(d_first_bytes, byte_ptr));
    auto map_idx          = d_offsets[offset_idx];
    auto const last_idx = (offset_idx + 1) < unique_count ? d_offsets[offset_idx + 1] : num_targets;
    // check for targets that begin with chr
    while (map_idx < last_idx) {
      auto const target_idx = d_indices[map_idx++];
      if (!d_results[target_idx][str_idx]) {  // not found before
        auto const d_target = d_targets.element<string_view>(target_idx);
        if ((d_str.size_bytes() - str_byte_idx) >= d_target.size_bytes()) {
          // first char already checked, only need to check the [2nd, end) chars
          bool found = true;
          for (auto i = 1; i < d_target.size_bytes() && found; i++) {
            if (*(d_str.data() + str_byte_idx + i) != *(d_target.data() + i)) { found = false; }
          }
          if (found) { d_results[target_idx][str_idx] = true; }
        }
      }
    }
  }
}

}  // namespace

std::unique_ptr<table> contains_multiple(strings_column_view const& input,
                                         strings_column_view const& targets,
                                         rmm::cuda_stream_view stream,
                                         rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(not targets.is_empty(), "Must specify at least one target string.");
  CUDF_EXPECTS(not targets.has_nulls(), "Target strings cannot be null");

  auto const num_targets = targets.size();

  auto const d_strings = column_device_view::create(input.parent(), stream);
  auto const d_targets = column_device_view::create(targets.parent(), stream);

  // copy the first byte of each target and sort them
  auto first_bytes = rmm::device_uvector<u_char>(targets.size(), stream);
  auto indices     = rmm::device_uvector<size_type>(targets.size(), stream);
  {
    auto tgt_itr = thrust::make_transform_iterator(
      d_targets->begin<string_view>(),
      cuda::proclaim_return_type<u_char>([] __device__(auto const& d_tgt) -> u_char {
        return d_tgt.empty() ? u_char{0} : static_cast<u_char>(d_tgt.data()[0]);
      }));
    auto count_itr = thrust::make_counting_iterator<size_type>(0);
    auto keys_out  = first_bytes.begin();
    auto vals_out  = indices.begin();
    auto cmp_op    = thrust::less();
    auto sv        = stream.value();

    std::size_t tmp_bytes = 0;
    hipcub::DeviceMergeSort::SortPairsCopy(
      nullptr, tmp_bytes, tgt_itr, count_itr, keys_out, vals_out, num_targets, cmp_op, sv);
    auto tmp_stg = rmm::device_buffer(tmp_bytes, stream);
    hipcub::DeviceMergeSort::SortPairsCopy(
      tmp_stg.data(), tmp_bytes, tgt_itr, count_itr, keys_out, vals_out, num_targets, cmp_op, sv);
  }

  // remove duplicates to speed up lower_bound
  auto offsets = rmm::device_uvector<size_type>(targets.size(), stream);
  thrust::sequence(rmm::exec_policy_nosync(stream), offsets.begin(), offsets.end());
  auto end = thrust::unique_by_key(
    rmm::exec_policy_nosync(stream), first_bytes.begin(), first_bytes.end(), offsets.begin());
  auto ucount = static_cast<size_type>(thrust::distance(first_bytes.begin(), end.first));

  // create output columns
  auto const results_iter = cudf::detail::make_counting_transform_iterator(0, [&](int i) {
    return make_numeric_column(data_type{type_id::BOOL8},
                               input.size(),
                               cudf::detail::copy_bitmask(input.parent(), stream, mr),
                               input.null_count(),
                               stream,
                               mr);
  });
  auto results_list =
    std::vector<std::unique_ptr<column>>(results_iter, results_iter + targets.size());
  auto device_results_list = [&] {
    auto host_results_pointer_iter =
      thrust::make_transform_iterator(results_list.begin(), [](auto const& results_column) {
        return results_column->mutable_view().template data<bool>();
      });
    auto host_results_pointers = std::vector<bool*>(
      host_results_pointer_iter, host_results_pointer_iter + results_list.size());
    return cudf::detail::make_device_uvector_async(host_results_pointers, stream, mr);
  }();

  constexpr cudf::thread_index_type block_size = 256;

  auto d_first_bytes = first_bytes.data();
  auto d_indices     = indices.data();
  auto d_offsets     = offsets.data();

  // Smaller strings perform better with a row per string
  bool const row_parallel = ((input.null_count() == input.size()) ||
                             ((input.chars_size(stream) / (input.size() - input.null_count())) <=
                              AVG_CHAR_BYTES_THRESHOLD));
  if (row_parallel) {
    cudf::detail::grid_1d grid{static_cast<cudf::thread_index_type>(input.size()), block_size};
    multi_contains_row_parallel<<<grid.num_blocks, grid.num_threads_per_block, 0, stream.value()>>>(
      *d_strings, *d_targets, d_first_bytes, d_indices, d_offsets, ucount, device_results_list);
  } else {
    cudf::detail::grid_1d grid{
      static_cast<cudf::thread_index_type>(input.size()) * cudf::detail::warp_size, block_size};
    auto shared_mem_size    = block_size * targets.size();
    size_type work_mem_size = 0;
    if (shared_mem_size > (16 * block_size)) {  // TODO: Need to find a good value for this
      shared_mem_size = 0;
      work_mem_size   = targets.size() * input.size() * cudf::detail::warp_size;
    }
    auto working_memory = rmm::device_uvector<bool>(work_mem_size, stream);
    multi_contains_warp_parallel<<<grid.num_blocks,
                                   grid.num_threads_per_block,
                                   shared_mem_size,
                                   stream.value()>>>(*d_strings,
                                                     *d_targets,
                                                     d_first_bytes,
                                                     d_indices,
                                                     d_offsets,
                                                     ucount,
                                                     working_memory.data(),
                                                     device_results_list);
  }

  return std::make_unique<table>(std::move(results_list));
}

}  // namespace detail

std::unique_ptr<table> contains_multiple(strings_column_view const& strings,
                                         strings_column_view const& targets,
                                         rmm::cuda_stream_view stream,
                                         rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::contains_multiple(strings, targets, stream, mr);
}

}  // namespace strings
}  // namespace cudf
