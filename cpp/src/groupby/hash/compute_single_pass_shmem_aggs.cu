#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "compute_single_pass_shmem_aggs.hpp"
#include "create_sparse_results_table.hpp"
#include "global_memory_aggregator.cuh"
#include "helpers.cuh"
#include "shared_memory_aggregator.cuh"
#include "single_pass_functors.cuh"

#include <cudf/aggregation.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/detail/utilities/cuda.hpp>
#include <cudf/detail/utilities/integer_utils.hpp>
#include <cudf/table/table_device_view.cuh>
#include <cudf/types.hpp>

#include <rmm/cuda_stream_view.hpp>

#include <hip/hip_cooperative_groups.h>

#include <cstddef>

namespace cudf::groupby::detail::hash {
namespace {
__device__ void calculate_columns_to_aggregate(cudf::size_type& col_start,
                                               cudf::size_type& col_end,
                                               cudf::mutable_table_device_view output_values,
                                               cudf::size_type output_size,
                                               std::byte** s_aggregates_pointer,
                                               bool** s_aggregates_valid_pointer,
                                               std::byte* shared_set_aggregates,
                                               cudf::size_type cardinality,
                                               cudf::size_type total_agg_size)
{
  col_start                       = col_end;
  cudf::size_type bytes_allocated = 0;

  auto const valid_col_size = round_to_multiple_of_8(sizeof(bool) * cardinality);

  while (bytes_allocated < total_agg_size && col_end < output_size) {
    auto const next_col_size =
      round_to_multiple_of_8(sizeof(output_values.column(col_end).type()) * cardinality);
    auto const next_col_total_size = next_col_size + valid_col_size;

    if (bytes_allocated + next_col_total_size > total_agg_size) { break; }

    s_aggregates_pointer[col_end] = shared_set_aggregates + bytes_allocated;
    s_aggregates_valid_pointer[col_end] =
      reinterpret_cast<bool*>(shared_set_aggregates + bytes_allocated + next_col_size);

    bytes_allocated += next_col_total_size;
    ++col_end;
  }
}

__device__ void initialize_shmem_aggregations(cudf::size_type col_start,
                                              cudf::size_type col_end,
                                              cudf::mutable_table_device_view output_values,
                                              std::byte** s_aggregates_pointer,
                                              bool** s_aggregates_valid_pointer,
                                              cudf::size_type cardinality,
                                              cudf::aggregation::Kind const* d_agg_kinds)
{
  for (auto col_idx = col_start; col_idx < col_end; col_idx++) {
    for (auto idx = threadIdx.x; idx < cardinality; idx += blockDim.x) {
      cudf::detail::dispatch_type_and_aggregation(output_values.column(col_idx).type(),
                                                  d_agg_kinds[col_idx],
                                                  initialize_shmem{},
                                                  s_aggregates_pointer[col_idx],
                                                  idx,
                                                  s_aggregates_valid_pointer[col_idx]);
    }
  }
}

__device__ void compute_pre_aggregrations(cudf::size_type col_start,
                                          cudf::size_type col_end,
                                          bitmask_type const* row_bitmask,
                                          bool skip_rows_with_nulls,
                                          cudf::table_device_view input_values,
                                          cudf::size_type num_input_rows,
                                          cudf::size_type* local_mapping_index,
                                          std::byte** s_aggregates_pointer,
                                          bool** s_aggregates_valid_pointer,
                                          cudf::aggregation::Kind const* d_agg_kinds)
{
  // TODO grid_1d utility
  for (auto cur_idx = blockDim.x * blockIdx.x + threadIdx.x; cur_idx < num_input_rows;
       cur_idx += blockDim.x * gridDim.x) {
    if (not skip_rows_with_nulls or cudf::bit_is_set(row_bitmask, cur_idx)) {
      auto map_idx = local_mapping_index[cur_idx];

      for (auto col_idx = col_start; col_idx < col_end; col_idx++) {
        auto input_col = input_values.column(col_idx);

        cudf::detail::dispatch_type_and_aggregation(input_col.type(),
                                                    d_agg_kinds[col_idx],
                                                    shmem_element_aggregator{},
                                                    s_aggregates_pointer[col_idx],
                                                    map_idx,
                                                    s_aggregates_valid_pointer[col_idx],
                                                    input_col,
                                                    cur_idx);
      }
    }
  }
}

__device__ void compute_final_aggregations(cudf::size_type col_start,
                                           cudf::size_type col_end,
                                           cudf::table_device_view input_values,
                                           cudf::mutable_table_device_view output_values,
                                           cudf::size_type cardinality,
                                           cudf::size_type* global_mapping_index,
                                           std::byte** s_aggregates_pointer,
                                           bool** s_aggregates_valid_pointer,
                                           cudf::aggregation::Kind const* d_agg_kinds)
{
  for (auto cur_idx = threadIdx.x; cur_idx < cardinality; cur_idx += blockDim.x) {
    auto out_idx = global_mapping_index[blockIdx.x * GROUPBY_SHM_MAX_ELEMENTS + cur_idx];
    for (auto col_idx = col_start; col_idx < col_end; col_idx++) {
      auto output_col = output_values.column(col_idx);

      cudf::detail::dispatch_type_and_aggregation(input_values.column(col_idx).type(),
                                                  d_agg_kinds[col_idx],
                                                  gmem_element_aggregator{},
                                                  output_col,
                                                  out_idx,
                                                  input_values.column(col_idx),
                                                  s_aggregates_pointer[col_idx],
                                                  cur_idx,
                                                  s_aggregates_valid_pointer[col_idx]);
    }
  }
}

/* Takes the local_mapping_index and global_mapping_index to compute
 * pre (shared) and final (global) aggregates*/
CUDF_KERNEL void single_pass_shmem_aggs_kernel(cudf::size_type num_rows,
                                               bitmask_type const* row_bitmask,
                                               bool skip_rows_with_nulls,
                                               cudf::size_type* local_mapping_index,
                                               cudf::size_type* global_mapping_index,
                                               cudf::size_type* block_cardinality,
                                               cudf::table_device_view input_values,
                                               cudf::mutable_table_device_view output_values,
                                               cudf::aggregation::Kind const* d_agg_kinds,
                                               cudf::size_type total_agg_size,
                                               cudf::size_type pointer_size)
{
  auto const block       = cooperative_groups::this_thread_block();
  auto const cardinality = block_cardinality[block.group_index().x];
  if (cardinality >= GROUPBY_CARDINALITY_THRESHOLD) { return; }

  auto const num_cols = output_values.num_columns();

  __shared__ cudf::size_type col_start;
  __shared__ cudf::size_type col_end;
  extern __shared__ std::byte shared_set_aggregates[];
  std::byte** s_aggregates_pointer =
    reinterpret_cast<std::byte**>(shared_set_aggregates + total_agg_size);
  bool** s_aggregates_valid_pointer =
    reinterpret_cast<bool**>(shared_set_aggregates + total_agg_size + pointer_size);

  if (block.thread_rank() == 0) {
    col_start = 0;
    col_end   = 0;
  }
  block.sync();

  while (col_end < num_cols) {
    if (block.thread_rank() == 0) {
      calculate_columns_to_aggregate(col_start,
                                     col_end,
                                     output_values,
                                     num_cols,
                                     s_aggregates_pointer,
                                     s_aggregates_valid_pointer,
                                     shared_set_aggregates,
                                     cardinality,
                                     total_agg_size);
    }
    block.sync();
    initialize_shmem_aggregations(col_start,
                                  col_end,
                                  output_values,
                                  s_aggregates_pointer,
                                  s_aggregates_valid_pointer,
                                  cardinality,
                                  d_agg_kinds);
    block.sync();
    compute_pre_aggregrations(col_start,
                              col_end,
                              row_bitmask,
                              skip_rows_with_nulls,
                              input_values,
                              num_rows,
                              local_mapping_index,
                              s_aggregates_pointer,
                              s_aggregates_valid_pointer,
                              d_agg_kinds);
    block.sync();
    compute_final_aggregations(col_start,
                               col_end,
                               input_values,
                               output_values,
                               cardinality,
                               global_mapping_index,
                               s_aggregates_pointer,
                               s_aggregates_valid_pointer,
                               d_agg_kinds);
    block.sync();
  }
}

constexpr size_t get_previous_multiple_of_8(size_t number) { return number / 8 * 8; }

}  // namespace

size_t available_shared_memory_size(cudf::size_type grid_size)
{
  auto const active_blocks_per_sm =
    cudf::util::div_rounding_up_safe(grid_size, cudf::detail::num_multiprocessors());

  size_t dynamic_shmem_size = 0;
  CUDF_CUDA_TRY(cudaOccupancyAvailableDynamicSMemPerBlock(
    &dynamic_shmem_size, single_pass_shmem_aggs_kernel, active_blocks_per_sm, GROUPBY_BLOCK_SIZE));
  return get_previous_multiple_of_8(0.5 * dynamic_shmem_size);
}

size_t shmem_agg_pointer_size(cudf::size_type num_cols) { return sizeof(void*) * num_cols; }

void compute_single_pass_shmem_aggs(cudf::size_type grid_size,
                                    cudf::size_type num_input_rows,
                                    bitmask_type const* row_bitmask,
                                    bool skip_rows_with_nulls,
                                    cudf::size_type* local_mapping_index,
                                    cudf::size_type* global_mapping_index,
                                    cudf::size_type* block_cardinality,
                                    cudf::table_device_view input_values,
                                    cudf::mutable_table_device_view output_values,
                                    cudf::aggregation::Kind const* d_agg_kinds,
                                    rmm::cuda_stream_view stream)
{
  auto const shmem_size = available_shared_memory_size(grid_size);
  // For each aggregation, need two pointers to arrays in shmem
  // One where the aggregation is performed, one indicating the validity of the aggregation
  auto const shmem_pointer_size = shmem_agg_pointer_size(output_values.num_columns());
  // The rest of shmem is utilized for the actual arrays in shmem
  CUDF_EXPECTS(shmem_size > shmem_pointer_size * 2,
               "No enough space for shared memory aggregations");
  auto const shmem_agg_size = shmem_size - shmem_pointer_size * 2;
  single_pass_shmem_aggs_kernel<<<grid_size, GROUPBY_BLOCK_SIZE, shmem_size, stream>>>(
    num_input_rows,
    row_bitmask,
    skip_rows_with_nulls,
    local_mapping_index,
    global_mapping_index,
    block_cardinality,
    input_values,
    output_values,
    d_agg_kinds,
    shmem_agg_size,
    shmem_pointer_size);
}
}  // namespace cudf::groupby::detail::hash
