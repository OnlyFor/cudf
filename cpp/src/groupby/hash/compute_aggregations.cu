#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "compute_aggregations.hpp"
#include "create_sparse_results_table.hpp"
#include "global_memory_aggregator.cuh"
#include "helpers.cuh"
#include "shared_memory_aggregator.cuh"
#include "single_pass_functors.cuh"

#include <cudf/aggregation.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/detail/utilities/cuda.hpp>
#include <cudf/detail/utilities/integer_utils.hpp>
#include <cudf/table/table_device_view.cuh>
#include <cudf/types.hpp>

#include <rmm/cuda_stream_view.hpp>

#include <hip/hip_cooperative_groups.h>

#include <cstddef>

namespace cudf::groupby::detail::hash {
namespace {
__device__ void calculate_columns_to_aggregate(int& col_start,
                                               int& col_end,
                                               cudf::mutable_table_device_view output_values,
                                               int num_input_cols,
                                               std::byte** s_aggregates_pointer,
                                               bool** s_aggregates_valid_pointer,
                                               std::byte* shared_set_aggregates,
                                               cudf::size_type cardinality,
                                               int total_agg_size)
{
  if (threadIdx.x == 0) {
    col_start           = col_end;
    int bytes_allocated = 0;
    int valid_col_size  = round_to_multiple_of_8(sizeof(bool) * cardinality);
    while ((bytes_allocated < total_agg_size) && (col_end < num_input_cols)) {
      int next_col_size =
        round_to_multiple_of_8(sizeof(output_values.column(col_end).type()) * cardinality);
      int next_col_total_size = valid_col_size + next_col_size;
      if (bytes_allocated + next_col_total_size > total_agg_size) { break; }
      s_aggregates_pointer[col_end] = shared_set_aggregates + bytes_allocated;
      s_aggregates_valid_pointer[col_end] =
        reinterpret_cast<bool*>(shared_set_aggregates + bytes_allocated + next_col_size);
      bytes_allocated += next_col_total_size;
      col_end++;
    }
  }
}

__device__ void initialize_shared_memory_aggregates(int col_start,
                                                    int col_end,
                                                    cudf::mutable_table_device_view output_values,
                                                    std::byte** s_aggregates_pointer,
                                                    bool** s_aggregates_valid_pointer,
                                                    cudf::size_type cardinality,
                                                    cudf::aggregation::Kind const* d_agg_kinds)
{
  for (auto col_idx = col_start; col_idx < col_end; col_idx++) {
    for (auto idx = threadIdx.x; idx < cardinality; idx += blockDim.x) {
      cudf::detail::dispatch_type_and_aggregation(output_values.column(col_idx).type(),
                                                  d_agg_kinds[col_idx],
                                                  initialize_shmem{},
                                                  s_aggregates_pointer[col_idx],
                                                  idx,
                                                  s_aggregates_valid_pointer[col_idx]);
    }
  }
}

__device__ void compute_pre_aggregrates(int col_start,
                                        int col_end,
                                        bitmask_type const* row_bitmask,
                                        bool skip_rows_with_nulls,
                                        cudf::table_device_view input_values,
                                        cudf::size_type num_input_rows,
                                        cudf::size_type* local_mapping_index,
                                        std::byte** s_aggregates_pointer,
                                        bool** s_aggregates_valid_pointer,
                                        cudf::aggregation::Kind const* d_agg_kinds)
{
  // TODO grid_1d utility
  for (auto cur_idx = blockDim.x * blockIdx.x + threadIdx.x; cur_idx < num_input_rows;
       cur_idx += blockDim.x * gridDim.x) {
    if (not skip_rows_with_nulls or cudf::bit_is_set(row_bitmask, cur_idx)) {
      auto map_idx = local_mapping_index[cur_idx];

      for (auto col_idx = col_start; col_idx < col_end; col_idx++) {
        auto input_col = input_values.column(col_idx);

        cudf::detail::dispatch_type_and_aggregation(input_col.type(),
                                                    d_agg_kinds[col_idx],
                                                    shmem_element_aggregator{},
                                                    s_aggregates_pointer[col_idx],
                                                    map_idx,
                                                    s_aggregates_valid_pointer[col_idx],
                                                    input_col,
                                                    cur_idx);
      }
    }
  }
}

__device__ void compute_final_aggregates(int col_start,
                                         int col_end,
                                         cudf::table_device_view input_values,
                                         cudf::mutable_table_device_view output_values,
                                         cudf::size_type cardinality,
                                         cudf::size_type* global_mapping_index,
                                         std::byte** s_aggregates_pointer,
                                         bool** s_aggregates_valid_pointer,
                                         cudf::aggregation::Kind const* d_agg_kinds)
{
  for (auto cur_idx = threadIdx.x; cur_idx < cardinality; cur_idx += blockDim.x) {
    auto out_idx = global_mapping_index[blockIdx.x * GROUPBY_SHM_MAX_ELEMENTS + cur_idx];
    for (auto col_idx = col_start; col_idx < col_end; col_idx++) {
      auto output_col = output_values.column(col_idx);

      cudf::detail::dispatch_type_and_aggregation(input_values.column(col_idx).type(),
                                                  d_agg_kinds[col_idx],
                                                  gmem_element_aggregator{},
                                                  output_col,
                                                  out_idx,
                                                  input_values.column(col_idx),
                                                  s_aggregates_pointer[col_idx],
                                                  cur_idx,
                                                  s_aggregates_valid_pointer[col_idx]);
    }
  }
}

/* Takes the local_mapping_index and global_mapping_index to compute
 * pre (shared) and final (global) aggregates*/
CUDF_KERNEL void compute_aggs_kernel(cudf::size_type num_rows,
                                     bitmask_type const* row_bitmask,
                                     bool skip_rows_with_nulls,
                                     cudf::size_type* local_mapping_index,
                                     cudf::size_type* global_mapping_index,
                                     cudf::size_type* block_cardinality,
                                     cudf::table_device_view input_values,
                                     cudf::mutable_table_device_view output_values,
                                     cudf::aggregation::Kind const* d_agg_kinds,
                                     int total_agg_size,
                                     int pointer_size)
{
  auto const block       = cooperative_groups::this_thread_block();
  auto const cardinality = block_cardinality[block.group_index().x];
  if (cardinality >= GROUPBY_CARDINALITY_THRESHOLD) { return; }

  auto const num_cols = output_values.num_columns();

  __shared__ int col_start;
  __shared__ int col_end;
  extern __shared__ std::byte shared_set_aggregates[];
  std::byte** s_aggregates_pointer =
    reinterpret_cast<std::byte**>(shared_set_aggregates + total_agg_size);
  bool** s_aggregates_valid_pointer =
    reinterpret_cast<bool**>(shared_set_aggregates + total_agg_size + pointer_size);

  if (block.thread_rank() == 0) {
    col_start = 0;
    col_end   = 0;
  }
  block.sync();

  while (col_end < num_cols) {
    calculate_columns_to_aggregate(col_start,
                                   col_end,
                                   output_values,
                                   num_cols,
                                   s_aggregates_pointer,
                                   s_aggregates_valid_pointer,
                                   shared_set_aggregates,
                                   cardinality,
                                   total_agg_size);
    block.sync();
    initialize_shared_memory_aggregates(col_start,
                                        col_end,
                                        output_values,
                                        s_aggregates_pointer,
                                        s_aggregates_valid_pointer,
                                        cardinality,
                                        d_agg_kinds);
    block.sync();
    compute_pre_aggregrates(col_start,
                            col_end,
                            row_bitmask,
                            skip_rows_with_nulls,
                            input_values,
                            num_rows,
                            local_mapping_index,
                            s_aggregates_pointer,
                            s_aggregates_valid_pointer,
                            d_agg_kinds);
    block.sync();
    compute_final_aggregates(col_start,
                             col_end,
                             input_values,
                             output_values,
                             cardinality,
                             global_mapping_index,
                             s_aggregates_pointer,
                             s_aggregates_valid_pointer,
                             d_agg_kinds);
    block.sync();
  }
}

constexpr size_t get_previous_multiple_of_8(size_t number) { return number / 8 * 8; }

}  // namespace

std::pair<bool, size_t> can_use_shmem_aggs(int grid_size) noexcept
{
  auto const active_blocks_per_sm =
    cudf::util::div_rounding_up_safe(grid_size, cudf::detail::num_multiprocessors());

  size_t dynamic_shmem_size = 0;

  auto const status = cudaOccupancyAvailableDynamicSMemPerBlock(
    &dynamic_shmem_size, compute_aggs_kernel, active_blocks_per_sm, GROUPBY_BLOCK_SIZE);
  auto const success = status == hipSuccess;
  if (!success) { hipGetLastError(); }

  return {success, get_previous_multiple_of_8(0.5 * dynamic_shmem_size)};
}

void compute_aggregations(int grid_size,
                          cudf::size_type num_input_rows,
                          bitmask_type const* row_bitmask,
                          bool skip_rows_with_nulls,
                          cudf::size_type* local_mapping_index,
                          cudf::size_type* global_mapping_index,
                          cudf::size_type* block_cardinality,
                          cudf::table_device_view input_values,
                          cudf::mutable_table_device_view output_values,
                          cudf::aggregation::Kind const* d_agg_kinds,
                          size_t shmem_size,
                          rmm::cuda_stream_view stream)
{
  // For each aggregation, need two pointers to arrays in shmem
  // One where the aggregation is performed, one indicating the validity of the aggregation
  auto const shmem_agg_pointer_size =
    round_to_multiple_of_8(sizeof(std::byte*) * output_values.num_columns());
  // The rest of shmem is utilized for the actual arrays in shmem
  auto const shmem_agg_size = shmem_size - shmem_agg_pointer_size * 2;
  compute_aggs_kernel<<<grid_size, GROUPBY_BLOCK_SIZE, shmem_size, stream>>>(
    num_input_rows,
    row_bitmask,
    skip_rows_with_nulls,
    local_mapping_index,
    global_mapping_index,
    block_cardinality,
    input_values,
    output_values,
    d_agg_kinds,
    shmem_agg_size,
    shmem_agg_pointer_size);
}
}  // namespace cudf::groupby::detail::hash
