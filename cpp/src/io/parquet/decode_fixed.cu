#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "page_data.cuh"
#include "page_decode.cuh"
#include "parquet_gpu.hpp"
#include "rle_stream.cuh"

#include <cudf/detail/utilities/cuda.cuh>

namespace cudf::io::parquet::detail {

namespace {

// Unlike cub's algorithm, this provides warp-wide and block-wide results simultaneously.
// Also, this provides the ability to compute warp_bits & lane_mask manually, which we need for
// lists.
struct block_scan_results {
  uint32_t warp_bits;
  int thread_count_within_warp;
  int warp_count;

  int thread_count_within_block;
  int block_count;
};

template <int decode_block_size>
__device__ inline static void scan_block_exclusive_sum(int thread_bit, block_scan_results& results)
{
  int const t = threadIdx.x;
  int const warp_index     = t / cudf::detail::warp_size;
  int const warp_lane      = t % cudf::detail::warp_size;
  uint32_t const lane_mask = (uint32_t(1) << warp_lane) - 1;

  uint32_t warp_bits = ballot(thread_bit);
  scan_block_exclusive_sum<decode_block_size>(warp_bits, warp_lane, warp_index, lane_mask, results);
}

template <int decode_block_size>
__device__ inline static void scan_block_exclusive_sum(uint32_t warp_bits, int warp_lane, int warp_index, uint32_t lane_mask, block_scan_results& results)
{
  //Compute # warps
  constexpr int num_warps = decode_block_size / cudf::detail::warp_size;
  
  //Compute the warp-wide results
  results.warp_bits                = warp_bits;
  results.warp_count               = __popc(results.warp_bits);
  results.thread_count_within_warp = __popc(results.warp_bits & lane_mask);

  //Share the warp counts amongst the block threads
  __shared__ int warp_counts[num_warps];
  if (warp_lane == 0) { warp_counts[warp_index] = results.warp_count; }
  __syncthreads();

  //Compute block-wide results
  results.block_count               = 0;
  results.thread_count_within_block = results.thread_count_within_warp;
  for (int warp_idx = 0; warp_idx < num_warps; ++warp_idx) {
    results.block_count += warp_counts[warp_idx];
    if (warp_idx < warp_index) { results.thread_count_within_block += warp_counts[warp_idx]; }
  }
}

template <int block_size, bool has_lists_t, typename state_buf>
__device__ inline void gpuDecodeFixedWidthValues(
  page_state_s* s, state_buf* const sb, int start, int end, int t)
{
  constexpr int num_warps      = block_size / cudf::detail::warp_size;
  constexpr int max_batch_size = num_warps * cudf::detail::warp_size;

  PageNestingDecodeInfo* nesting_info_base = s->nesting_info;
  int const dtype                          = s->col.physical_type;

  int const leaf_level_index = s->col.max_nesting_depth - 1;
  uint32_t dtype_len = s->dtype_len;
  auto const data_out = nesting_info_base[leaf_level_index].data_out;
  uint32_t const skipped_leaf_values = s->page.skipped_leaf_values;

  static constexpr bool enable_print = false;
  static constexpr bool enable_print_range_error = false;
//  static constexpr bool enable_print_large_list = true;

  if constexpr (enable_print) {
    if(t == 0) { printf("DECODE VALUES: start %d, end %d, first_row %d, leaf_level_index %d, dtype_len %u, "
      "data_out %p, dict_base %p, dict_size %d, dict_bits %d, dict_val %d, data_start %p, skipped_leaf_values %u, input_row_count %d\n", 
      start, end, s->first_row, leaf_level_index, dtype_len, data_out, s->dict_base, s->dict_bits, s->dict_val, 
      s->dict_size, s->data_start, skipped_leaf_values, s->input_row_count);
    }
  }

  // decode values
  int pos = start;
  while (pos < end) {
    int const batch_size = min(max_batch_size, end - pos);

    int const target_pos = pos + batch_size;
    int src_pos    = pos + t;

    // the position in the output column/buffer
//Index from rolling buffer of values (which doesn't include nulls) to final array (which includes gaps for nulls)
    auto offset = sb->nz_idx[rolling_index<state_buf::nz_buf_size>(src_pos)];
    int dst_pos = offset;
    if constexpr (!has_lists_t) {
      dst_pos -= s->first_row;
    }

    if constexpr (has_lists_t && enable_print_range_error) {
      if((dst_pos < 0) && (src_pos < target_pos)) { printf("WHOA: decode dst_pos %d out of bounds, src_pos %d, start %d\n", dst_pos, src_pos, start); }
    }

    int dict_idx = rolling_index<state_buf::dict_buf_size>(src_pos + skipped_leaf_values);
    int dict_pos = sb->dict_idx[dict_idx];
    if constexpr (enable_print) {
      if(t == 0) { 
        printf("DECODE OFFSETS: pos %d, src_pos %d, offset %d, dst_pos %d, target_pos %d, dict_idx %d, dict_pos %d\n", 
          pos, src_pos, offset, dst_pos, target_pos, dict_idx, dict_pos);
      }
    }

    // target_pos will always be properly bounded by num_rows, but dst_pos may be negative (values
    // before first_row) in the flat hierarchy case.
    if (src_pos < target_pos && dst_pos >= 0) {
      // nesting level that is storing actual leaf values

      // src_pos represents the logical row position we want to read from. But in the case of
      // nested hierarchies (lists), there is no 1:1 mapping of rows to values.  So our true read position
      // has to take into account the # of values we have to skip in the page to get to the
      // desired logical row.  For flat hierarchies, skipped_leaf_values will always be 0.
      if constexpr (has_lists_t) {
        src_pos += skipped_leaf_values;
      }

      void* dst = data_out + static_cast<size_t>(dst_pos) * dtype_len;
      if constexpr (enable_print) {
        if(dst_pos == 0) {
          printf("WRITTEN TO dst_pos ZERO: t %d, data_out %p, dst %p, src_pos %d, dict_idx %d, dict_pos %d, dict_base %p\n", 
            t, data_out, dst, src_pos, dict_idx, dict_pos, s->dict_base);
        }
      }

      if (s->col.logical_type.has_value() && s->col.logical_type->type == LogicalType::DECIMAL) {
        switch (dtype) {
          case INT32: gpuOutputFast(s, sb, src_pos, static_cast<uint32_t*>(dst)); break;
          case INT64: gpuOutputFast(s, sb, src_pos, static_cast<uint2*>(dst)); break;
          default:
            if (s->dtype_len_in <= sizeof(int32_t)) {
              gpuOutputFixedLenByteArrayAsInt(s, sb, src_pos, static_cast<int32_t*>(dst));
            } else if (s->dtype_len_in <= sizeof(int64_t)) {
              gpuOutputFixedLenByteArrayAsInt(s, sb, src_pos, static_cast<int64_t*>(dst));
            } else {
              gpuOutputFixedLenByteArrayAsInt(s, sb, src_pos, static_cast<__int128_t*>(dst));
            }
            break;
        }
      } else if (dtype == INT96) {
        gpuOutputInt96Timestamp(s, sb, src_pos, static_cast<int64_t*>(dst));
      } else if (dtype_len == 8) {
        if (s->dtype_len_in == 4) {
          // Reading INT32 TIME_MILLIS into 64-bit DURATION_MILLISECONDS
          // TIME_MILLIS is the only duration type stored as int32:
          // https://github.com/apache/parquet-format/blob/master/LogicalTypes.md#deprecated-time-convertedtype
          gpuOutputFast(s, sb, src_pos, static_cast<uint32_t*>(dst));
        } else if (s->ts_scale) {
          gpuOutputInt64Timestamp(s, sb, src_pos, static_cast<int64_t*>(dst));
        } else {
          gpuOutputFast(s, sb, src_pos, static_cast<uint2*>(dst));
        }
      } else if (dtype_len == 4) {
        gpuOutputFast(s, sb, src_pos, static_cast<uint32_t*>(dst));
      } else {
        gpuOutputGeneric(s, sb, src_pos, static_cast<uint8_t*>(dst), dtype_len);
      }
/*
      if constexpr (enable_print_large_list) {
        if (dtype == INT32) {
          int value_stored = *static_cast<uint32_t*>(dst);
          int overall_index = blockIdx.x * 20000 * 4 + src_pos;
          if((overall_index % 1024) != value_stored) {
            printf("WHOA BAD VALUE: WROTE %d to %d!\n", value_stored, overall_index);
          }
        }
      }
      */
    }

    pos += batch_size;
  }
}

template <int block_size, bool has_lists_t, typename state_buf>
struct decode_fixed_width_values_func {
  __device__ inline void operator()(page_state_s* s, state_buf* const sb, int start, int end, int t)
  {
    gpuDecodeFixedWidthValues<block_size, has_lists_t, state_buf>(s, sb, start, end, t);
  }
};

template <int block_size, bool has_lists_t, typename state_buf>
__device__ inline void gpuDecodeFixedWidthSplitValues(
  page_state_s* s, state_buf* const sb, int start, int end, int t)
{
  using cudf::detail::warp_size;
  constexpr int num_warps      = block_size / warp_size;
  constexpr int max_batch_size = num_warps * warp_size;

  PageNestingDecodeInfo* nesting_info_base = s->nesting_info;
  int const dtype                          = s->col.physical_type;
  auto const data_len                      = thrust::distance(s->data_start, s->data_end);
  auto const num_values                    = data_len / s->dtype_len_in;
  uint32_t const skipped_leaf_values = s->page.skipped_leaf_values;

  // decode values
  int pos = start;
  while (pos < end) {
    int const batch_size = min(max_batch_size, end - pos);

    int const target_pos = pos + batch_size;
    int src_pos    = pos + t;

    // the position in the output column/buffer
    int dst_pos = sb->nz_idx[rolling_index<state_buf::nz_buf_size>(src_pos)];
    if constexpr (!has_lists_t) {
      dst_pos -= s->first_row;
    }

    // target_pos will always be properly bounded by num_rows, but dst_pos may be negative (values
    // before first_row) in the flat hierarchy case.
    if (src_pos < target_pos && dst_pos >= 0) {
      // nesting level that is storing actual leaf values
      int const leaf_level_index = s->col.max_nesting_depth - 1;

      // src_pos represents the logical row position we want to read from. But in the case of
      // nested hierarchies (lists), there is no 1:1 mapping of rows to values.  So our true read position
      // has to take into account the # of values we have to skip in the page to get to the
      // desired logical row.  For flat hierarchies, skipped_leaf_values will always be 0.
      if constexpr (has_lists_t) {
        src_pos += skipped_leaf_values;
      }

      uint32_t dtype_len = s->dtype_len;
      uint8_t const* src = s->data_start + src_pos;
      uint8_t* dst =
        nesting_info_base[leaf_level_index].data_out + static_cast<size_t>(dst_pos) * dtype_len;
      auto const is_decimal =
        s->col.logical_type.has_value() and s->col.logical_type->type == LogicalType::DECIMAL;

      // Note: non-decimal FIXED_LEN_BYTE_ARRAY will be handled in the string reader
      if (is_decimal) {
        switch (dtype) {
          case INT32: gpuOutputByteStreamSplit<int32_t>(dst, src, num_values); break;
          case INT64: gpuOutputByteStreamSplit<int64_t>(dst, src, num_values); break;
          case FIXED_LEN_BYTE_ARRAY:
            if (s->dtype_len_in <= sizeof(int32_t)) {
              gpuOutputSplitFixedLenByteArrayAsInt(
                reinterpret_cast<int32_t*>(dst), src, num_values, s->dtype_len_in);
              break;
            } else if (s->dtype_len_in <= sizeof(int64_t)) {
              gpuOutputSplitFixedLenByteArrayAsInt(
                reinterpret_cast<int64_t*>(dst), src, num_values, s->dtype_len_in);
              break;
            } else if (s->dtype_len_in <= sizeof(__int128_t)) {
              gpuOutputSplitFixedLenByteArrayAsInt(
                reinterpret_cast<__int128_t*>(dst), src, num_values, s->dtype_len_in);
              break;
            }
            // unsupported decimal precision
            [[fallthrough]];

          default: s->set_error_code(decode_error::UNSUPPORTED_ENCODING);
        }
      } else if (dtype_len == 8) {
        if (s->dtype_len_in == 4) {
          // Reading INT32 TIME_MILLIS into 64-bit DURATION_MILLISECONDS
          // TIME_MILLIS is the only duration type stored as int32:
          // https://github.com/apache/parquet-format/blob/master/LogicalTypes.md#deprecated-time-convertedtype
          gpuOutputByteStreamSplit<int32_t>(dst, src, num_values);
          // zero out most significant bytes
          memset(dst + 4, 0, 4);
        } else if (s->ts_scale) {
          gpuOutputSplitInt64Timestamp(
            reinterpret_cast<int64_t*>(dst), src, num_values, s->ts_scale);
        } else {
          gpuOutputByteStreamSplit<int64_t>(dst, src, num_values);
        }
      } else if (dtype_len == 4) {
        gpuOutputByteStreamSplit<int32_t>(dst, src, num_values);
      } else {
        s->set_error_code(decode_error::UNSUPPORTED_ENCODING);
      }
    }

    pos += batch_size;
  }
}

template <int block_size, bool has_lists_t, typename state_buf>
struct decode_fixed_width_split_values_func {
  __device__ inline void operator()(page_state_s* s, state_buf* const sb, int start, int end, int t)
  {
    gpuDecodeFixedWidthSplitValues<block_size, has_lists_t, state_buf>(s, sb, start, end, t);
  }
};

template <int decode_block_size, typename level_t, typename state_buf>
static __device__ int gpuUpdateValidityAndRowIndicesNested(
  int32_t target_value_count, page_state_s* s, state_buf* sb, level_t const* const def, int t)
{
  constexpr int num_warps      = decode_block_size / cudf::detail::warp_size;
  constexpr int max_batch_size = num_warps * cudf::detail::warp_size;

  // how many (input) values we've processed in the page so far
  int value_count = s->input_value_count;

  // cap by last row so that we don't process any rows past what we want to output.
  int const first_row                 = s->first_row;
  int const last_row                  = first_row + s->num_rows;
  int const capped_target_value_count = min(target_value_count, last_row);

  static constexpr bool enable_print = false;
  if constexpr (enable_print) {
    if (t == 0) { printf("NESTED: s->input_value_count %d, first_row %d, last_row %d, target_value_count %d, capped_target_value_count %d\n", 
      s->input_value_count, first_row, last_row, target_value_count, capped_target_value_count); }
  }

  int const row_index_lower_bound = s->row_index_lower_bound;

  int const max_depth       = s->col.max_nesting_depth - 1;
  auto& max_depth_ni        = s->nesting_info[max_depth];
  int max_depth_valid_count = max_depth_ni.valid_count;

  __syncthreads();

  while (value_count < capped_target_value_count) {
    if constexpr (enable_print) {
      if(t == 0) { printf("NESTED VALUE COUNT: %d\n", value_count); }
    }
    int const batch_size = min(max_batch_size, capped_target_value_count - value_count);

    // definition level
    int d = 1;
    if (t >= batch_size) {
      d = -1;
    } else if (def) {
      d = static_cast<int>(def[rolling_index<state_buf::nz_buf_size>(value_count + t)]);
    }

    int const thread_value_count = t;
    int const block_value_count  = batch_size;

    // compute our row index, whether we're in row bounds, and validity
    int const row_index           = thread_value_count + value_count;
    int const in_row_bounds       = (row_index >= row_index_lower_bound) && (row_index < last_row);
    int const in_write_row_bounds = ballot(row_index >= first_row && row_index < last_row);
    int const write_start = __ffs(in_write_row_bounds) - 1;  // first bit in the warp to store

    if constexpr (enable_print) {
      if(t == 0) { printf("NESTED ROWS: row_index %d, row_index_lower_bound %d, last_row %d, in_row_bounds %d\n", 
        row_index, row_index_lower_bound, last_row, in_row_bounds); }
    }

    // iterate by depth
    for (int d_idx = 0; d_idx <= max_depth; d_idx++) {
      auto& ni = s->nesting_info[d_idx];

      int const is_valid = ((d >= ni.max_def_level) && in_row_bounds) ? 1 : 0;

      // thread and block validity count
      using block_scan = hipcub::BlockScan<int, decode_block_size>;
      __shared__ typename block_scan::TempStorage scan_storage;
      int thread_valid_count, block_valid_count;
      block_scan(scan_storage).ExclusiveSum(is_valid, thread_valid_count, block_valid_count);

      // validity is processed per-warp
      //
      // nested schemas always read and write to the same bounds (that is, read and write
      // positions are already pre-bounded by first_row/num_rows). flat schemas will start reading
      // at the first value, even if that is before first_row, because we cannot trivially jump to
      // the correct position to start reading. since we are about to write the validity vector
      // here we need to adjust our computed mask to take into account the write row bounds.
      int warp_null_count = 0;
      if (ni.valid_map != nullptr) {
        uint32_t const warp_validity_mask = ballot(is_valid);
        // lane 0 from each warp writes out validity
        if ((write_start >= 0) && ((t % cudf::detail::warp_size) == 0)) {
          int const valid_map_offset = ni.valid_map_offset;
          int const vindex     = value_count + thread_value_count;  // absolute input value index
          int const bit_offset = (valid_map_offset + vindex + write_start) -
                                 first_row;  // absolute bit offset into the output validity map
          int const write_end =
            cudf::detail::warp_size - __clz(in_write_row_bounds);  // last bit in the warp to store
          int const bit_count = write_end - write_start;
          warp_null_count     = bit_count - __popc(warp_validity_mask >> write_start);

          store_validity(bit_offset, ni.valid_map, warp_validity_mask >> write_start, bit_count);
        }
      }

      // sum null counts. we have to do it this way instead of just incrementing by (value_count -
      // valid_count) because valid_count also includes rows that potentially start before our row
      // bounds. if we could come up with a way to clean that up, we could remove this and just
      // compute it directly at the end of the kernel.
      size_type const block_null_count =
        cudf::detail::single_lane_block_sum_reduce<decode_block_size, 0>(warp_null_count);
      if (t == 0) { ni.null_count += block_null_count; }

      // if this is valid and we're at the leaf, output dst_pos
      if (d_idx == max_depth) {
        if (is_valid) {
          int const dst_pos = value_count + thread_value_count;
          int const src_pos = max_depth_valid_count + thread_valid_count;
          sb->nz_idx[rolling_index<state_buf::nz_buf_size>(src_pos)] = dst_pos;
          if constexpr (enable_print) {
            if(t == 0) {printf("NESTED STORE: first_row %d, row_index %d dst_pos %d, src_pos %d\n", 
              first_row, row_index, dst_pos, src_pos);}
          }
        }
        // update stuff
        max_depth_valid_count += block_valid_count;
      }

    }  // end depth loop

    value_count += block_value_count;
  }  // end loop

  if (t == 0) {
    // update valid value count for decoding and total # of values we've processed
    max_depth_ni.valid_count = max_depth_valid_count;
    s->nz_count              = max_depth_valid_count;
    s->input_value_count     = value_count;
    s->input_row_count       = value_count;
  }

  return max_depth_valid_count;
}

template <int decode_block_size, typename level_t, typename state_buf>
static __device__ int gpuUpdateValidityAndRowIndicesFlat(
  int32_t target_value_count, page_state_s* s, state_buf* sb, level_t const* const def, int t)
{
  constexpr int num_warps      = decode_block_size / cudf::detail::warp_size;
  constexpr int max_batch_size = num_warps * cudf::detail::warp_size;

  auto& ni = s->nesting_info[0];

  // how many (input) values we've processed in the page so far
  int value_count = s->input_value_count;
  int valid_count = ni.valid_count;

  // cap by last row so that we don't process any rows past what we want to output.
  int const first_row                 = s->first_row;
  int const last_row                  = first_row + s->num_rows;
  int const capped_target_value_count = min(target_value_count, last_row);

  static constexpr bool enable_print = false;
  if constexpr (enable_print) {
    if (t == 0) { printf("FLAT: s->input_value_count %d, first_row %d, last_row %d, target_value_count %d, capped_target_value_count %d\n", 
      s->input_value_count, first_row, last_row, target_value_count, capped_target_value_count); }
  }

  int const valid_map_offset      = ni.valid_map_offset;
  int const row_index_lower_bound = s->row_index_lower_bound;

  __syncthreads();

  while (value_count < capped_target_value_count) {
    if constexpr (enable_print) {
      if(t == 0) { printf("FLAT VALUE COUNT: %d\n", value_count); }
    }

    int const batch_size = min(max_batch_size, capped_target_value_count - value_count);

    int const thread_value_count = t;
    int const block_value_count  = batch_size;

    // compute our row index, whether we're in row bounds, and validity
    int const row_index     = thread_value_count + value_count;
    int const in_row_bounds = (row_index >= row_index_lower_bound) && (row_index < last_row);

    // use definition level & row bounds to determine if is valid
    int is_valid;
    if (t >= batch_size) {
      is_valid = 0;
    } else if (def) {
      int const def_level =
        static_cast<int>(def[rolling_index<state_buf::nz_buf_size>(value_count + t)]);
      is_valid = ((def_level > 0) && in_row_bounds) ? 1 : 0;
    } else {
      is_valid = in_row_bounds;
    }

    // thread and block validity count
    using block_scan = hipcub::BlockScan<int, decode_block_size>;
    __shared__ typename block_scan::TempStorage scan_storage;
    int thread_valid_count, block_valid_count;
    block_scan(scan_storage).ExclusiveSum(is_valid, thread_valid_count, block_valid_count);
    uint32_t const warp_validity_mask = ballot(is_valid);

    // validity is processed per-warp
    //
    // nested schemas always read and write to the same bounds (that is, read and write
    // positions are already pre-bounded by first_row/num_rows). flat schemas will start reading
    // at the first value, even if that is before first_row, because we cannot trivially jump to
    // the correct position to start reading. since we are about to write the validity vector
    // here we need to adjust our computed mask to take into account the write row bounds.
    int const in_write_row_bounds = ballot(row_index >= first_row && row_index < last_row);
    int const write_start = __ffs(in_write_row_bounds) - 1;  // first bit in the warp to store
    int warp_null_count   = 0;
    // lane 0 from each warp writes out validity
    if ((write_start >= 0) && ((t % cudf::detail::warp_size) == 0)) {
      int const vindex     = value_count + thread_value_count;  // absolute input value index
      int const bit_offset = (valid_map_offset + vindex + write_start) -
                             first_row;  // absolute bit offset into the output validity map
      int const write_end =
        cudf::detail::warp_size - __clz(in_write_row_bounds);  // last bit in the warp to store
      int const bit_count = write_end - write_start;
      warp_null_count     = bit_count - __popc(warp_validity_mask >> write_start);

      store_validity(bit_offset, ni.valid_map, warp_validity_mask >> write_start, bit_count);
    }

    // sum null counts. we have to do it this way instead of just incrementing by (value_count -
    // valid_count) because valid_count also includes rows that potentially start before our row
    // bounds. if we could come up with a way to clean that up, we could remove this and just
    // compute it directly at the end of the kernel.
    size_type const block_null_count =
      cudf::detail::single_lane_block_sum_reduce<decode_block_size, 0>(warp_null_count);
    if (t == 0) { ni.null_count += block_null_count; }

    // output offset
    if (is_valid) {
      int const dst_pos = value_count + thread_value_count;
      int const src_pos = valid_count + thread_valid_count;

      sb->nz_idx[rolling_index<state_buf::nz_buf_size>(src_pos)] = dst_pos;
    }

    // update stuff
    value_count += block_value_count;
    valid_count += block_valid_count;
  }

  if (t == 0) {
    // update valid value count for decoding and total # of values we've processed
    ni.valid_count       = valid_count;
    ni.value_count       = value_count;
    s->nz_count          = valid_count;
    s->input_value_count = value_count;
    s->input_row_count   = value_count;
  }

  return valid_count;
}

template <int decode_block_size, typename state_buf>
static __device__ int gpuUpdateValidityAndRowIndicesNonNullable(int32_t target_value_count,
                                                                page_state_s* s,
                                                                state_buf* sb,
                                                                int t)
{
  constexpr int num_warps      = decode_block_size / cudf::detail::warp_size;
  constexpr int max_batch_size = num_warps * cudf::detail::warp_size;

  // cap by last row so that we don't process any rows past what we want to output.
  int const first_row                 = s->first_row;
  int const last_row                  = first_row + s->num_rows;
  int const capped_target_value_count = min(target_value_count, last_row);
  int const row_index_lower_bound     = s->row_index_lower_bound;

  // how many (input) values we've processed in the page so far
  int value_count = s->input_value_count;

  int const max_depth = s->col.max_nesting_depth - 1;
  auto& ni            = s->nesting_info[max_depth];
  int valid_count     = ni.valid_count;

  __syncthreads();

  while (value_count < capped_target_value_count) {
    int const batch_size = min(max_batch_size, capped_target_value_count - value_count);

    int const thread_value_count = t;
    int const block_value_count  = batch_size;

    // compute our row index, whether we're in row bounds, and validity
    int const row_index     = thread_value_count + value_count;
    int const in_row_bounds = (row_index >= row_index_lower_bound) && (row_index < last_row);

    int const is_valid           = in_row_bounds;
    int const thread_valid_count = thread_value_count;
    int const block_valid_count  = block_value_count;

    // if this is valid and we're at the leaf, output dst_pos
    if (is_valid) {
      // for non-list types, the value count is always the same across
      int const dst_pos = value_count + thread_value_count;
      int const src_pos = valid_count + thread_valid_count;

      sb->nz_idx[rolling_index<state_buf::nz_buf_size>(src_pos)] = dst_pos;
    }

    // update stuff
    value_count += block_value_count;
    valid_count += block_valid_count;
  }  // end loop

  if (t == 0) {
    // update valid value count for decoding and total # of values we've processed
    ni.valid_count       = valid_count;
    ni.value_count       = value_count;
    s->nz_count          = valid_count;
    s->input_value_count = value_count;
    s->input_row_count   = value_count;
  }

  return valid_count;
}

template <int decode_block_size, bool nullable, typename level_t, typename state_buf>
static __device__ int gpuUpdateValidityAndRowIndicesLists(
  int32_t target_value_count, page_state_s* s, state_buf* sb, level_t const* const def, 
  level_t const* const rep, int t)
{
  //What is the output of this? Validity bits and offsets to list starts
  constexpr int num_warps      = decode_block_size / cudf::detail::warp_size;
  constexpr int max_batch_size = num_warps * cudf::detail::warp_size;

  // how many (input) values we've processed in the page so far, prior to this loop iteration
  int value_count = s->input_value_count;

  static constexpr bool enable_print = false;
  static constexpr bool enable_print_range_error = false;
  static constexpr bool enable_print_large_list = false;

  // how many rows we've processed in the page so far
  int input_row_count = s->input_row_count;
  if constexpr (enable_print) {
    if (t == 0) { printf("value_count %d, input_row_count %d\n", value_count, input_row_count); }
  }

  // cap by last row so that we don't process any rows past what we want to output.
  int const first_row                 = s->first_row;
  int const last_row                  = first_row + s->num_rows;
  if constexpr (enable_print) {
    if (t == 0) { printf("LIST s->input_value_count %d, first_row %d, last_row %d, target_value_count %d\n", 
      s->input_value_count, first_row, last_row, target_value_count); }
  }

  int const row_index_lower_bound = s->row_index_lower_bound;
  int const max_depth = s->col.max_nesting_depth - 1;
  int max_depth_valid_count = s->nesting_info[max_depth].valid_count;

  __syncthreads();
  
  int const warp_index     = t / cudf::detail::warp_size;
  int const warp_lane      = t % cudf::detail::warp_size;
  bool const is_first_lane = (warp_lane == 0);

  while (value_count < target_value_count) {

    if constexpr (enable_print) {
      if(t == 0) { printf("LIST VALUE COUNT: %d\n", value_count); }
    }
    bool const within_batch = value_count + t < target_value_count;

    // get definition level, use repitition level to get start/end depth
    // different for each thread, as each thread has a different r/d
    int def_level = -1, start_depth = -1, end_depth = -1;
    if (within_batch) {
      int const index = rolling_index<state_buf::nz_buf_size>(value_count + t);
      int rep_level = static_cast<int>(rep[index]);
      if constexpr (nullable) {
        def_level = static_cast<int>(def[index]);
        end_depth = s->nesting_info[def_level].end_depth;
      } else {
        end_depth = max_depth;
      }

      //computed by generate_depth_remappings()
      if constexpr (enable_print || enable_print_range_error) {
        if((rep_level < 0) || (rep_level > max_depth)) {
          printf("WHOA: rep level %d out of bounds %d!\n", rep_level, max_depth);
        }
        if(nullable && ((def_level < 0)/* || (def_level > (max_depth + 1)) */ )) {
          printf("WHOA: def level %d out of bounds (max_depth %d) (index %d)!\n", def_level, max_depth, index);
        }
      }

      start_depth = s->nesting_info[rep_level].start_depth;
      if constexpr (enable_print || enable_print_range_error) {
        if((start_depth < 0) || (start_depth > (max_depth + 1))) {
          printf("WHOA: start_depth %d out of bounds (max_depth %d) (index %d)!\n", start_depth, max_depth, index);
        }
        if((end_depth < 0) || (end_depth > (max_depth + 1))) {
          printf("WHOA: end_depth %d out of bounds (max_depth %d) (index %d)!\n", end_depth, max_depth, index);
        }
      }
      if constexpr (enable_print) {
        if (t == 0) { printf("t %d, def_level %d, rep_level %d, start_depth %d, end_depth %d, max_depth %d\n", \
          t, def_level, rep_level, start_depth, end_depth, max_depth); }
      }
    }

    //Determine value count & row index
    // track (page-relative) row index for the thread so we can compare against input bounds
    // keep track of overall # of rows we've read.
    int const is_new_row = start_depth == 0 ? 1 : 0;
    int num_prior_new_rows, total_num_new_rows;
    {
      block_scan_results new_row_scan_results;
      scan_block_exclusive_sum<decode_block_size>(is_new_row, new_row_scan_results);
      num_prior_new_rows = new_row_scan_results.thread_count_within_block;
      total_num_new_rows = new_row_scan_results.block_count;
    }

if constexpr (enable_print_large_list) {
  if(within_batch && (bool(is_new_row) != (t % 4 == 0))) {
    printf("CUB GARBAGE: blockIdx.x %d, value_count %d, target_value_count %d, t %d, is_new_row %d, start_depth %d\n", 
      blockIdx.x, value_count, target_value_count, t, is_new_row, start_depth);
  }
  if(within_batch && (num_prior_new_rows != ((t + 3) / 4))) {
    printf("CUB GARBAGE: blockIdx.x %d, value_count %d, target_value_count %d, t %d, num_prior_new_rows %d\n", 
      blockIdx.x, value_count, target_value_count, t, num_prior_new_rows);
  }
  if((value_count + 128 <= target_value_count) && (total_num_new_rows != 32)) {
    printf("CUB GARBAGE: blockIdx.x %d, value_count %d, target_value_count %d, t %d, total_num_new_rows %d\n", 
      blockIdx.x, value_count, target_value_count, t, total_num_new_rows);
  }
}

    if constexpr (enable_print) {
      if (t == 0) { printf("num_prior_new_rows %d, total_num_new_rows %d\n", num_prior_new_rows, total_num_new_rows); }
    }

    int const row_index = input_row_count + (num_prior_new_rows + is_new_row - 1);
    input_row_count += total_num_new_rows;
    int const in_row_bounds = (row_index >= row_index_lower_bound) && (row_index < last_row);

    // thread and block value count

    // if we are within the range of nesting levels we should be adding value indices for
    // is from/in current rep level to/in the rep level AT the depth with the def value
    int in_nesting_bounds = ((0 >= start_depth && 0 <= end_depth) && in_row_bounds) ? 1 : 0;

    if constexpr (enable_print) {
      if(t == 0) { printf("LIST ROWS: row_index %d, row_index_lower_bound %d, last_row %d, in_row_bounds %d, in_nesting_bounds %d\n", 
        row_index, row_index_lower_bound, last_row, in_row_bounds, in_nesting_bounds); }
      if (t < 32) { printf("t %d, is_new_row %d, num_prior_new_rows %d, row_index %d, in_row_bounds %d\n", 
        t, is_new_row, num_prior_new_rows, row_index, in_row_bounds); }
    }

    // queries is_valid from all threads, stores prior total and total total

    //WARP VALUE COUNT:
    int thread_value_count_within_warp, warp_value_count, thread_value_count, block_value_count;
    {
      block_scan_results value_count_scan_results;
      scan_block_exclusive_sum<decode_block_size>(in_nesting_bounds, value_count_scan_results);

      thread_value_count_within_warp = value_count_scan_results.thread_count_within_warp;
      warp_value_count = value_count_scan_results.warp_count;
      thread_value_count = value_count_scan_results.thread_count_within_block;
      block_value_count = value_count_scan_results.block_count;
    }

if constexpr (enable_print_large_list) {
  if(within_batch && in_row_bounds && (in_nesting_bounds != (t % 4 == 0))) {
    printf("CUB GARBAGE: blockIdx.x %d, value_count %d, target_value_count %d, t %d, in_nesting_bounds %d, start_depth %d, end_depth %d, "
      "in_row_bounds %d, row_index %d, input_row_count %d, row_index_lower_bound %d, last_row %d, first_row %d, s->num_rows %d\n", 
      blockIdx.x, value_count, target_value_count, t, in_nesting_bounds, start_depth, end_depth, in_row_bounds, row_index, input_row_count, 
      row_index_lower_bound, last_row, first_row, s->num_rows);
  }
  if(within_batch && in_row_bounds && (thread_value_count != ((t + 3) / 4))) {
    printf("CUB GARBAGE: blockIdx.x %d, value_count %d, target_value_count %d, t %d, thread_value_count %d\n", 
      blockIdx.x, value_count, target_value_count, t, thread_value_count);
  }
  if((value_count + 128 <= target_value_count) && (input_row_count + total_num_new_rows <= last_row) && (block_value_count != 32)) {
    printf("CUB GARBAGE: blockIdx.x %d, value_count %d, target_value_count %d, t %d, block_value_count %d\n", 
      blockIdx.x, value_count, target_value_count, t, block_value_count);
  }
}

    if constexpr (enable_print) {
      if (t == 0) { printf("block_value_count %d\n", block_value_count); }
      if (t < 32) { printf("t %d, thread_value_count %d, in_nesting_bounds %d\n", 
        t, thread_value_count, in_nesting_bounds); }
    }

    // column is either nullable or is a list (or both): iterate by depth
    for (int d_idx = 0; d_idx <= max_depth; d_idx++) {

      auto& ni = s->nesting_info[d_idx];

      // everything up to the max_def_level is a non-null value
      int is_valid;
      if constexpr (nullable) {
        is_valid = ((def_level >= ni.max_def_level) && in_nesting_bounds) ? 1 : 0;
      } else {
        is_valid = in_nesting_bounds;
      }

      if constexpr (enable_print) {
        if (t == 0) { printf("nullable %d, depth %d, max_depth %d, max_def_level %d, value_count %d\n", 
          int(nullable), d_idx, max_depth, ni.max_def_level, value_count); }
        if (t < 32) { printf("t %d, def_level %d, in_nesting_bounds %d, is_valid %d\n", 
          t, def_level, in_nesting_bounds, is_valid); }
      }

      // thread and block validity count
      // queries is_valid of all threads, stores prior total and total total

      // for nested lists, it's more complicated.  This block will visit 128 incoming values,
      // however not all of them will necessarily represent a value at this nesting level. so
      // the validity bit for thread t might actually represent output value t-6. the correct
      // position for thread t's bit is thread_value_count. 


//WARP VALID COUNT:
        // for nested schemas, it's more complicated.  This warp will visit 32 incoming values,
        // however not all of them will necessarily represent a value at this nesting level. so
        // the validity bit for thread t might actually represent output value t-6. the correct
        // position for thread t's bit is thread_value_count. for cuda 11 we could use
        // __reduce_or_sync(), but until then we have to do a warp reduce.
        uint32_t const warp_valid_mask = WarpReduceOr32((uint32_t)is_valid << thread_value_count_within_warp);
        int thread_valid_count, block_valid_count;
        {
          auto thread_mask = (uint32_t(1) << thread_value_count_within_warp) - 1;

          block_scan_results valid_count_scan_results;
          scan_block_exclusive_sum<decode_block_size>(warp_valid_mask, warp_lane, warp_index, thread_mask, valid_count_scan_results);
          thread_valid_count = valid_count_scan_results.thread_count_within_block;
          block_valid_count = valid_count_scan_results.block_count;
        }

if constexpr (enable_print_large_list) {
  if(within_batch && in_row_bounds && (((d_idx == 0) && (is_valid != (t % 4 == 0))) || ((d_idx == 1) && !is_valid))) {
    printf("CUB GARBAGE: blockIdx.x %d, value_count %d, target_value_count %d, t %d, d_idx %d, is_valid %d, in_nesting_bounds %d\n", 
      blockIdx.x, value_count, target_value_count, t, d_idx, is_valid, in_nesting_bounds);
  }
  if (within_batch && in_row_bounds && (((d_idx == 0) && (thread_valid_count != ((t + 3)/ 4))) || ((d_idx == 1) && (thread_valid_count != t)))) {
    printf("CUB GARBAGE: blockIdx.x %d, value_count %d, target_value_count %d, t %d, d_idx %d, thread_valid_count %d\n", 
      blockIdx.x, value_count, target_value_count, t, d_idx, thread_valid_count);
  }
  if((value_count + 128 <= target_value_count) && (input_row_count + total_num_new_rows <= last_row) && (((d_idx == 0) && (block_valid_count != 32)) || ((d_idx == 1) && (block_valid_count != 128)))) {
    printf("CUB GARBAGE: blockIdx.x %d, value_count %d, target_value_count %d, t %d, d_idx %d, block_valid_count %d\n", 
      blockIdx.x, value_count, target_value_count, t, d_idx, block_valid_count);
  }
}

      if constexpr (enable_print) {
        if((block_valid_count == 0) && (t == 0) && (d_idx == max_depth)) { 
          printf("EMPTY VALID MASK: def_level %d, max_def_level %d, in_nesting_bounds %d, start_depth %d, "
            "end_depth %d, in_row_bounds %d, row_index %d, row_index_lower_bound %d, last_row %d, input_row_count %d\n", 
            def_level, ni.max_def_level, in_nesting_bounds, start_depth, end_depth, in_row_bounds, row_index, 
            row_index_lower_bound, last_row, input_row_count); }

        if (t == 0) { printf("block_valid_count %u\n", int(block_valid_count)); }
        if (t < 32) { printf("t %d, thread_valid_count %d\n", t, thread_valid_count); }
      }

      // compute warp and thread value counts for the -next- nesting level. we need to
      // do this for nested schemas so that we can emit an offset for the -current- nesting
      // level. more concretely : the offset for the current nesting level == current length of the
      // next nesting level
      int next_thread_value_count_within_warp = 0, next_warp_value_count = 0;
      int next_thread_value_count = 0, next_block_value_count = 0;
      int next_in_nesting_bounds = 0;
      if (d_idx < max_depth) {
        //mask is different between depths
        next_in_nesting_bounds = 
          (d_idx + 1 >= start_depth && d_idx + 1 <= end_depth && in_row_bounds) ? 1 : 0;

//NEXT WARP VALUE COUNT:
        {
          block_scan_results next_value_count_scan_results;
          scan_block_exclusive_sum<decode_block_size>(next_in_nesting_bounds, next_value_count_scan_results);

          next_thread_value_count_within_warp = next_value_count_scan_results.thread_count_within_warp;
          next_warp_value_count = next_value_count_scan_results.warp_count;
          next_thread_value_count = next_value_count_scan_results.thread_count_within_block;
          next_block_value_count = next_value_count_scan_results.block_count;
        }

if constexpr (enable_print_large_list) {
  if(within_batch && in_row_bounds && (next_in_nesting_bounds != 1)) {
    printf("CUB GARBAGE: blockIdx.x %d, value_count %d, target_value_count %d, t %d, next_in_nesting_bounds %d, start_depth %d, end_depth %d, in_row_bounds %d, row_index %d, input_row_count %d\n", 
      blockIdx.x, value_count, target_value_count, t, next_in_nesting_bounds, start_depth, end_depth, in_row_bounds, row_index, input_row_count);
  }
  if(within_batch && in_row_bounds && (next_thread_value_count != t)) {
    printf("CUB GARBAGE: blockIdx.x %d, value_count %d, target_value_count %d, t %d, next_thread_value_count %d\n", 
      blockIdx.x, value_count, target_value_count, t, next_thread_value_count);
  }
  if((value_count + 128 <= target_value_count) && (input_row_count + total_num_new_rows <= last_row) && (next_block_value_count != 128)) {
    printf("CUB GARBAGE: blockIdx.x %d, value_count %d, target_value_count %d, t %d, next_block_value_count %d\n", 
      blockIdx.x, value_count, target_value_count, t, next_block_value_count);
  }
}

        if constexpr (enable_print) {
          if (t == 0) { printf("next depth %d, next_block_value_count %d\n", d_idx + 1, next_block_value_count); }
          if (t < 32) { printf("t %d, start_depth %d, end_depth %d, in_row_bounds %d, next_in_nesting_bounds %d\n", 
            t, start_depth, end_depth, in_row_bounds, next_in_nesting_bounds); }
          if (t < 32) { printf("t %d, next_thread_value_count %d\n", t, next_thread_value_count); }
        }

        // if we're -not- at a leaf column and we're within nesting/row bounds
        // and we have a valid data_out pointer, it implies this is a list column, so
        // emit an offset.
        if (in_nesting_bounds && ni.data_out != nullptr) {
          const auto& next_ni = s->nesting_info[d_idx + 1];
          int const idx             = ni.value_count + thread_value_count;
          cudf::size_type const ofs = next_ni.value_count + next_thread_value_count + next_ni.page_start_value;

          //STORE THE OFFSET FOR THE NEW LIST LOCATION
          (reinterpret_cast<cudf::size_type*>(ni.data_out))[idx] = ofs;

/*
if constexpr (enable_print_large_list) {
  int overall_index = 4*(blockIdx.x * 20000 + idx);
  if(overall_index != ofs) {
    printf("WHOA BAD OFFSET\n");
    printf("WHOA BAD OFFSET: WROTE %d to %d! t %d, blockIdx.x %d, idx %d, d_idx %d, start_depth %d, end_depth %d, max_depth %d, "
      "in_row_bounds %d, in_nesting_bounds %d, next_in_nesting_bounds %d, row_index %d, row_index_lower_bound %d, last_row %d, "
      "input_row_count %d, num_prior_new_rows %d, is_new_row %d, total_num_new_rows %d, def_level %d, ni.value_count %d, "
      "thread_value_count %d, next_ni.value_count %d, next_thread_value_count %d, next_ni.page_start_value %d, value_count %d, "
      "target_value_count %d, block_value_count %d, next_block_value_count %d\n", 
      ofs, overall_index, t, blockIdx.x, idx, d_idx, start_depth, end_depth, max_depth, in_row_bounds, in_nesting_bounds, 
      next_in_nesting_bounds, row_index, row_index_lower_bound, last_row, input_row_count, num_prior_new_rows, is_new_row, 
      total_num_new_rows, def_level, ni.value_count, thread_value_count, next_ni.value_count, 
      next_thread_value_count, next_ni.page_start_value, value_count, target_value_count, block_value_count, next_block_value_count);
  }
}
*/
          if constexpr (enable_print || enable_print_range_error) {
            if((idx < 0) || (idx > 50000)){ printf("WHOA: offset index %d out of bounds!\n", idx); }
            if(ofs < 0){ printf("WHOA: offset value %d out of bounds!\n", ofs); }
          }

          if constexpr (enable_print) {
            if(idx < 0) { printf("WHOA: offset index out of bounds!\n"); }
            if (t < 32) { printf("OFFSETS: t %d, idx %d, next value count %d, next page_start_value %d, ofs %d\n", 
              t, idx, next_ni.value_count, next_ni.page_start_value, ofs); }
          }
        }
      }

      // validity is processed per-warp (on lane 0's), because writes are atomic
      //
      // nested schemas always read and write to the same bounds 
      // (that is, read and write positions are already pre-bounded by first_row/num_rows). 
      // since we are about to write the validity vector
      // here we need to adjust our computed mask to take into account the write row bounds.
      if constexpr (nullable) {
//TODO: Consider OR'ING for next_thread_value_count and popc() for next_thread_value_count
//so that we don't have to take a ballot here. Is uint128 so may deconstruct to this anyway ...

        if(is_first_lane && (ni.valid_map != nullptr) && (warp_value_count > 0)) {
          // last bit in the warp to store //in old is warp_valid_mask_bit_count
//so it's a count of everything in nesting bounds, though bits can be zero if NULL at this level            

          // absolute bit offset into the output validity map
          //is cumulative sum of warp_value_count at the given nesting depth
          // DON'T subtract by first_row: since it's lists it's not 1-row-per-value
          int const bit_offset = ni.valid_map_offset + thread_value_count;
          store_validity(bit_offset, ni.valid_map, warp_valid_mask, warp_value_count);

          if constexpr (enable_print) {
              printf("STORE VALIDITY: t %d, depth %d, thread_value_count %d, valid_map_offset %d, bit_offset %d, warp_value_count %d, warp_valid_mask %u\n", 
                t, d_idx, thread_value_count, ni.valid_map_offset, bit_offset, warp_value_count, warp_valid_mask);
            }
        }

        if (t == 0) { 
          size_type const block_null_count = block_value_count - block_valid_count;
          if constexpr (enable_print) {
            if (t == 0) { printf("BLOCK NULLS: depth %d, prior %d, block_null_count %u\n", 
              d_idx, ni.null_count, block_null_count); }
          }
          ni.null_count += block_null_count;
        }
      }

      // if this is valid and we're at the leaf, output dst_pos
      // Read these before the sync, so that when thread 0 modifies them we've already read their values
      int current_value_count = ni.value_count;
      __syncthreads();  // handle modification of ni.value_count from below
      if (d_idx == max_depth) {
        if (is_valid) {
          // for non-list types, the value count is always the same across
          int const dst_pos = current_value_count + thread_value_count;
          int const src_pos = max_depth_valid_count + thread_valid_count;
          int const output_index = rolling_index<state_buf::nz_buf_size>(src_pos);

          if constexpr (enable_print || enable_print_range_error) {
            if((output_index < 0) || (output_index >= state_buf::nz_buf_size)) {
              printf("WHOA: output index STORE %d out of bounds!\n", output_index);
            }
            if(dst_pos < 0) { printf("WHOA: dst_pos STORE %d out of bounds!\n", dst_pos); }
          }

          if constexpr (enable_print) {
            if (t == 0) { printf("ni.value_count %d, max_depth_valid_count %d\n", int(ni.value_count), max_depth_valid_count); }
            if (t < 32) { printf("t %d, src_pos %d, output_index %d\n", t, src_pos, output_index); }

            if((t == 0) && (src_pos == 0)) {printf("SPECIAL: output_index %d, dst_pos %d, ni.value_count %d, max_depth_valid_count %d, thread_value_count %d, thread_valid_count %d\n", 
              output_index, dst_pos, ni.value_count, max_depth_valid_count, thread_value_count, thread_valid_count);}

            if (t == 0) { printf("OUTPUT_INDICES: output_index %d, dst_pos %d\n", output_index, dst_pos); }
          }

          //Index from rolling buffer of values (which doesn't include nulls) to final array (which includes gaps for nulls)        
          sb->nz_idx[output_index] = dst_pos;
        }
        max_depth_valid_count += block_valid_count;
      }

      // update stuff
      if (t == 0) {
        ni.value_count += block_value_count;
        ni.valid_map_offset += block_value_count;
      }
      __syncthreads();  // handle modification of ni.value_count from below

      // propagate value counts for the next depth level
      block_value_count  = next_block_value_count;
      thread_value_count = next_thread_value_count;
      in_nesting_bounds  = next_in_nesting_bounds;
      warp_value_count = next_warp_value_count;
      thread_value_count_within_warp = next_thread_value_count_within_warp;
    } //END OF DEPTH LOOP

    if constexpr (enable_print) {
      if (t == 0) { printf("END DEPTH LOOP\n"); }
    }

    int const batch_size = min(max_batch_size, target_value_count - value_count);
    value_count += batch_size;
  }

  if constexpr (enable_print) {
    if (t == 0) { printf("END LOOP\n"); }
  }

  if (t == 0) {
    // update valid value count for decoding and total # of values we've processed
    s->nesting_info[max_depth].valid_count = max_depth_valid_count;
    s->nz_count          = max_depth_valid_count;
    s->input_value_count = value_count;

    // If we have lists # rows != # values
    s->input_row_count = input_row_count;
  }

  return max_depth_valid_count;
}

// is the page marked nullable or not
__device__ inline bool is_nullable(page_state_s* s)
{
  auto const lvl           = level_type::DEFINITION;
  auto const max_def_level = s->col.max_level[lvl];
  return max_def_level > 0;
}

// for a nullable page, check to see if it could have nulls
__device__ inline bool maybe_has_nulls(page_state_s* s)
{
  auto const lvl      = level_type::DEFINITION;
  auto const init_run = s->initial_rle_run[lvl];
  // literal runs, lets assume they could hold nulls
  if (is_literal_run(init_run)) { return true; }

  // repeated run with number of items in the run not equal
  // to the rows in the page, assume that means we could have nulls
  if (s->page.num_input_values != (init_run >> 1)) { return true; }

  auto const lvl_bits = s->col.level_bits[lvl];
  auto const run_val  = lvl_bits == 0 ? 0 : s->initial_rle_value[lvl];

  // the encoded repeated value isn't valid, we have (all) nulls
  return run_val != s->col.max_level[lvl];
}

template <int decode_block_size_t, typename stream_type>
__device__ int skip_decode(stream_type& parquet_stream, int num_to_skip, int t)
{
  static constexpr bool enable_print = false;

  // it could be that (e.g.) we skip 5000 but starting at row 4000 we have a run of length 2000:
  // in that case skip_decode() only skips 4000, and we have to process the remaining 1000 up front
  // modulo 2 * block_size of course, since that's as many as we process at once
  int num_skipped = parquet_stream.skip_decode(t, num_to_skip);
  if constexpr (enable_print) {
    if (t == 0) { printf("SKIPPED: num_skipped %d, for %d\n", num_skipped, num_to_skip); }
  }
  while (num_skipped < num_to_skip) {
    auto const to_decode = min(2 * decode_block_size_t, num_to_skip - num_skipped);
    num_skipped += parquet_stream.decode_next(t, to_decode);
    if constexpr (enable_print) {
      if (t == 0) { printf("EXTRA SKIPPED: to_decode %d, at %d, for %d\n", to_decode, num_skipped, num_to_skip); }
    }
    __syncthreads();
  }

  return num_skipped;
}

/**
 * @brief Kernel for computing fixed width non dictionary column data stored in the pages
 *
 * This function will write the page data and the page data's validity to the
 * output specified in the page's column chunk. If necessary, additional
 * conversion will be performed to translate from the Parquet datatype to
 * desired output datatype.
 *
 * @param pages List of pages
 * @param chunks List of column chunks
 * @param min_row Row index to start reading at
 * @param num_rows Maximum number of rows to read
 * @param error_code Error code to set if an error is encountered
 */
template <typename level_t,
          int decode_block_size_t,
          decode_kernel_mask kernel_mask_t,
          bool has_dict_t,
          bool has_nesting_t,
          bool has_lists_t,
          template <int block_size, bool decode_has_lists_t, typename state_buf>
          typename DecodeValuesFunc>
CUDF_KERNEL void __launch_bounds__(decode_block_size_t)
  gpuDecodePageDataGeneric(PageInfo* pages,
                           device_span<ColumnChunkDesc const> chunks,
                           size_t min_row,
                           size_t num_rows,
                           kernel_error::pointer error_code)
{
  constexpr int rolling_buf_size    = decode_block_size_t * 2;
  constexpr int rle_run_buffer_size = rle_stream_required_run_buffer_size<decode_block_size_t>();

  __shared__ __align__(16) page_state_s state_g;
  using state_buf_t = page_state_buffers_s<rolling_buf_size,  // size of nz_idx buffer
                                           has_dict_t ? rolling_buf_size : 1,
                                           1>;
  __shared__ __align__(16) state_buf_t state_buffers;

  page_state_s* const s = &state_g;
  auto* const sb        = &state_buffers;
  int const page_idx    = blockIdx.x;
  int const t           = threadIdx.x;
  PageInfo* pp          = &pages[page_idx];

  if (!(BitAnd(pages[page_idx].kernel_mask, kernel_mask_t))) { return; }

  // must come after the kernel mask check
  [[maybe_unused]] null_count_back_copier _{s, t};

  if (!setupLocalPageInfo(s,
                          pp,
                          chunks,
                          min_row,
                          num_rows,
                          mask_filter{kernel_mask_t},
                          page_processing_stage::DECODE)) {
    return;
  }

  // if we have no work to do (eg, in a skip_rows/num_rows case) in this page.
  if (s->num_rows == 0) { return; }

  DecodeValuesFunc<decode_block_size_t, has_lists_t, state_buf_t> decode_values;

  bool const should_process_nulls = is_nullable(s) && maybe_has_nulls(s);

  // shared buffer. all shared memory is suballocated out of here
  static constexpr auto align_test = false;
  static constexpr size_t buffer_alignment = align_test ? 128 : 16;
  constexpr int shared_rep_size = has_lists_t ? cudf::util::round_up_unsafe(rle_run_buffer_size *
    sizeof(rle_run<level_t>), buffer_alignment) : 0;
  constexpr int shared_dict_size =
    has_dict_t
      ? cudf::util::round_up_unsafe(rle_run_buffer_size * sizeof(rle_run<uint32_t>), buffer_alignment)
      : 0;
  constexpr int shared_def_size =
    cudf::util::round_up_unsafe(rle_run_buffer_size * sizeof(rle_run<level_t>), buffer_alignment);
  constexpr int shared_buf_size = shared_rep_size + shared_dict_size + shared_def_size;
  __shared__ __align__(buffer_alignment) uint8_t shared_buf[shared_buf_size];

  // setup all shared memory buffers
  int shared_offset = 0;
  rle_run<level_t>* rep_runs = reinterpret_cast<rle_run<level_t>*>(shared_buf + shared_offset);
  if constexpr (has_lists_t){ shared_offset += shared_rep_size; }

  rle_run<uint32_t>* dict_runs = reinterpret_cast<rle_run<uint32_t>*>(shared_buf + shared_offset);
  if constexpr (has_dict_t) { shared_offset += shared_dict_size; }
  rle_run<level_t>* def_runs = reinterpret_cast<rle_run<level_t>*>(shared_buf + shared_offset);

  // initialize the stream decoders (requires values computed in setupLocalPageInfo)
  rle_stream<level_t, decode_block_size_t, rolling_buf_size> def_decoder{def_runs};
  level_t* const def = reinterpret_cast<level_t*>(pp->lvl_decode_buf[level_type::DEFINITION]);
  if (should_process_nulls) {
    def_decoder.init(s->col.level_bits[level_type::DEFINITION],
                     s->abs_lvl_start[level_type::DEFINITION],
                     s->abs_lvl_end[level_type::DEFINITION],
                     def,
                     s->page.num_input_values);
  }
  
  rle_stream<level_t, decode_block_size_t, rolling_buf_size> rep_decoder{rep_runs};
  level_t* const rep = reinterpret_cast<level_t*>(pp->lvl_decode_buf[level_type::REPETITION]);
  if constexpr (has_lists_t){
    rep_decoder.init(s->col.level_bits[level_type::REPETITION],
                     s->abs_lvl_start[level_type::REPETITION],
                     s->abs_lvl_end[level_type::REPETITION],
                     rep,
                     s->page.num_input_values);
  }

  static constexpr bool enable_print = false;

  rle_stream<uint32_t, decode_block_size_t, rolling_buf_size> dict_stream{dict_runs};
  if constexpr (has_dict_t) {
    dict_stream.init(
      s->dict_bits, s->data_start, s->data_end, sb->dict_idx, s->page.num_input_values);
    if constexpr (enable_print) {
      if(t == 0) { printf("INIT DICT: dict_bits %d, data_start %p, data_end %p, dict_idx %p, page.num_input_values %d, s->dict_pos %d \n", 
        s->dict_bits, s->data_start, s->data_end, sb->dict_idx, s->page.num_input_values, s->dict_pos); }
    }
  }

  if constexpr (enable_print) {
    if((t == 0) && (page_idx == 0)){
      printf("SIZES: shared_rep_size %d, shared_dict_size %d, shared_def_size %d\n", shared_rep_size, shared_dict_size, shared_def_size);
    }
    if constexpr (has_lists_t){
      printf("Is fixed list page\n");
    } else {
      printf("Is fixed non-list page\n");
    }
  }

  // We use two counters in the loop below: processed_count and valid_count.
  // - processed_count: number of values out of num_input_values that we have decoded so far.
  //   the definition stream returns the number of total rows it has processed in each call
  //   to decode_next and we accumulate in process_count.
  // - valid_count: number of non-null values we have decoded so far. In each iteration of the
  //   loop below, we look at the number of valid items (which could be all for non-nullable),
  //   and valid_count is that running count.
  int processed_count = 0;
  int valid_count     = 0;
  // the core loop. decode batches of level stream data using rle_stream objects
  // and pass the results to gpuDecodeValues

  //For lists (which can have skipped values, skip ahead in the decoding so that we don't repeat work
  if constexpr (has_lists_t){
    if(s->page.skipped_leaf_values > 0) {
      if (should_process_nulls) {
        skip_decode<decode_block_size_t>(def_decoder, s->page.skipped_leaf_values, t);
      }
      processed_count = skip_decode<decode_block_size_t>(rep_decoder, s->page.skipped_leaf_values, t);
      if constexpr (has_dict_t) {
        skip_decode<decode_block_size_t>(dict_stream, s->page.skipped_leaf_values, t);
      }
    }
  }

  if constexpr (enable_print) {
    if(t == 0) { printf("page_idx %d, should_process_nulls %d, has_lists_t %d, has_dict_t %d, num_rows %lu, page.num_input_values %d\n", 
      page_idx, int(should_process_nulls), int(has_lists_t), int(has_dict_t), num_rows, s->page.num_input_values); }
  }

  auto print_nestings = [&](bool is_post){
    if constexpr (enable_print) {
      auto print_nesting_level = [&](const PageNestingDecodeInfo& ni) {
        printf("page_idx %d, max_def_level %d, start_depth %d, end_depth %d, page_start_value %d, null_count %d, "
          "valid_map_offset %d, valid_count %d, value_count %d\n", 
          page_idx, ni.max_def_level, ni.start_depth, ni.end_depth, ni.page_start_value, ni.null_count, 
          ni.valid_map_offset, ni.valid_count, ni.value_count);
      };

      if(t == 0) {
        printf("POST %d NESTING 0: ", int(is_post));
        print_nesting_level(s->nesting_info[0]);
        printf("POST %d NESTING 1: ", int(is_post));
        print_nesting_level(s->nesting_info[1]);
        //printf("POST %d NESTING 2: ", int(is_post));
        //print_nesting_level(s->nesting_info[2]);
      }
    }
  };

  print_nestings(false);
  if constexpr (enable_print) {
    if(t == 0) {printf("LOOP START page_idx %d\n", page_idx);}
  }

  int last_row = s->first_row + s->num_rows;
  while ((s->error == 0) && (processed_count < s->page.num_input_values) &&
         (s->input_row_count <= last_row)) {
    int next_valid_count;

    // only need to process definition levels if this is a nullable column
    if (should_process_nulls) {
      processed_count += def_decoder.decode_next(t);
      __syncthreads();

      if constexpr (has_lists_t) {
        rep_decoder.decode_next(t);
        __syncthreads();

        int value_count = s->input_value_count;
        next_valid_count = gpuUpdateValidityAndRowIndicesLists<decode_block_size_t, true, level_t>(
          processed_count, s, sb, def, rep, t);
        if constexpr (enable_print) {
          if(t == 0) { printf("LISTS NEXT: next_valid_count %d\n", next_valid_count); }
          if(t == 0) { printf("PROCESSING: page total values %d, num_input_values %d, pre value_count %d, post value_count %d, "
            "processed_count %d, valid_count %d, next_valid_count %d\n", 
            s->page.num_input_values, s->input_value_count, value_count, s->input_value_count, processed_count, valid_count, next_valid_count); }
        }
      } else if constexpr (has_nesting_t) {
        next_valid_count = gpuUpdateValidityAndRowIndicesNested<decode_block_size_t, level_t>(
          processed_count, s, sb, def, t);
        if constexpr (enable_print) {
          if(t == 0) { printf("NESTED NEXT: next_valid_count %d\n", next_valid_count); }
        }
      } else {
        next_valid_count = gpuUpdateValidityAndRowIndicesFlat<decode_block_size_t, level_t>(
          processed_count, s, sb, def, t);
      }
    }
    // if we wanted to split off the skip_rows/num_rows case into a separate kernel, we could skip
    // this function call entirely since all it will ever generate is a mapping of (i -> i) for
    // nz_idx.  gpuDecodeFixedWidthValues would be the only work that happens.
    else {
      if constexpr (has_lists_t) {
        processed_count += rep_decoder.decode_next(t);
        __syncthreads();

        next_valid_count =
          gpuUpdateValidityAndRowIndicesLists<decode_block_size_t, false, level_t>(
            processed_count, s, sb, nullptr, rep, t);
      } else {
        processed_count += min(rolling_buf_size, s->page.num_input_values - processed_count);
        next_valid_count = gpuUpdateValidityAndRowIndicesNonNullable<decode_block_size_t>(processed_count, s, sb, t);
      }
    }
    __syncthreads();

    // if we have dictionary data
    if constexpr (has_dict_t) {
      // We want to limit the number of dictionary items we decode, that correspond to
      // the rows we have processed in this iteration that are valid.
      // We know the number of valid rows to process with: next_valid_count - valid_count.
      dict_stream.decode_next(t, next_valid_count - valid_count);
      __syncthreads();
    }

    // decode the values themselves
    decode_values(s, sb, valid_count, next_valid_count, t);
    __syncthreads();

    valid_count = next_valid_count;

    if constexpr (enable_print) {
      if(t == 0) { printf("LOOP: processed_count %d, #page values %d, error %d\n", 
        processed_count, s->page.num_input_values, s->error); }
    }
  }
  if (t == 0 and s->error != 0) { set_error(s->error, error_code); }

  print_nestings(true);
}

}  // anonymous namespace

void __host__ DecodePageDataFixed(cudf::detail::hostdevice_span<PageInfo> pages,
                                  cudf::detail::hostdevice_span<ColumnChunkDesc const> chunks,
                                  size_t num_rows,
                                  size_t min_row,
                                  int level_type_size,
                                  bool has_nesting,
                                  bool is_list,
                                  kernel_error::pointer error_code,
                                  rmm::cuda_stream_view stream)
{
  constexpr int decode_block_size = 128;

  dim3 dim_block(decode_block_size, 1);
  dim3 dim_grid(pages.size(), 1);  // 1 threadblock per page
  if (level_type_size == 1) {
    if (is_list) {
      gpuDecodePageDataGeneric<uint8_t,
                               decode_block_size,
                               decode_kernel_mask::FIXED_WIDTH_NO_DICT_LIST,
                               false,
                               true,
                               true,
                               decode_fixed_width_values_func>
        <<<dim_grid, dim_block, 0, stream.value()>>>(
          pages.device_ptr(), chunks, min_row, num_rows, error_code);
    } else if (has_nesting) {
      gpuDecodePageDataGeneric<uint8_t,
                               decode_block_size,
                               decode_kernel_mask::FIXED_WIDTH_NO_DICT_NESTED,
                               false,
                               true,
                               false,
                               decode_fixed_width_values_func>
        <<<dim_grid, dim_block, 0, stream.value()>>>(
          pages.device_ptr(), chunks, min_row, num_rows, error_code);
    } else {
      gpuDecodePageDataGeneric<uint8_t,
                               decode_block_size,
                               decode_kernel_mask::FIXED_WIDTH_NO_DICT,
                               false,
                               false,
                               false,
                               decode_fixed_width_values_func>
        <<<dim_grid, dim_block, 0, stream.value()>>>(
          pages.device_ptr(), chunks, min_row, num_rows, error_code);
    }
  } else {
    if (is_list) {
      gpuDecodePageDataGeneric<uint16_t,
                               decode_block_size,
                               decode_kernel_mask::FIXED_WIDTH_NO_DICT_LIST,
                               false,
                               true,
                               true,
                               decode_fixed_width_values_func>
        <<<dim_grid, dim_block, 0, stream.value()>>>(
          pages.device_ptr(), chunks, min_row, num_rows, error_code);
    } else if (has_nesting) {
      gpuDecodePageDataGeneric<uint16_t,
                               decode_block_size,
                               decode_kernel_mask::FIXED_WIDTH_NO_DICT_NESTED,
                               false,
                               true,
                               false,
                               decode_fixed_width_values_func>
        <<<dim_grid, dim_block, 0, stream.value()>>>(
          pages.device_ptr(), chunks, min_row, num_rows, error_code);
    } else {
      gpuDecodePageDataGeneric<uint16_t,
                               decode_block_size,
                               decode_kernel_mask::FIXED_WIDTH_NO_DICT,
                               false,
                               false,
                               false,
                               decode_fixed_width_values_func>
        <<<dim_grid, dim_block, 0, stream.value()>>>(
          pages.device_ptr(), chunks, min_row, num_rows, error_code);
    }
  }
}

void __host__ DecodePageDataFixedDict(cudf::detail::hostdevice_span<PageInfo> pages,
                                      cudf::detail::hostdevice_span<ColumnChunkDesc const> chunks,
                                      size_t num_rows,
                                      size_t min_row,
                                      int level_type_size,
                                      bool has_nesting,
                                      bool is_list,
                                      kernel_error::pointer error_code,
                                      rmm::cuda_stream_view stream)
{
  constexpr int decode_block_size = 128;

  dim3 dim_block(decode_block_size, 1);  // decode_block_size = 128 threads per block
  dim3 dim_grid(pages.size(), 1);        // 1 thread block per page => # blocks

  if (level_type_size == 1) {
    if (is_list) {
      gpuDecodePageDataGeneric<uint8_t,
                               decode_block_size,
                               decode_kernel_mask::FIXED_WIDTH_DICT_LIST,
                               true,
                               true,
                               true,
                               decode_fixed_width_values_func>
        <<<dim_grid, dim_block, 0, stream.value()>>>(
          pages.device_ptr(), chunks, min_row, num_rows, error_code);
    } else if (has_nesting) {
      gpuDecodePageDataGeneric<uint8_t,
                               decode_block_size,
                               decode_kernel_mask::FIXED_WIDTH_DICT_NESTED,
                               true,
                               true,
                               false,
                               decode_fixed_width_values_func>
        <<<dim_grid, dim_block, 0, stream.value()>>>(
          pages.device_ptr(), chunks, min_row, num_rows, error_code);
    } else {
      gpuDecodePageDataGeneric<uint8_t,
                               decode_block_size,
                               decode_kernel_mask::FIXED_WIDTH_DICT,
                               true,
                               false,
                               false,
                               decode_fixed_width_values_func>
        <<<dim_grid, dim_block, 0, stream.value()>>>(
          pages.device_ptr(), chunks, min_row, num_rows, error_code);
    }
  } else {
    if (is_list) {
      gpuDecodePageDataGeneric<uint16_t,
                               decode_block_size,
                               decode_kernel_mask::FIXED_WIDTH_DICT_LIST,
                               true,
                               true,
                               true,
                               decode_fixed_width_values_func>
        <<<dim_grid, dim_block, 0, stream.value()>>>(
          pages.device_ptr(), chunks, min_row, num_rows, error_code);
    } else if (has_nesting) {
      gpuDecodePageDataGeneric<uint16_t,
                               decode_block_size,
                               decode_kernel_mask::FIXED_WIDTH_DICT_NESTED,
                               true,
                               true,
                               false,
                               decode_fixed_width_values_func>
        <<<dim_grid, dim_block, 0, stream.value()>>>(
          pages.device_ptr(), chunks, min_row, num_rows, error_code);
    } else {
      gpuDecodePageDataGeneric<uint16_t,
                               decode_block_size,
                               decode_kernel_mask::FIXED_WIDTH_DICT,
                               true,
                               false,
                               true,
                               decode_fixed_width_values_func>
        <<<dim_grid, dim_block, 0, stream.value()>>>(
          pages.device_ptr(), chunks, min_row, num_rows, error_code);
    }
  }
}

void __host__
DecodeSplitPageFixedWidthData(cudf::detail::hostdevice_span<PageInfo> pages,
                              cudf::detail::hostdevice_span<ColumnChunkDesc const> chunks,
                              size_t num_rows,
                              size_t min_row,
                              int level_type_size,
                              bool has_nesting,
                              bool is_list,
                              kernel_error::pointer error_code,
                              rmm::cuda_stream_view stream)
{
  constexpr int decode_block_size = 128;

  dim3 dim_block(decode_block_size, 1);  // decode_block_size = 128 threads per block
  dim3 dim_grid(pages.size(), 1);        // 1 thread block per page => # blocks

  if (level_type_size == 1) {
    if (is_list) {
      gpuDecodePageDataGeneric<uint8_t,
                               decode_block_size,
                               decode_kernel_mask::BYTE_STREAM_SPLIT_FIXED_WIDTH_LIST,
                               true,
                               true,
                               true,
                               decode_fixed_width_split_values_func>
        <<<dim_grid, dim_block, 0, stream.value()>>>(
          pages.device_ptr(), chunks, min_row, num_rows, error_code);
    } else if (has_nesting) {
      gpuDecodePageDataGeneric<uint8_t,
                               decode_block_size,
                               decode_kernel_mask::BYTE_STREAM_SPLIT_FIXED_WIDTH_NESTED,
                               false,
                               true,
                               false,
                               decode_fixed_width_split_values_func>
        <<<dim_grid, dim_block, 0, stream.value()>>>(
          pages.device_ptr(), chunks, min_row, num_rows, error_code);
    } else {
      gpuDecodePageDataGeneric<uint8_t,
                               decode_block_size,
                               decode_kernel_mask::BYTE_STREAM_SPLIT_FIXED_WIDTH_FLAT,
                               false,
                               false,
                               false,
                               decode_fixed_width_split_values_func>
        <<<dim_grid, dim_block, 0, stream.value()>>>(
          pages.device_ptr(), chunks, min_row, num_rows, error_code);
    }
  } else {
    if (is_list) {
      gpuDecodePageDataGeneric<uint16_t,
                               decode_block_size,
                               decode_kernel_mask::BYTE_STREAM_SPLIT_FIXED_WIDTH_LIST,
                               true,
                               true,
                               true,
                               decode_fixed_width_split_values_func>
        <<<dim_grid, dim_block, 0, stream.value()>>>(
          pages.device_ptr(), chunks, min_row, num_rows, error_code);
    } else if (has_nesting) {
      gpuDecodePageDataGeneric<uint16_t,
                               decode_block_size,
                               decode_kernel_mask::BYTE_STREAM_SPLIT_FIXED_WIDTH_NESTED,
                               false,
                               true,
                               false,
                               decode_fixed_width_split_values_func>
        <<<dim_grid, dim_block, 0, stream.value()>>>(
          pages.device_ptr(), chunks, min_row, num_rows, error_code);
    } else {
      gpuDecodePageDataGeneric<uint16_t,
                               decode_block_size,
                               decode_kernel_mask::BYTE_STREAM_SPLIT_FIXED_WIDTH_FLAT,
                               false,
                               false,
                               false,
                               decode_fixed_width_split_values_func>
        <<<dim_grid, dim_block, 0, stream.value()>>>(
          pages.device_ptr(), chunks, min_row, num_rows, error_code);
    }
  }
}

}  // namespace cudf::io::parquet::detail
