#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "error.hpp"
#include "reader_impl.hpp"

#include <cudf/detail/iterator.cuh>
#include <cudf/detail/utilities/integer_utils.hpp>
#include <cudf/detail/utilities/vector_factories.hpp>

#include <rmm/exec_policy.hpp>

#include <thrust/binary_search.h>
#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/iterator_categories.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/logical.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/transform.h>
#include <thrust/unique.h>

#include <numeric>

namespace cudf::io::parquet::detail {
namespace {

#if defined(PREPROCESS_DEBUG)
void print_pages(cudf::detail::hostdevice_vector<PageInfo>& pages, rmm::cuda_stream_view _stream)
{
  pages.device_to_host_sync(_stream);
  for (size_t idx = 0; idx < pages.size(); idx++) {
    auto const& p = pages[idx];
    // skip dictionary pages
    if (p.flags & PAGEINFO_FLAGS_DICTIONARY) { continue; }
    printf(
      "P(%lu, s:%d): chunk_row(%d), num_rows(%d), skipped_values(%d), skipped_leaf_values(%d), "
      "str_bytes(%d)\n",
      idx,
      p.src_col_schema,
      p.chunk_row,
      p.num_rows,
      p.skipped_values,
      p.skipped_leaf_values,
      p.str_bytes);
  }
}
#endif  // PREPROCESS_DEBUG

/**
 * @brief Generate depth remappings for repetition and definition levels.
 *
 * When dealing with columns that contain lists, we must examine incoming
 * repetition and definition level pairs to determine what range of output nesting
 * is indicated when adding new values.  This function generates the mappings of
 * the R/D levels to those start/end bounds
 *
 * @param remap Maps column schema index to the R/D remapping vectors for that column
 * @param src_col_schema The column schema to generate the new mapping for
 * @param md File metadata information
 */
void generate_depth_remappings(std::map<int, std::pair<std::vector<int>, std::vector<int>>>& remap,
                               int src_col_schema,
                               aggregate_reader_metadata const& md)
{
  // already generated for this level
  if (remap.find(src_col_schema) != remap.end()) { return; }
  auto schema   = md.get_schema(src_col_schema);
  int max_depth = md.get_output_nesting_depth(src_col_schema);

  CUDF_EXPECTS(remap.find(src_col_schema) == remap.end(),
               "Attempting to remap a schema more than once");
  auto inserted =
    remap.insert(std::pair<int, std::pair<std::vector<int>, std::vector<int>>>{src_col_schema, {}});
  auto& depth_remap = inserted.first->second;

  std::vector<int>& rep_depth_remap = (depth_remap.first);
  rep_depth_remap.resize(schema.max_repetition_level + 1);
  std::vector<int>& def_depth_remap = (depth_remap.second);
  def_depth_remap.resize(schema.max_definition_level + 1);

  // the key:
  // for incoming level values  R/D
  // add values starting at the shallowest nesting level X has repetition level R
  // until you reach the deepest nesting level Y that corresponds to the repetition level R1
  // held by the nesting level that has definition level D
  //
  // Example: a 3 level struct with a list at the bottom
  //
  //                     R / D   Depth
  // level0              0 / 1     0
  //   level1            0 / 2     1
  //     level2          0 / 3     2
  //       list          0 / 3     3
  //         element     1 / 4     4
  //
  // incoming R/D : 0, 0  -> add values from depth 0 to 3   (def level 0 always maps to depth 0)
  // incoming R/D : 0, 1  -> add values from depth 0 to 3
  // incoming R/D : 0, 2  -> add values from depth 0 to 3
  // incoming R/D : 1, 4  -> add values from depth 4 to 4
  //
  // Note : the -validity- of values is simply checked by comparing the incoming D value against the
  // D value of the given nesting level (incoming D >= the D for the nesting level == valid,
  // otherwise NULL).  The tricky part is determining what nesting levels to add values at.
  //
  // For schemas with no repetition level (no lists), X is always 0 and Y is always max nesting
  // depth.
  //

  // compute "X" from above
  for (int s_idx = schema.max_repetition_level; s_idx >= 0; s_idx--) {
    auto find_shallowest = [&](int r) {
      int shallowest = -1;
      int cur_depth  = max_depth - 1;
      int schema_idx = src_col_schema;
      while (schema_idx > 0) {
        auto cur_schema = md.get_schema(schema_idx);
        if (cur_schema.max_repetition_level == r) {
          // if this is a repeated field, map it one level deeper
          shallowest = cur_schema.is_stub() ? cur_depth + 1 : cur_depth;
        }
        // if it's one-level encoding list
        else if (cur_schema.is_one_level_list(md.get_schema(cur_schema.parent_idx))) {
          shallowest = cur_depth - 1;
        }
        if (!cur_schema.is_stub()) { cur_depth--; }
        schema_idx = cur_schema.parent_idx;
      }
      return shallowest;
    };
    rep_depth_remap[s_idx] = find_shallowest(s_idx);
  }

  // compute "Y" from above
  for (int s_idx = schema.max_definition_level; s_idx >= 0; s_idx--) {
    auto find_deepest = [&](int d) {
      SchemaElement prev_schema;
      int schema_idx = src_col_schema;
      int r1         = 0;
      while (schema_idx > 0) {
        SchemaElement cur_schema = md.get_schema(schema_idx);
        if (cur_schema.max_definition_level == d) {
          // if this is a repeated field, map it one level deeper
          r1 = cur_schema.is_stub() ? prev_schema.max_repetition_level
                                    : cur_schema.max_repetition_level;
          break;
        }
        prev_schema = cur_schema;
        schema_idx  = cur_schema.parent_idx;
      }

      // we now know R1 from above. return the deepest nesting level that has the
      // same repetition level
      schema_idx = src_col_schema;
      int depth  = max_depth - 1;
      while (schema_idx > 0) {
        SchemaElement cur_schema = md.get_schema(schema_idx);
        if (cur_schema.max_repetition_level == r1) {
          // if this is a repeated field, map it one level deeper
          depth = cur_schema.is_stub() ? depth + 1 : depth;
          break;
        }
        if (!cur_schema.is_stub()) { depth--; }
        prev_schema = cur_schema;
        schema_idx  = cur_schema.parent_idx;
      }
      return depth;
    };
    def_depth_remap[s_idx] = find_deepest(s_idx);
  }
}

/**
 * @brief Reads compressed page data to device memory.
 *
 * @param sources Dataset sources
 * @param page_data Buffers to hold compressed page data for each chunk
 * @param chunks List of column chunk descriptors
 * @param begin_chunk Index of first column chunk to read
 * @param end_chunk Index after the last column chunk to read
 * @param column_chunk_offsets File offset for all chunks
 * @param chunk_source_map Association between each column chunk and its source
 * @param stream CUDA stream used for device memory operations and kernel launches
 *
 * @return A future object for reading synchronization
 */
[[nodiscard]] std::future<void> read_column_chunks_async(
  std::vector<std::unique_ptr<datasource>> const& sources,
  std::vector<std::unique_ptr<datasource::buffer>>& page_data,
  cudf::detail::hostdevice_vector<ColumnChunkDesc>& chunks,
  size_t begin_chunk,
  size_t end_chunk,
  std::vector<size_t> const& column_chunk_offsets,
  std::vector<size_type> const& chunk_source_map,
  rmm::cuda_stream_view stream)
{
  // Transfer chunk data, coalescing adjacent chunks
  std::vector<std::future<size_t>> read_tasks;
  for (size_t chunk = begin_chunk; chunk < end_chunk;) {
    size_t const io_offset   = column_chunk_offsets[chunk];
    size_t io_size           = chunks[chunk].compressed_size;
    size_t next_chunk        = chunk + 1;
    bool const is_compressed = (chunks[chunk].codec != Compression::UNCOMPRESSED);
    while (next_chunk < end_chunk) {
      size_t const next_offset      = column_chunk_offsets[next_chunk];
      bool const is_next_compressed = (chunks[next_chunk].codec != Compression::UNCOMPRESSED);
      if (next_offset != io_offset + io_size || is_next_compressed != is_compressed ||
          chunk_source_map[chunk] != chunk_source_map[next_chunk]) {
        // Can't merge if not contiguous or mixing compressed and uncompressed
        // Not coalescing uncompressed with compressed chunks is so that compressed buffers can be
        // freed earlier (immediately after decompression stage) to limit peak memory requirements
        break;
      }
      io_size += chunks[next_chunk].compressed_size;
      next_chunk++;
    }
    if (io_size != 0) {
      auto& source = sources[chunk_source_map[chunk]];
      if (source->is_device_read_preferred(io_size)) {
        // Buffer needs to be padded.
        // Required by `gpuDecodePageData`.
        auto buffer =
          rmm::device_buffer(cudf::util::round_up_safe(io_size, BUFFER_PADDING_MULTIPLE), stream);
        auto fut_read_size = source->device_read_async(
          io_offset, io_size, static_cast<uint8_t*>(buffer.data()), stream);
        read_tasks.emplace_back(std::move(fut_read_size));
        page_data[chunk] = datasource::buffer::create(std::move(buffer));
      } else {
        auto const read_buffer = source->host_read(io_offset, io_size);
        // Buffer needs to be padded.
        // Required by `gpuDecodePageData`.
        auto tmp_buffer = rmm::device_buffer(
          cudf::util::round_up_safe(read_buffer->size(), BUFFER_PADDING_MULTIPLE), stream);
        CUDF_CUDA_TRY(hipMemcpyAsync(
          tmp_buffer.data(), read_buffer->data(), read_buffer->size(), hipMemcpyDefault, stream));
        page_data[chunk] = datasource::buffer::create(std::move(tmp_buffer));
      }
      auto d_compdata = page_data[chunk]->data();
      do {
        chunks[chunk].compressed_data = d_compdata;
        d_compdata += chunks[chunk].compressed_size;
      } while (++chunk != next_chunk);
    } else {
      chunk = next_chunk;
    }
  }
  auto sync_fn = [](decltype(read_tasks) read_tasks) {
    for (auto& task : read_tasks) {
      task.wait();
    }
  };
  return std::async(std::launch::deferred, sync_fn, std::move(read_tasks));
}

/**
 * @brief Return the number of total pages from the given column chunks.
 *
 * @param chunks List of column chunk descriptors
 * @param stream CUDA stream used for device memory operations and kernel launches
 *
 * @return The total number of pages
 */
[[nodiscard]] size_t count_page_headers(cudf::detail::hostdevice_vector<ColumnChunkDesc>& chunks,
                                        rmm::cuda_stream_view stream)
{
  size_t total_pages = 0;

  kernel_error error_code(stream);
  chunks.host_to_device_async(stream);
  DecodePageHeaders(chunks.device_ptr(), nullptr, chunks.size(), error_code.data(), stream);
  chunks.device_to_host_sync(stream);

  if (error_code.value() != 0) {
    CUDF_FAIL("Parquet header parsing failed with code(s) " + error_code.str());
  }

  for (size_t c = 0; c < chunks.size(); c++) {
    total_pages += chunks[c].num_data_pages + chunks[c].num_dict_pages;
  }

  return total_pages;
}

/**
 * @brief Decode the page information for a given pass.
 *
 * @param pass_intermediate_data The struct containing pass information
 *
 */
void decode_page_headers(pass_intermediate_data& pass, rmm::cuda_stream_view stream)
{
  cudf::detail::hostdevice_vector<chunk_page_info> chunk_page_info(pass.chunks.size(), stream);

  // IMPORTANT : if you change how pages are stored within a chunk (dist pages, then data pages),
  // please update preprocess_nested_columns to reflect this.
  for (size_t c = 0, page_count = 0; c < pass.chunks.size(); c++) {
    pass.chunks[c].max_num_pages = pass.chunks[c].num_data_pages + pass.chunks[c].num_dict_pages;
    chunk_page_info[c].pages     = pass.pages.device_ptr(page_count);
    page_count += pass.chunks[c].max_num_pages;
  }

  kernel_error error_code(stream);
  pass.chunks.host_to_device_async(stream);
  chunk_page_info.host_to_device_async(stream);
  DecodePageHeaders(pass.chunks.device_ptr(),
                    chunk_page_info.device_ptr(),
                    pass.chunks.size(),
                    error_code.data(),
                    stream);

  if (error_code.value() != 0) {
    // TODO(ets): if an unsupported encoding was detected, do extra work to figure out which one
    CUDF_FAIL("Parquet header parsing failed with code(s)" + error_code.str());
  }

  // compute max bytes needed for level data
  auto level_bit_size = cudf::detail::make_counting_transform_iterator(
    0, [chunks = pass.chunks.d_begin()] __device__(int i) {
      auto c = chunks[i];
      return static_cast<int>(
        max(c.level_bits[level_type::REPETITION], c.level_bits[level_type::DEFINITION]));
    });
  // max level data bit size.
  int const max_level_bits = thrust::reduce(rmm::exec_policy(stream),
                                            level_bit_size,
                                            level_bit_size + pass.chunks.size(),
                                            0,
                                            thrust::maximum<int>());
  pass.level_type_size     = std::max(1, cudf::util::div_rounding_up_safe(max_level_bits, 8));

  // sort the pages in schema order.
  //
  // ordering of pages is by input column schema, repeated across row groups.  so
  // if we had 3 columns, each with 2 pages, and 1 row group, our schema values might look like
  //
  // 1, 1, 2, 2, 3, 3
  //
  // However, if we had more than one row group, the pattern would be
  //
  // 1, 1, 2, 2, 3, 3, 1, 1, 2, 2, 3, 3
  // ^ row group 0     |
  //                   ^ row group 1
  //
  // To process pages by key (exclusive_scan_by_key, reduce_by_key, etc), the ordering we actually
  // want is
  //
  // 1, 1, 1, 1, 2, 2, 2, 2, 3, 3, 3, 3
  //
  // We also need to preserve key-relative page ordering, so we need to use a stable sort.
  {
    rmm::device_uvector<int32_t> page_keys{pass.pages.size(), stream};
    thrust::transform(rmm::exec_policy(stream),
                      pass.pages.d_begin(),
                      pass.pages.d_begin() + pass.pages.size(),
                      page_keys.begin(),
                      [] __device__(PageInfo const& page) { return page.src_col_schema; });
    thrust::stable_sort_by_key(rmm::exec_policy(stream),
                               page_keys.begin(),
                               page_keys.end(),
                               pass.pages.d_begin(),
                               thrust::less<int>());
  }

  // compute offsets to each group of input pages.
  // page_keys:   1, 1, 1, 1, 2, 2, 2, 2, 3, 3, 3, 3
  //
  // result:      0,          4,          8
  rmm::device_uvector<size_type> page_counts(pass.pages.size() + 1, stream);
  auto page_keys             = make_page_key_iterator(pass.pages);
  auto const page_counts_end = thrust::reduce_by_key(rmm::exec_policy(stream),
                                                     page_keys,
                                                     page_keys + pass.pages.size(),
                                                     thrust::make_constant_iterator(1),
                                                     thrust::make_discard_iterator(),
                                                     page_counts.begin())
                                 .second;
  auto const num_page_counts = page_counts_end - page_counts.begin();
  pass.page_offsets = cudf::detail::hostdevice_vector<size_type>(num_page_counts + 1, stream);
  thrust::exclusive_scan(rmm::exec_policy(stream),
                         page_counts.begin(),
                         page_counts.begin() + num_page_counts + 1,
                         pass.page_offsets.d_begin());

  // setup dict_page for each chunk if necessary
  thrust::for_each(rmm::exec_policy(stream),
                   pass.pages.d_begin(),
                   pass.pages.d_end(),
                   [chunks = pass.chunks.d_begin()] __device__(PageInfo const& p) {
                     if (p.flags & PAGEINFO_FLAGS_DICTIONARY) {
                       chunks[p.chunk_idx].dict_page = &p;
                     }
                   });

  pass.page_offsets.device_to_host_async(stream);
  pass.pages.device_to_host_async(stream);
  pass.chunks.device_to_host_async(stream);
  stream.synchronize();

  // validate page encodings
  CUDF_EXPECTS(std::all_of(pass.pages.begin(),
                           pass.pages.end(),
                           [](auto const& page) { return is_supported_encoding(page.encoding); }),
               "Unsupported page encoding detected");
}

}  // namespace

void reader::impl::build_string_dict_indices()
{
  auto& pass = *_pass_itm_data;

  auto is_dict_chunk = [](ColumnChunkDesc const& chunk) {
    return (chunk.data_type & 0x7) == BYTE_ARRAY && chunk.num_dict_pages > 0;
  };

  // Count the number of string dictionary entries
  // NOTE: Assumes first page in the chunk is always the dictionary page
  size_t total_str_dict_indexes = 0;
  for (size_t c = 0, page_count = 0; c < pass.chunks.size(); c++) {
    if (is_dict_chunk(pass.chunks[c])) {
      total_str_dict_indexes += pass.pages[page_count].num_input_values;
    }
    page_count += pass.chunks[c].max_num_pages;
  }

  // Build index for string dictionaries since they can't be indexed
  // directly due to variable-sized elements
  pass.str_dict_index = cudf::detail::make_zeroed_device_uvector_async<string_index_pair>(
    total_str_dict_indexes, _stream, rmm::mr::get_current_device_resource());

  // Update chunks with pointers to string dict indices
  for (size_t c = 0, page_count = 0, str_ofs = 0; c < pass.chunks.size(); c++) {
    input_column_info const& input_col = _input_columns[pass.chunks[c].src_col_index];
    CUDF_EXPECTS(input_col.schema_idx == pass.chunks[c].src_col_schema,
                 "Column/page schema index mismatch");
    if (is_dict_chunk(pass.chunks[c])) {
      pass.chunks[c].str_dict_index = pass.str_dict_index.data() + str_ofs;
      str_ofs += pass.pages[page_count].num_input_values;
    }

    // column_data_base will always point to leaf data, even for nested types.
    page_count += pass.chunks[c].max_num_pages;
  }

  if (total_str_dict_indexes > 0) {
    pass.chunks.host_to_device_async(_stream);
    BuildStringDictionaryIndex(pass.chunks.device_ptr(), pass.chunks.size(), _stream);
  }
}

void reader::impl::allocate_nesting_info()
{
  auto& pass    = *_pass_itm_data;
  auto& subpass = *pass.subpass;

  // auto const& chunks             = pass.chunks;
  auto const num_columns         = _input_columns.size();
  auto& pages                    = subpass.pages;
  auto& page_nesting_info        = subpass.page_nesting_info;
  auto& page_nesting_decode_info = subpass.page_nesting_decode_info;

  // compute total # of page_nesting infos needed and allocate space. doing this in one
  // buffer to keep it to a single gpu allocation
  auto counting_iter = thrust::make_counting_iterator(size_t{0});
  size_t const total_page_nesting_infos =
    std::accumulate(counting_iter, counting_iter + num_columns, 0, [&](int total, size_t index) {
      // the schema of the input column
      auto const schema_idx = _input_columns[index].schema_idx;
      auto const& schema    = _metadata->get_schema(schema_idx);
      auto const per_page_nesting_info_size =
        max(schema.max_definition_level + 1, _metadata->get_output_nesting_depth(schema_idx));
      return total + (per_page_nesting_info_size * subpass.column_page_count[index]);
    });

  page_nesting_info =
    cudf::detail::hostdevice_vector<PageNestingInfo>{total_page_nesting_infos, _stream};
  page_nesting_decode_info =
    cudf::detail::hostdevice_vector<PageNestingDecodeInfo>{total_page_nesting_infos, _stream};

  // update pointers in the PageInfos
  int target_page_index = 0;
  int src_info_index    = 0;
  for (size_t idx = 0; idx < _input_columns.size(); idx++) {
    auto const src_col_schema             = _input_columns[idx].schema_idx;
    auto& schema                          = _metadata->get_schema(src_col_schema);
    auto const per_page_nesting_info_size = std::max(
      schema.max_definition_level + 1, _metadata->get_output_nesting_depth(src_col_schema));

    for (size_t p_idx = 0; p_idx < subpass.column_page_count[idx]; p_idx++) {
      pages[target_page_index + p_idx].nesting = page_nesting_info.device_ptr() + src_info_index;
      pages[target_page_index + p_idx].nesting_decode =
        page_nesting_decode_info.device_ptr() + src_info_index;

      pages[target_page_index + p_idx].nesting_info_size = per_page_nesting_info_size;
      pages[target_page_index + p_idx].num_output_nesting_levels =
        _metadata->get_output_nesting_depth(src_col_schema);

      src_info_index += per_page_nesting_info_size;
    }
    target_page_index += subpass.column_page_count[idx];
  }

  // fill in
  int nesting_info_index = 0;
  std::map<int, std::pair<std::vector<int>, std::vector<int>>> depth_remapping;
  for (size_t idx = 0; idx < _input_columns.size(); idx++) {
    auto const src_col_schema = _input_columns[idx].schema_idx;

    // schema of the input column
    auto& schema = _metadata->get_schema(src_col_schema);
    // real depth of the output cudf column hierarchy (1 == no nesting, 2 == 1 level, etc)
    int max_depth = _metadata->get_output_nesting_depth(src_col_schema);

    // # of nesting infos stored per page for this column
    auto const per_page_nesting_info_size = std::max(schema.max_definition_level + 1, max_depth);

    // if this column has lists, generate depth remapping
    std::map<int, std::pair<std::vector<int>, std::vector<int>>> depth_remapping;
    if (schema.max_repetition_level > 0) {
      generate_depth_remappings(depth_remapping, src_col_schema, *_metadata);
    }

    // fill in host-side nesting info
    int schema_idx  = src_col_schema;
    auto cur_schema = _metadata->get_schema(schema_idx);
    int cur_depth   = max_depth - 1;
    while (schema_idx > 0) {
      // stub columns (basically the inner field of a list scheme element) are not real columns.
      // we can ignore them for the purposes of output nesting info
      if (!cur_schema.is_stub()) {
        // initialize each page within the chunk
        for (size_t p_idx = 0; p_idx < subpass.column_page_count[idx]; p_idx++) {
          PageNestingInfo* pni =
            &page_nesting_info[nesting_info_index + (p_idx * per_page_nesting_info_size)];

          PageNestingDecodeInfo* nesting_info =
            &page_nesting_decode_info[nesting_info_index + (p_idx * per_page_nesting_info_size)];

          // if we have lists, set our start and end depth remappings
          if (schema.max_repetition_level > 0) {
            auto remap = depth_remapping.find(src_col_schema);
            CUDF_EXPECTS(remap != depth_remapping.end(),
                         "Could not find depth remapping for schema");
            std::vector<int> const& rep_depth_remap = (remap->second.first);
            std::vector<int> const& def_depth_remap = (remap->second.second);

            for (size_t m = 0; m < rep_depth_remap.size(); m++) {
              nesting_info[m].start_depth = rep_depth_remap[m];
            }
            for (size_t m = 0; m < def_depth_remap.size(); m++) {
              nesting_info[m].end_depth = def_depth_remap[m];
            }
          }

          // values indexed by output column index
          nesting_info[cur_depth].max_def_level = cur_schema.max_definition_level;
          pni[cur_depth].size                   = 0;
          pni[cur_depth].type =
            to_type_id(cur_schema, _strings_to_categorical, _timestamp_type.id());
          pni[cur_depth].nullable = cur_schema.repetition_type == OPTIONAL;
        }

        // move up the hierarchy
        cur_depth--;
      }

      // next schema
      schema_idx = cur_schema.parent_idx;
      cur_schema = _metadata->get_schema(schema_idx);
    }

    nesting_info_index += (per_page_nesting_info_size * subpass.column_page_count[idx]);
  }

  // copy nesting info to the device
  page_nesting_info.host_to_device_async(_stream);
  page_nesting_decode_info.host_to_device_async(_stream);
}

void reader::impl::allocate_level_decode_space()
{
  auto& pass    = *_pass_itm_data;
  auto& subpass = *pass.subpass;

  auto& pages = subpass.pages;

  // TODO: this could be made smaller if we ignored dictionary pages and pages with no
  // repetition data.
  size_t const per_page_decode_buf_size = LEVEL_DECODE_BUF_SIZE * 2 * pass.level_type_size;
  auto const decode_buf_size            = per_page_decode_buf_size * pages.size();
  subpass.level_decode_data =
    rmm::device_buffer(decode_buf_size, _stream, rmm::mr::get_current_device_resource());

  // distribute the buffers
  uint8_t* buf = static_cast<uint8_t*>(subpass.level_decode_data.data());
  for (size_t idx = 0; idx < pages.size(); idx++) {
    auto& p = pages[idx];

    p.lvl_decode_buf[level_type::DEFINITION] = buf;
    buf += (LEVEL_DECODE_BUF_SIZE * pass.level_type_size);
    p.lvl_decode_buf[level_type::REPETITION] = buf;
    buf += (LEVEL_DECODE_BUF_SIZE * pass.level_type_size);
  }
}

std::pair<bool, std::vector<std::future<void>>> reader::impl::read_column_chunks()
{
  auto const& row_groups_info = _pass_itm_data->row_groups;

  auto& raw_page_data = _pass_itm_data->raw_page_data;
  auto& chunks        = _pass_itm_data->chunks;

  // Descriptors for all the chunks that make up the selected columns
  auto const num_input_columns = _input_columns.size();
  auto const num_chunks        = row_groups_info.size() * num_input_columns;

  // Association between each column chunk and its source
  std::vector<size_type> chunk_source_map(num_chunks);

  // Tracker for eventually deallocating compressed and uncompressed data
  raw_page_data = std::vector<std::unique_ptr<datasource::buffer>>(num_chunks);

  // Keep track of column chunk file offsets
  std::vector<size_t> column_chunk_offsets(num_chunks);

  // Initialize column chunk information
  size_t total_decompressed_size = 0;
  // TODO: make this respect the pass-wide skip_rows/num_rows instead of the file-wide
  // skip_rows/num_rows
  // auto remaining_rows            = num_rows;
  std::vector<std::future<void>> read_chunk_tasks;
  size_type chunk_count = 0;
  for (auto const& rg : row_groups_info) {
    auto const& row_group       = _metadata->get_row_group(rg.index, rg.source_index);
    auto const row_group_source = rg.source_index;
    // auto const row_group_rows   = std::min<int>(remaining_rows, row_group.num_rows);

    // generate ColumnChunkDesc objects for everything to be decoded (all input columns)
    for (size_t i = 0; i < num_input_columns; ++i) {
      auto const& col = _input_columns[i];
      // look up metadata
      auto& col_meta = _metadata->get_column_metadata(rg.index, rg.source_index, col.schema_idx);

      column_chunk_offsets[chunk_count] =
        (col_meta.dictionary_page_offset != 0)
          ? std::min(col_meta.data_page_offset, col_meta.dictionary_page_offset)
          : col_meta.data_page_offset;

      // Map each column chunk to its column index and its source index
      chunk_source_map[chunk_count] = row_group_source;

      if (col_meta.codec != Compression::UNCOMPRESSED) {
        total_decompressed_size += col_meta.total_uncompressed_size;
      }

      chunk_count++;
    }
    // remaining_rows -= row_group_rows;
  }

  // Read compressed chunk data to device memory
  read_chunk_tasks.push_back(read_column_chunks_async(_sources,
                                                      raw_page_data,
                                                      chunks,
                                                      0,
                                                      chunks.size(),
                                                      column_chunk_offsets,
                                                      chunk_source_map,
                                                      _stream));

  // CUDF_EXPECTS(remaining_rows == 0, "All rows data must be read.");

  return {total_decompressed_size > 0, std::move(read_chunk_tasks)};
}

void reader::impl::load_compressed_data()
{
  auto& pass = *_pass_itm_data;

  // This function should never be called if `num_rows == 0`.
  // CUDF_EXPECTS(_pass_itm_data->num_rows > 0, "Number of reading rows must not be zero.");

  auto& chunks = pass.chunks;
  auto& pages  = pass.pages;

  auto const [has_compressed_data, read_chunks_tasks] = read_column_chunks();
  pass.has_compressed_data                            = has_compressed_data;

  for (auto& task : read_chunks_tasks) {
    task.wait();
  }

  // Process dataset chunk pages into output columns
  auto const total_pages = count_page_headers(chunks, _stream);
  if (total_pages <= 0) { return; }
  pages = cudf::detail::hostdevice_vector<PageInfo>(total_pages, total_pages, _stream);

  // decoding of column/page information
  decode_page_headers(pass, _stream);
  CUDF_EXPECTS(pass.page_offsets.size() - 1 == static_cast<size_t>(_input_columns.size()),
               "Encountered page_offsets / num_columns mismatch");
}

namespace {

struct cumulative_row_info {
  size_t row_count;   // cumulative row count
  size_t size_bytes;  // cumulative size in bytes
  int key;            // schema index
};

struct get_page_chunk_idx {
  __device__ size_type operator()(PageInfo const& page) { return page.chunk_idx; }
};

struct get_page_num_rows {
  __device__ size_type operator()(PageInfo const& page) { return page.num_rows; }
};

struct input_col_info {
  int const schema_idx;
  size_type const nesting_depth;
};

/**
 * @brief Converts a 1-dimensional index into page, depth and column indices used in
 * allocate_columns to compute columns sizes.
 *
 * The input index will iterate through pages, nesting depth and column indices in that order.
 */
struct reduction_indices {
  size_t const page_idx;
  size_type const depth_idx;
  size_type const col_idx;

  __device__ reduction_indices(size_t index_, size_type max_depth_, size_t num_pages_)
    : page_idx(index_ % num_pages_),
      depth_idx((index_ / num_pages_) % max_depth_),
      col_idx(index_ / (max_depth_ * num_pages_))
  {
  }
};

/**
 * @brief Returns the size field of a PageInfo struct for a given depth, keyed by schema.
 */
struct get_page_nesting_size {
  input_col_info const* const input_cols;
  size_type const max_depth;
  size_t const num_pages;
  PageInfo const* const pages;

  __device__ size_type operator()(size_t index) const
  {
    auto const indices = reduction_indices{index, max_depth, num_pages};

    auto const& page = pages[indices.page_idx];
    if (page.src_col_schema != input_cols[indices.col_idx].schema_idx ||
        page.flags & PAGEINFO_FLAGS_DICTIONARY ||
        indices.depth_idx >= input_cols[indices.col_idx].nesting_depth) {
      return 0;
    }

    return page.nesting[indices.depth_idx].batch_size;
  }
};

struct get_reduction_key {
  size_t const num_pages;
  __device__ size_t operator()(size_t index) const { return index / num_pages; }
};

/**
 * @brief Writes to the chunk_row field of the PageInfo struct.
 */
struct chunk_row_output_iter {
  PageInfo* p;
  using value_type        = size_type;
  using difference_type   = size_type;
  using pointer           = size_type*;
  using reference         = size_type&;
  using iterator_category = thrust::output_device_iterator_tag;

  __host__ __device__ chunk_row_output_iter operator+(int i)
  {
    return chunk_row_output_iter{p + i};
  }

  __host__ __device__ void operator++() { p++; }

  __device__ reference operator[](int i) { return p[i].chunk_row; }
  __device__ reference operator*() { return p->chunk_row; }
};

/**
 * @brief Writes to the page_start_value field of the PageNestingInfo struct, keyed by schema.
 */
struct start_offset_output_iterator {
  PageInfo const* pages;
  size_t cur_index;
  input_col_info const* input_cols;
  size_type max_depth;
  size_t num_pages;
  int empty               = 0;
  using value_type        = size_type;
  using difference_type   = size_type;
  using pointer           = size_type*;
  using reference         = size_type&;
  using iterator_category = thrust::output_device_iterator_tag;

  constexpr void operator=(start_offset_output_iterator const& other)
  {
    pages      = other.pages;
    cur_index  = other.cur_index;
    input_cols = other.input_cols;
    max_depth  = other.max_depth;
    num_pages  = other.num_pages;
  }

  constexpr start_offset_output_iterator operator+(size_t i)
  {
    return start_offset_output_iterator{pages, cur_index + i, input_cols, max_depth, num_pages};
  }

  constexpr void operator++() { cur_index++; }

  __device__ reference operator[](size_t i) { return dereference(cur_index + i); }
  __device__ reference operator*() { return dereference(cur_index); }

 private:
  __device__ reference dereference(size_t index)
  {
    auto const indices = reduction_indices{index, max_depth, num_pages};

    PageInfo const& p = pages[indices.page_idx];
    if (p.src_col_schema != input_cols[indices.col_idx].schema_idx ||
        p.flags & PAGEINFO_FLAGS_DICTIONARY ||
        indices.depth_idx >= input_cols[indices.col_idx].nesting_depth) {
      return empty;
    }
    return p.nesting_decode[indices.depth_idx].page_start_value;
  }
};

struct page_to_string_size {
  ColumnChunkDesc const* chunks;

  __device__ size_t operator()(PageInfo const& page) const
  {
    auto const chunk = chunks[page.chunk_idx];

    if (not is_string_col(chunk) || (page.flags & PAGEINFO_FLAGS_DICTIONARY) != 0) { return 0; }
    return page.str_bytes;
  }
};

struct page_offset_output_iter {
  PageInfo* p;

  using value_type        = size_type;
  using difference_type   = size_type;
  using pointer           = size_type*;
  using reference         = size_type&;
  using iterator_category = thrust::output_device_iterator_tag;

  __host__ __device__ page_offset_output_iter operator+(int i)
  {
    return page_offset_output_iter{p + i};
  }

  __host__ __device__ void operator++() { p++; }

  __device__ reference operator[](int i) { return p[i].str_offset; }
  __device__ reference operator*() { return p->str_offset; }
};

}  // anonymous namespace

void reader::impl::preprocess_file(
  int64_t skip_rows,
  std::optional<size_type> const& num_rows,
  host_span<std::vector<size_type> const> row_group_indices,
  std::optional<std::reference_wrapper<ast::expression const>> filter)
{
  CUDF_EXPECTS(!_file_preprocessed, "Attempted to preprocess file more than once");

  // if filter is not empty, then create output types as vector and pass for filtering.
  std::vector<data_type> output_types;
  if (filter.has_value()) {
    std::transform(_output_buffers.cbegin(),
                   _output_buffers.cend(),
                   std::back_inserter(output_types),
                   [](auto const& col) { return col.type; });
  }
  std::tie(
    _file_itm_data.global_skip_rows, _file_itm_data.global_num_rows, _file_itm_data.row_groups) =
    _metadata->select_row_groups(
      row_group_indices, skip_rows, num_rows, output_types, filter, _stream);

  if (_file_itm_data.global_num_rows > 0 && not _file_itm_data.row_groups.empty() &&
      not _input_columns.empty()) {
    // fills in chunk information without physically loading or decompressing
    // the associated data
    create_global_chunk_info();

    // compute schedule of input reads.
    compute_input_passes();
  }

  _file_preprocessed = true;
}

// update chunk_row field in subpass page from pass page
struct update_subpass_chunk_row {
  device_span<PageInfo> pass_pages;
  device_span<PageInfo> subpass_pages;
  device_span<size_t> page_src_index;

  void operator()(size_t i)
  {
    subpass_pages[i].chunk_row = pass_pages[page_src_index[i]].chunk_row;
  }
};

// update num_rows field from pass page to subpass page
struct update_pass_num_rows {
  device_span<PageInfo> pass_pages;
  device_span<PageInfo> subpass_pages;
  device_span<size_t> page_src_index;

  __device__ void operator()(size_t i)
  {
    pass_pages[page_src_index[i]].num_rows = subpass_pages[i].num_rows;
  }
};

void reader::impl::preprocess_subpass_pages(bool uses_custom_row_bounds, size_t chunk_read_limit)
{
  auto& pass    = *_pass_itm_data;
  auto& subpass = *pass.subpass;

  // iterate over all input columns and determine if they contain lists.
  // TODO: we could do this once at the file level instead of every time we get in here. the set of
  // columns we are processing does not change over multiple passes/subpasses/output chunks.
  bool has_lists = false;
  for (size_t idx = 0; idx < _input_columns.size(); idx++) {
    auto const& input_col  = _input_columns[idx];
    size_t const max_depth = input_col.nesting_depth();

    auto* cols = &_output_buffers;
    for (size_t l_idx = 0; l_idx < max_depth; l_idx++) {
      auto& out_buf = (*cols)[input_col.nesting[l_idx]];
      cols          = &out_buf.children;

      // if this has a list parent, we have to get column sizes from the
      // data computed during ComputePageSizes
      if (out_buf.user_data & PARQUET_COLUMN_BUFFER_FLAG_HAS_LIST_PARENT) {
        has_lists = true;
        break;
      }
    }
    if (has_lists) { break; }
  }

  // in some cases we will need to do further preprocessing of pages.
  // - if we have lists, the num_rows field in PageInfo will be incorrect coming out of the file
  // - if we are doing a chunked read, we need to compute the size of all string data
  if (has_lists || chunk_read_limit > 0) {
    // computes:
    // PageNestingInfo::num_rows for each page. the true number of rows (taking repetition into
    // account), not just the number of values. PageNestingInfo::size for each level of nesting, for
    // each page.
    //
    // we will be applying a later "trim" pass if skip_rows/num_rows is being used, which can happen
    // if:
    // - user has passed custom row bounds
    // - we will be doing a chunked read
    ComputePageSizes(subpass.pages,
                     pass.chunks,
                     0,  // 0-max size_t. process all possible rows
                     std::numeric_limits<size_t>::max(),
                     true,                  // compute num_rows
                     chunk_read_limit > 0,  // compute string sizes
                     _pass_itm_data->level_type_size,
                     _stream);
  }

  // copy our now-correct row counts  back to the base pages stored in the pass.
  auto iter = thrust::make_counting_iterator(0);
  thrust::for_each(rmm::exec_policy(_stream),
                   iter,
                   iter + subpass.pages.size(),
                   update_pass_num_rows{pass.pages, subpass.pages, subpass.page_src_index});

  // computes:
  // PageInfo::chunk_row (the chunk-relative row index) for all pages in the pass. The start_row
  // field in ColumnChunkDesc is the absolute row index for the whole file. chunk_row in PageInfo is
  // relative to the beginning of the chunk. so in the kernels, chunk.start_row + page.chunk_row
  // gives us the absolute row index. NOTE: this is recomputing chunk_row for -all- pages in the
  // pass, not just the pages in the current subpass.  the reason we do this is that we may visit
  // the same page multiple times over multiple subpasses (if we didn't process all rows in a given
  // subpass). this greatly simplifies the logic.
  auto key_input  = thrust::make_transform_iterator(pass.pages.d_begin(), get_page_chunk_idx{});
  auto page_input = thrust::make_transform_iterator(pass.pages.d_begin(), get_page_num_rows{});
  thrust::exclusive_scan_by_key(rmm::exec_policy(_stream),
                                key_input,
                                key_input + pass.pages.size(),
                                page_input,
                                chunk_row_output_iter{pass.pages.device_ptr()});

  // finally, copy chunk row into the subpass.
  thrust::for_each(rmm::exec_policy(_stream),
                   iter,
                   iter + subpass.pages.size(),
                   update_subpass_chunk_row{pass.pages, subpass.pages, subpass.page_src_index});

  // retrieve pages back
  subpass.pages.device_to_host_sync(_stream);

  // at this point we have an accurate row count so we can compute how many rows we will actually be
  // able to decode for this pass. we will have selected a set of pages for each column in the
  // row group, but not every page will have the same number of rows. so, we can only read as many
  // rows as the smallest batch (by column) we have decompressed.
  size_t page_index = 0;
  size_t max_row    = std::numeric_limits<size_t>::max();
  for (size_t idx = 0; idx < subpass.column_page_count.size(); idx++) {
    auto const& last_page = subpass.pages[page_index + (subpass.column_page_count[idx] - 1)];
    auto const& chunk     = pass.chunks[last_page.chunk_idx];
    max_row =
      min(max_row, static_cast<size_t>(chunk.start_row + last_page.chunk_row + last_page.num_rows));
    page_index += subpass.column_page_count[idx];
  }
  CUDF_EXPECTS(max_row > pass.processed_rows, "Encountered invalid row read count");
  subpass.skip_rows = pass.skip_rows + pass.processed_rows;
  subpass.num_rows  = max_row - pass.processed_rows;

  // now split up the output into chunks as necessary
  compute_chunks_for_subpass();
}

void reader::impl::allocate_columns(size_t skip_rows, size_t num_rows, bool uses_custom_row_bounds)
{
  auto& pass    = *_pass_itm_data;
  auto& subpass = *pass.subpass;

  // Should not reach here if there is no page data.
  CUDF_EXPECTS(subpass.pages.size() > 0, "There are no pages present in the subpass");

  // computes:
  // PageNestingInfo::batch_size for each level of nesting, for each page, taking row bounds into
  // account. PageInfo::skipped_values, which tells us where to start decoding in the input to
  // respect the user bounds. It is only necessary to do this second pass if uses_custom_row_bounds
  // is set (if the user has specified artificial bounds).
  if (uses_custom_row_bounds) {
    ComputePageSizes(subpass.pages,
                     pass.chunks,
                     skip_rows,
                     num_rows,
                     false,  // num_rows is already computed
                     false,  // no need to compute string sizes
                     pass.level_type_size,
                     _stream);

    // print_pages(pages, _stream);
  }

  // iterate over all input columns and allocate any associated output
  // buffers if they are not part of a list hierarchy. mark down
  // if we have any list columns that need further processing.
  bool has_lists = false;
  for (size_t idx = 0; idx < _input_columns.size(); idx++) {
    auto const& input_col  = _input_columns[idx];
    size_t const max_depth = input_col.nesting_depth();

    auto* cols = &_output_buffers;
    for (size_t l_idx = 0; l_idx < max_depth; l_idx++) {
      auto& out_buf = (*cols)[input_col.nesting[l_idx]];
      cols          = &out_buf.children;

      // if this has a list parent, we have to get column sizes from the
      // data computed during ComputePageSizes
      if (out_buf.user_data & PARQUET_COLUMN_BUFFER_FLAG_HAS_LIST_PARENT) {
        has_lists = true;
      }
      // if we haven't already processed this column because it is part of a struct hierarchy
      else if (out_buf.size == 0) {
        // add 1 for the offset if this is a list column
        out_buf.create(
          out_buf.type.id() == type_id::LIST && l_idx < max_depth ? num_rows + 1 : num_rows,
          _stream,
          _mr);
      }
    }
  }

  // compute output column sizes by examining the pages of the -input- columns
  if (has_lists) {
    std::vector<input_col_info> h_cols_info;
    h_cols_info.reserve(_input_columns.size());
    std::transform(_input_columns.cbegin(),
                   _input_columns.cend(),
                   std::back_inserter(h_cols_info),
                   [](auto& col) -> input_col_info {
                     return {col.schema_idx, static_cast<size_type>(col.nesting_depth())};
                   });

    auto const max_depth =
      (*std::max_element(h_cols_info.cbegin(),
                         h_cols_info.cend(),
                         [](auto& l, auto& r) { return l.nesting_depth < r.nesting_depth; }))
        .nesting_depth;

    auto const d_cols_info = cudf::detail::make_device_uvector_async(
      h_cols_info, _stream, rmm::mr::get_current_device_resource());

    auto const num_keys = _input_columns.size() * max_depth * subpass.pages.size();
    // size iterator. indexes pages by sorted order
    rmm::device_uvector<size_type> size_input{num_keys, _stream};
    thrust::transform(
      rmm::exec_policy(_stream),
      thrust::make_counting_iterator<size_type>(0),
      thrust::make_counting_iterator<size_type>(num_keys),
      size_input.begin(),
      get_page_nesting_size{
        d_cols_info.data(), max_depth, subpass.pages.size(), subpass.pages.d_begin()});
    auto const reduction_keys =
      cudf::detail::make_counting_transform_iterator(0, get_reduction_key{subpass.pages.size()});
    cudf::detail::hostdevice_vector<size_t> sizes{_input_columns.size() * max_depth, _stream};

    // find the size of each column
    thrust::reduce_by_key(rmm::exec_policy(_stream),
                          reduction_keys,
                          reduction_keys + num_keys,
                          size_input.cbegin(),
                          thrust::make_discard_iterator(),
                          sizes.d_begin());

    // for nested hierarchies, compute per-page start offset
    thrust::exclusive_scan_by_key(
      rmm::exec_policy(_stream),
      reduction_keys,
      reduction_keys + num_keys,
      size_input.cbegin(),
      start_offset_output_iterator{
        subpass.pages.d_begin(), 0, d_cols_info.data(), max_depth, subpass.pages.size()});

    sizes.device_to_host_sync(_stream);
    for (size_type idx = 0; idx < static_cast<size_type>(_input_columns.size()); idx++) {
      auto const& input_col = _input_columns[idx];
      auto* cols            = &_output_buffers;
      for (size_type l_idx = 0; l_idx < static_cast<size_type>(input_col.nesting_depth());
           l_idx++) {
        auto& out_buf = (*cols)[input_col.nesting[l_idx]];
        cols          = &out_buf.children;
        // if this buffer is part of a list hierarchy, we need to determine it's
        // final size and allocate it here.
        //
        // for struct columns, higher levels of the output columns are shared between input
        // columns. so don't compute any given level more than once.
        if ((out_buf.user_data & PARQUET_COLUMN_BUFFER_FLAG_HAS_LIST_PARENT) && out_buf.size == 0) {
          auto size = sizes[(idx * max_depth) + l_idx];

          // if this is a list column add 1 for non-leaf levels for the terminating offset
          if (out_buf.type.id() == type_id::LIST && l_idx < max_depth) { size++; }

          // allocate
          out_buf.create(size, _stream, _mr);
        }
      }
    }
  }
}

std::vector<size_t> reader::impl::calculate_page_string_offsets()
{
  auto& pass    = *_pass_itm_data;
  auto& subpass = *pass.subpass;

  auto page_keys = make_page_key_iterator(subpass.pages);

  std::vector<size_t> col_sizes(_input_columns.size(), 0L);
  rmm::device_uvector<size_t> d_col_sizes(col_sizes.size(), _stream);

  // use page_index to fetch page string sizes in the proper order
  auto val_iter = thrust::make_transform_iterator(subpass.pages.d_begin(),
                                                  page_to_string_size{pass.chunks.d_begin()});

  // do scan by key to calculate string offsets for each page
  thrust::exclusive_scan_by_key(rmm::exec_policy(_stream),
                                page_keys,
                                page_keys + subpass.pages.size(),
                                val_iter,
                                page_offset_output_iter{subpass.pages.device_ptr()});

  // now sum up page sizes
  rmm::device_uvector<int> reduce_keys(col_sizes.size(), _stream);
  thrust::reduce_by_key(rmm::exec_policy(_stream),
                        page_keys,
                        page_keys + subpass.pages.size(),
                        val_iter,
                        reduce_keys.begin(),
                        d_col_sizes.begin());

  hipMemcpyAsync(col_sizes.data(),
                  d_col_sizes.data(),
                  sizeof(size_t) * col_sizes.size(),
                  hipMemcpyDeviceToHost,
                  _stream);
  _stream.synchronize();

  return col_sizes;
}

}  // namespace cudf::io::parquet::detail
