#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "parquet_gpu.cuh"

#include <cudf/detail/iterator.cuh>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/table/experimental/row_operators.cuh>

#include <rmm/exec_policy.hpp>

#include <hip/hip_cooperative_groups.h>
#include <cuda/atomic>

namespace cudf::io::parquet::detail {

namespace cg = cooperative_groups;

namespace {
constexpr int DEFAULT_BLOCK_SIZE = 256;
}

template <typename T>
struct equality_functor {
  column_device_view const& col;
  __device__ bool operator()(key_type const lhs_idx, key_type const rhs_idx) const
  {
    //  We don't call this for nulls so this is fine
    auto const equal  = cudf::experimental::row::equality::nan_equal_physical_equality_comparator{};
    auto const result = equal(col.element<T>(lhs_idx), col.element<T>(rhs_idx));
    printf("col_type_id:%d, equality idx1:%d, idx2:%d, eq:%d\n",
           col.type().id(),
           lhs_idx,
           rhs_idx,
           result);
    return result;
  }
};

template <typename T>
struct hash_functor {
  column_device_view const& col;
  __device__ auto operator()(key_type idx) const
  {
    auto const hashed = cudf::hashing::detail::MurmurHash3_x86_32<T>{}(col.element<T>(idx));
    printf("hashing idx: %d = %d\n", idx, hashed);
    return hashed;  // cudf::hashing::detail::MurmurHash3_x86_32<T>{}(col.element<T>(idx));
  }
};

struct map_insert_fn {
  storage_ref_type const& storage_ref;

  template <typename T>
  __device__ bool operator()(column_device_view const& col, key_type i)
  {
    if constexpr (column_device_view::has_element_accessor<T>()) {
      using equality_fn_type    = equality_functor<T>;
      using hash_fn_type        = hash_functor<T>;
      using probing_scheme_type = cuco::linear_probing<cg_size, hash_fn_type>;

      // Instantiate hash and equality functors.
      auto hash_fn  = hash_fn_type{col};
      auto equal_fn = equality_fn_type{col};

      // Make a view of the hash map
      cuco::static_map_ref<key_type,
                           mapped_type,
                           SCOPE,
                           equality_fn_type,
                           probing_scheme_type,
                           storage_ref_type>
        hash_map_ref{cuco::empty_key{KEY_SENTINEL},
                     cuco::empty_value{VALUE_SENTINEL},
                     {equal_fn},
                     {hash_fn},
                     {},
                     storage_ref};

      // Create another map with insert operator
      auto map_insert_ref = hash_map_ref.with_operators(cuco::insert_and_find);
      // Insert
      auto [iter, found] = map_insert_ref.insert_and_find(cuco::pair{i, i});
      printf("Inserted k=%d, v=%d, unique=%d\n", iter->first, iter->second, found);
      return found;
    } else {
      CUDF_UNREACHABLE("Unsupported type to insert in map");
    }
  }
};  // namespace cudf::io::parquet::detail

struct map_find_fn {
  storage_ref_type const& storage_ref;

  template <typename T>
  __device__ cuco::pair<key_type, mapped_type> operator()(column_device_view const& col, key_type i)
  {
    if constexpr (column_device_view::has_element_accessor<T>()) {
      using equality_fn_type    = equality_functor<T>;
      using hash_fn_type        = hash_functor<T>;
      using probing_scheme_type = cuco::linear_probing<cg_size, hash_fn_type>;

      // Instantiate hash and equality functors.
      auto hash_fn  = hash_fn_type{col};
      auto equal_fn = equality_fn_type{col};

      // Make a view of the hash map
      cuco::static_map_ref<key_type,
                           mapped_type,
                           SCOPE,
                           equality_fn_type,
                           probing_scheme_type,
                           storage_ref_type>
        hash_map_ref{cuco::empty_key{KEY_SENTINEL},
                     cuco::empty_value{VALUE_SENTINEL},
                     {equal_fn},
                     {hash_fn},
                     {},
                     storage_ref};

      // Create another map with find operator
      auto map_find_ref = hash_map_ref.with_operators(cuco::find);

      // Find the key = i
      auto found_slot = map_find_ref.find(i);

      // Check if we found the previously inserted key.
      cudf_assert(found_slot != map_find_ref.end() &&
                  "Unable to find value in map in dictionary index construction");

      // Return a pair of the found key and value.
      printf("Find=%d, Found slot: k=%d, v=%d\n", i, found_slot->first, found_slot->second);
      return {found_slot->first, found_slot->second};
    } else {
      CUDF_UNREACHABLE("Unsupported type to find in map");
    }
  }
};

template <int block_size>
CUDF_KERNEL void __launch_bounds__(block_size)
  populate_chunk_hash_maps_kernel(storage_type::window_type* map_storage,
                                  cudf::detail::device_2dspan<PageFragment const> frags)
{
  auto const col_idx = blockIdx.y;
  auto const block_x = blockIdx.x;
  auto const frag    = frags[col_idx][block_x];
  auto chunk         = frag.chunk;
  auto col           = chunk->col_desc;

  if (not chunk->use_dictionary) { return; }

  using block_reduce = hipcub::BlockReduce<size_type, block_size>;
  __shared__ typename block_reduce::TempStorage reduce_storage;

  [[maybe_unused]] auto const tile = cg::tiled_partition<cg_size>(cg::this_thread_block());
  auto const t                     = cg::this_thread_block().thread_rank();

  size_type start_row = frag.start_row;
  size_type end_row   = frag.start_row + frag.num_rows;

  // Find the bounds of values in leaf column to be inserted into the map for current chunk
  size_type const s_start_value_idx = row_to_value_idx(start_row, *col);
  size_type const end_value_idx     = row_to_value_idx(end_row, *col);

  column_device_view const& data_col = *col->leaf_column;
  storage_ref_type const storage_ref{chunk->dict_map_size, map_storage + chunk->dict_map_offset};

  __shared__ size_type total_num_dict_entries;
  thread_index_type val_idx = s_start_value_idx + t;

  while (val_idx - block_size < end_value_idx) {
    auto const is_valid =
      val_idx < end_value_idx and val_idx < data_col.size() and data_col.is_valid(val_idx);

    // insert element at val_idx to hash map and count successful insertions
    size_type is_unique      = 0;
    size_type uniq_elem_size = 0;
    if (is_valid) {
      auto const is_unique =
        type_dispatcher(data_col.type(), map_insert_fn{storage_ref}, data_col, val_idx);
      uniq_elem_size = [&]() -> size_type {
        if (not is_unique) { return 0; }
        switch (col->physical_type) {
          case Type::INT32: return 4;
          case Type::INT64: return 8;
          case Type::INT96: return 12;
          case Type::FLOAT: return 4;
          case Type::DOUBLE: return 8;
          case Type::BYTE_ARRAY: {
            auto const col_type = data_col.type().id();
            if (col_type == type_id::STRING) {
              // Strings are stored as 4 byte length + string bytes
              return 4 + data_col.element<string_view>(val_idx).size_bytes();
            } else if (col_type == type_id::LIST) {
              // Binary is stored as 4 byte length + bytes
              return 4 + get_element<statistics::byte_array_view>(data_col, val_idx).size_bytes();
            }
            CUDF_UNREACHABLE(
              "Byte array only supports string and list<byte> column types for dictionary "
              "encoding!");
          }
          case Type::FIXED_LEN_BYTE_ARRAY:
            if (data_col.type().id() == type_id::DECIMAL128) { return sizeof(__int128_t); }
            CUDF_UNREACHABLE(
              "Fixed length byte array only supports decimal 128 column types for dictionary "
              "encoding!");
          default: CUDF_UNREACHABLE("Unsupported type for dictionary encoding");
        }
      }();
    }
    auto num_unique = block_reduce(reduce_storage).Sum(is_unique);
    __syncthreads();
    auto uniq_data_size = block_reduce(reduce_storage).Sum(uniq_elem_size);
    if (t == 0) {
      total_num_dict_entries = atomicAdd(&chunk->num_dict_entries, num_unique);
      total_num_dict_entries += num_unique;
      atomicAdd(&chunk->uniq_data_size, uniq_data_size);
    }
    __syncthreads();

    // Check if the num unique values in chunk has already exceeded max dict size and early exit
    if (total_num_dict_entries > MAX_DICT_SIZE) { return; }

    val_idx += block_size;
  }  // while
}

template <int block_size>
CUDF_KERNEL void __launch_bounds__(block_size)
  collect_map_entries_kernel(storage_type::window_type* map_storage,
                             device_span<EncColumnChunk> chunks)
{
  auto& chunk = chunks[blockIdx.x];
  if (not chunk.use_dictionary) { return; }

  [[maybe_unused]] auto const tile = cg::tiled_partition<cg_size>(cg::this_thread_block());
  auto const t                     = cg::this_thread_block().thread_rank();

  __shared__ cuda::atomic<size_type, cuda::thread_scope_block> counter;
  using cuda::std::memory_order_relaxed;
  if (t == 0) { new (&counter) cuda::atomic<size_type, cuda::thread_scope_block>{0}; }
  __syncthreads();

  for (size_type i = 0; i < chunk.dict_map_size; i += block_size) {
    if (t + i < chunk.dict_map_size) {
      auto* slot     = map_storage + chunk.dict_map_offset + t + i;
      auto const key = slot->data()->first;
      if (key != KEY_SENTINEL) {
        auto loc = counter.fetch_add(1, memory_order_relaxed);
        cudf_assert(loc < MAX_DICT_SIZE && "Number of filled slots exceeds max dict size");
        // printf("Writing %d at loc: %d\n", key, loc);
        chunk.dict_data[loc] = key;
        // If sorting dict page ever becomes a hard requirement, enable the following statement and
        // add a dict sorting step before storing into the slot's second field.
        // chunk.dict_data_idx[loc] = t + i;
        slot->data()->second = loc;
      }
    }
  }
}

template <int block_size>
CUDF_KERNEL void __launch_bounds__(block_size)
  get_dictionary_indices_kernel(storage_type::window_type* map_storage,
                                cudf::detail::device_2dspan<PageFragment const> frags)
{
  auto const col_idx = blockIdx.y;
  auto const block_x = blockIdx.x;
  auto const frag    = frags[col_idx][block_x];
  auto chunk         = frag.chunk;
  auto const col     = chunk->col_desc;

  if (not chunk->use_dictionary) { return; }

  [[maybe_unused]] auto const tile = cg::tiled_partition<cg_size>(cg::this_thread_block());
  auto const t                     = cg::this_thread_block().thread_rank();

  size_type start_row = frag.start_row;
  size_type end_row   = frag.start_row + frag.num_rows;

  // Find the bounds of values in leaf column to be searched in the map for current chunk
  auto const s_start_value_idx  = row_to_value_idx(start_row, *col);
  auto const s_ck_start_val_idx = row_to_value_idx(chunk->start_row, *col);
  auto const end_value_idx      = row_to_value_idx(end_row, *col);

  column_device_view const& data_col = *col->leaf_column;
  storage_ref_type const storage_ref{chunk->dict_map_size, map_storage + chunk->dict_map_offset};

  thread_index_type val_idx = s_start_value_idx + t;
  while (val_idx < end_value_idx) {
    if (data_col.is_valid(val_idx)) {
      auto [found_key, found_value] =
        type_dispatcher(data_col.type(), map_find_fn{storage_ref}, data_col, val_idx);
      // No need for atomic as this is not going to be modified by any other thread
      chunk->dict_index[val_idx - s_ck_start_val_idx] = found_value;
    }

    val_idx += block_size;
  }
}

void populate_chunk_hash_maps(storage_type::window_type* map_storage,
                              cudf::detail::device_2dspan<PageFragment const> frags,
                              rmm::cuda_stream_view stream)
{
  dim3 const dim_grid(frags.size().second, frags.size().first);
  populate_chunk_hash_maps_kernel<DEFAULT_BLOCK_SIZE>
    <<<dim_grid, DEFAULT_BLOCK_SIZE, 0, stream.value()>>>(map_storage, frags);
}

void collect_map_entries(storage_type::window_type* map_storage,
                         device_span<EncColumnChunk> chunks,
                         rmm::cuda_stream_view stream)
{
  constexpr int block_size = 1024;
  collect_map_entries_kernel<block_size>
    <<<chunks.size(), block_size, 0, stream.value()>>>(map_storage, chunks);
}

void get_dictionary_indices(storage_type::window_type* map_storage,
                            cudf::detail::device_2dspan<PageFragment const> frags,
                            rmm::cuda_stream_view stream)
{
  dim3 const dim_grid(frags.size().second, frags.size().first);
  get_dictionary_indices_kernel<DEFAULT_BLOCK_SIZE>
    <<<dim_grid, DEFAULT_BLOCK_SIZE, 0, stream.value()>>>(map_storage, frags);
}
}  // namespace cudf::io::parquet::detail
