#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "io/comp/io_uncomp.hpp"
#include "io/json/legacy/read_json.hpp"
#include "io/json/nested_json.hpp"
#include "read_json.hpp"

#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/utilities/stream_pool.hpp>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/io/detail/json.hpp>
#include <cudf/utilities/error.hpp>

#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/iterator/constant_iterator.h>
#include <thrust/scatter.h>

#include <numeric>

namespace cudf::io::json::detail {

size_t sources_size(host_span<std::unique_ptr<datasource>> const sources,
                    size_t range_offset,
                    size_t range_size)
{
  return std::accumulate(sources.begin(), sources.end(), 0ul, [=](size_t sum, auto& source) {
    auto const size = source->size();
    // TODO take care of 0, 0, or *, 0 case.
    return sum +
           (range_size == 0 or range_offset + range_size > size ? size - range_offset : range_size);
  });
}

/**
 * @brief Read from array of data sources into RMM buffer
 *
 * @param sources Array of data sources
 * @param compression Compression format of source
 * @param range_offset Number of bytes to skip from source start
 * @param range_size Number of bytes to read from source
 * @param stream CUDA stream used for device memory operations and kernel launches
 */
rmm::device_uvector<char> ingest_raw_input(host_span<std::unique_ptr<datasource>> sources,
                                           compression_type compression,
                                           size_t range_offset,
                                           size_t range_size,
                                           rmm::cuda_stream_view stream)
{
  CUDF_FUNC_RANGE();
  // We append a line delimiter between two files to make sure the last line of file i and the first
  // line of file i+1 don't end up on the same JSON line, if file i does not already end with a line
  // delimiter.
  auto constexpr num_delimiter_chars = 1;
  auto const num_extra_delimiters    = num_delimiter_chars * (sources.size() - 1);

  // Iterate through the user defined sources and read the contents into the local buffer
  auto const total_source_size =
    sources_size(sources, range_offset, range_size) + num_extra_delimiters;

  if (compression == compression_type::NONE) {
    std::vector<size_type> delimiter_map{};
    delimiter_map.reserve(sources.size());
    auto d_buffer     = rmm::device_uvector<char>(total_source_size, stream);
    size_t bytes_read = 0;
    std::vector<std::unique_ptr<datasource::buffer>> h_buffers;
    for (auto const& source : sources) {
      if (!source->is_empty()) {
        auto data_size   = (range_size != 0) ? range_size : source->size();
        auto destination = reinterpret_cast<uint8_t*>(d_buffer.data()) + bytes_read;
        if (source->is_device_read_preferred(data_size)) {
          bytes_read += source->device_read(range_offset, data_size, destination, stream);
        } else {
          h_buffers.emplace_back(source->host_read(range_offset, data_size));
          auto const& h_buffer = h_buffers.back();
          CUDF_CUDA_TRY(hipMemcpyAsync(
            destination, h_buffer->data(), h_buffer->size(), hipMemcpyDefault, stream.value()));
          bytes_read += h_buffer->size();
        }
        delimiter_map.push_back(bytes_read);
        bytes_read += num_delimiter_chars;
      }
    }

    // If this is a multi-file source, we scatter the JSON line delimiters between files
    if (sources.size() > 1) {
      static_assert(num_delimiter_chars == 1,
                    "Currently only single-character delimiters are supported");
      auto const delimiter_source = thrust::make_constant_iterator('\n');
      auto const d_delimiter_map  = cudf::detail::make_device_uvector_async(
        host_span<size_type const>{delimiter_map.data(), delimiter_map.size() - 1},
        stream,
        rmm::mr::get_current_device_resource());
      thrust::scatter(rmm::exec_policy_nosync(stream),
                      delimiter_source,
                      delimiter_source + d_delimiter_map.size(),
                      d_delimiter_map.data(),
                      d_buffer.data());
    }

    stream.synchronize();
    return d_buffer;

  } else {
    auto buffer = std::vector<uint8_t>(total_source_size);
    // Single read because only a single compressed source is supported
    // Reading to host because decompression of a single block is much faster on the CPU
    sources[0]->host_read(range_offset, total_source_size, buffer.data());
    auto const uncomp_data = decompress(compression, buffer);
    return cudf::detail::make_device_uvector_sync(
      host_span<char const>{reinterpret_cast<char const*>(uncomp_data.data()), uncomp_data.size()},
      stream,
      rmm::mr::get_current_device_resource());
  }
}

size_type find_first_delimiter_in_chunk(host_span<std::unique_ptr<cudf::io::datasource>> sources,
                                        json_reader_options const& reader_opts,
                                        char const delimiter,
                                        rmm::cuda_stream_view stream)
{
  auto const buffer = ingest_raw_input(sources,
                                       reader_opts.get_compression(),
                                       reader_opts.get_byte_range_offset(),
                                       reader_opts.get_byte_range_size(),
                                       stream);
  return find_first_delimiter(buffer, delimiter, stream);
}

bool should_load_whole_source(json_reader_options const& opts, size_t source_size)
{
  auto const range_offset = opts.get_byte_range_offset();
  auto const range_size   = opts.get_byte_range_size();
  return range_offset == 0 and (range_size == 0 or range_size >= source_size);
}

/**
 * @brief Get the byte range between record starts and ends starting from the given range.
 *
 * if get_byte_range_offset == 0, then we can skip the first delimiter search
 * if get_byte_range_offset != 0, then we need to search for the first delimiter in given range.
 * if not found, skip this chunk, if found, then search for first delimiter in next range until we
 * find a delimiter. Use this as actual range for parsing.
 *
 * @param sources Data sources to read from
 * @param reader_opts JSON reader options with range offset and range size
 * @param stream CUDA stream used for device memory operations and kernel launches
 * @return Byte range for parsing
 */
auto get_record_range_raw_input(host_span<std::unique_ptr<datasource>> sources,
                                json_reader_options const& reader_opts,
                                rmm::cuda_stream_view stream)
{
  size_t const total_source_size = sources_size(sources, 0, 0);
  rmm::device_uvector<char> merged(0, stream);

  rmm::device_uvector<char> cur_chunk_buf = ingest_raw_input(sources,
                                                             reader_opts.get_compression(),
                                                             reader_opts.get_byte_range_offset(),
                                                             reader_opts.get_byte_range_size(),
                                                             stream);
  auto first_delim_pos                    = reader_opts.get_byte_range_offset() == 0
                                              ? 0
                                              : find_first_delimiter(cur_chunk_buf, '\n', stream);
  if (first_delim_pos == -1) {
    return rmm::device_uvector<char>{0, stream};
  } else if (!should_load_whole_source(reader_opts, sources[0]->size()) &&
             cur_chunk_buf.back_element(stream) != '\n' &&
             reader_opts.get_byte_range_offset() + reader_opts.get_byte_range_size() <
               total_source_size) {
    // Find next delimiter
    /*
     * NOTE: heuristic for choosing subchunk size: geometric mean of minimum subchunk size (set to
     * 10kb) and the byte range size
     */
    std::int64_t next_delim_pos = -1;
    constexpr int num_subchunks = 10;  // per byte_range_size
    auto geometric_mean         = [](double a, double b) { return std::pow(a * b, 0.5); };
    size_t size_per_subchunk =
      geometric_mean(reader_opts.get_byte_range_size() / num_subchunks, 10000);
    size_t next_subchunk_start =
      reader_opts.get_byte_range_offset() + reader_opts.get_byte_range_size();
    std::vector<rmm::device_uvector<char>> subchunk_buffers;

    while (next_subchunk_start < total_source_size && next_delim_pos == -1) {
      subchunk_buffers.emplace_back(ingest_raw_input(
        sources, reader_opts.get_compression(), next_subchunk_start, size_per_subchunk, stream));
      next_delim_pos = find_first_delimiter(subchunk_buffers.back(), '\n', stream);
      if (next_delim_pos == -1) { next_subchunk_start += size_per_subchunk; }
    }
    if (next_delim_pos == -1)
      next_delim_pos = total_source_size - (next_subchunk_start - size_per_subchunk);

    merged.resize(
      cur_chunk_buf.size() + ((subchunk_buffers.size() - 1) * size_per_subchunk) + next_delim_pos,
      stream);
    size_t offset = cur_chunk_buf.size() - first_delim_pos;
    if (subchunk_buffers.size() >= 3) {
      std::vector<rmm::cuda_stream_view> copy_streams =
        cudf::detail::fork_streams(stream, subchunk_buffers.size() - 1);
      for (size_t i = 0; i < subchunk_buffers.size() - 1; i++) {
        CUDF_CUDA_TRY(hipMemcpyAsync(merged.data() + offset,
                                      subchunk_buffers[i].data(),
                                      size_per_subchunk,
                                      hipMemcpyDeviceToDevice,
                                      copy_streams[i]));
        offset += size_per_subchunk;
      }
      cudf::detail::join_streams(copy_streams, stream);
    } else if (subchunk_buffers.size() == 2) {
      CUDF_CUDA_TRY(hipMemcpyAsync(merged.data() + offset,
                                    subchunk_buffers[0].data(),
                                    size_per_subchunk,
                                    hipMemcpyDeviceToDevice,
                                    stream));
      offset += size_per_subchunk;
    }
    CUDF_CUDA_TRY(hipMemcpyAsync(merged.data() + offset,
                                  subchunk_buffers.back().data(),
                                  next_delim_pos,
                                  hipMemcpyDeviceToDevice,
                                  stream));
  } else {
    merged.resize(cur_chunk_buf.size() - first_delim_pos, stream);
  }

  CUDF_CUDA_TRY(hipMemcpyAsync(merged.data(),
                                cur_chunk_buf.data() + first_delim_pos,
                                cur_chunk_buf.size() - first_delim_pos,
                                hipMemcpyDeviceToDevice,
                                stream));
  stream.synchronize();
  return merged;
}

table_with_metadata read_json(host_span<std::unique_ptr<datasource>> sources,
                              json_reader_options const& reader_opts,
                              rmm::cuda_stream_view stream,
                              rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();

  if (reader_opts.is_enabled_legacy()) {
    return legacy::read_json(sources, reader_opts, stream, mr);
  }

  if (reader_opts.get_byte_range_offset() != 0 or reader_opts.get_byte_range_size() != 0) {
    CUDF_EXPECTS(reader_opts.is_enabled_lines(),
                 "Specifying a byte range is supported only for JSON Lines");
    CUDF_EXPECTS(sources.size() == 1,
                 "Specifying a byte range is supported only for a single source");
  }

  if (sources.size() > 1) {
    CUDF_EXPECTS(reader_opts.get_compression() == compression_type::NONE,
                 "Multiple compressed inputs are not supported");
    CUDF_EXPECTS(reader_opts.is_enabled_lines(),
                 "Multiple inputs are supported only for JSON Lines format");
  }

  auto buffer = get_record_range_raw_input(sources, reader_opts, stream);

  // If input JSON buffer has single quotes and option to normalize single quotes is enabled,
  // invoke pre-processing FST
  if (reader_opts.is_enabled_normalize_single_quotes()) {
    buffer =
      normalize_single_quotes(std::move(buffer), stream, rmm::mr::get_current_device_resource());
  }

  // If input JSON buffer has unquoted spaces and tabs and option to normalize whitespaces is
  // enabled, invoke pre-processing FST
  if (reader_opts.is_enabled_normalize_whitespace()) {
    buffer =
      normalize_whitespace(std::move(buffer), stream, rmm::mr::get_current_device_resource());
  }

  return device_parse_nested_json(buffer, reader_opts, stream, mr);
  // For debug purposes, use host_parse_nested_json()
}

}  // namespace cudf::io::json::detail
