#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

// #define PRINT_DEBUG

// TODO: remove
#include <cudf_test/debug_utilities.hpp>

#include <cudf/concatenate.hpp>
//
//
//
#include "io/comp/gpuinflate.hpp"
#include "io/comp/nvcomp_adapter.hpp"
#include "io/orc/reader_impl.hpp"
#include "io/orc/reader_impl_chunking.hpp"
#include "io/orc/reader_impl_helpers.hpp"
#include "io/utilities/config_utils.hpp"

#include <cudf/detail/copy.hpp>
#include <cudf/detail/timezone.hpp>
#include <cudf/detail/transform.hpp>
#include <cudf/detail/utilities/integer_utils.hpp>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/table/table.hpp>
#include <cudf/utilities/bit.hpp>
#include <cudf/utilities/error.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_buffer.hpp>
#include <rmm/device_scalar.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <cuda/functional>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/pair.h>
#include <thrust/scan.h>
#include <thrust/transform.h>

#include <algorithm>
#include <iterator>

namespace cudf::io::orc::detail {

namespace {

// TODO: update
// TODO: compute num stripes from chunks
/**
 * @brief Decompresses the stripe data, at stream granularity.
 *
 * @param decompressor Block decompressor
 * @param stripe_data List of source stripe column data
 * @param stream_info List of stream to column mappings
 * @param chunks Vector of list of column chunk descriptors
 * @param row_groups Vector of list of row index descriptors
 * @param num_stripes Number of stripes making up column chunks
 * @param row_index_stride Distance between each row index
 * @param use_base_stride Whether to use base stride obtained from meta or use the computed value
 * @param stream CUDA stream used for device memory operations and kernel launches
 * @return Device buffer to decompressed page data
 */
rmm::device_buffer decompress_stripe_data(
  chunk const& load_stripe_chunk,
  chunk const& stripe_chunk,
  stream_id_map<stripe_level_comp_info> const& compinfo_map,
  OrcDecompressor const& decompressor,
  host_span<rmm::device_buffer const> stripe_data,
  host_span<orc_stream_info const> stream_info,
  cudf::detail::hostdevice_2dvector<gpu::ColumnDesc>& chunks,
  cudf::detail::hostdevice_2dvector<gpu::RowGroup>& row_groups,
  size_type num_stripes,
  size_type row_index_stride,
  bool use_base_stride,
  rmm::cuda_stream_view stream)
{
  // Count the exact number of compressed blocks
  std::size_t num_compressed_blocks   = 0;
  std::size_t num_uncompressed_blocks = 0;
  std::size_t total_decomp_size       = 0;

  // printf("decompress #stripe: %d, ")

  // TODO: use lvl_stripe_stream_chunks
  std::size_t count{0};
  for (auto const& info : stream_info) {
    if (info.id.stripe_idx < stripe_chunk.start_idx ||
        info.id.stripe_idx >= stripe_chunk.start_idx + stripe_chunk.count) {
      continue;
    }
    count++;
  }

  cudf::detail::hostdevice_vector<gpu::CompressedStreamInfo> compinfo(0, count, stream);

  for (auto const& info : stream_info) {
    if (info.id.stripe_idx < stripe_chunk.start_idx ||
        info.id.stripe_idx >= stripe_chunk.start_idx + stripe_chunk.count) {
      continue;
    }

#ifdef PRINT_DEBUG
    printf("collec stream  again [%d, %d, %d, %d]: dst = %lu,  length = %lu\n",
           (int)info.id.stripe_idx,
           (int)info.id.level,
           (int)info.id.orc_cold_idx,
           (int)info.id.kind,
           info.dst_pos,
           info.length);
    fflush(stdout);
#endif

    compinfo.push_back(gpu::CompressedStreamInfo(
      static_cast<uint8_t const*>(
        stripe_data[info.id.stripe_idx - load_stripe_chunk.start_idx].data()) +
        info.dst_pos,
      info.length));

    //    printf("line %d\n", __LINE__);
    //    fflush(stdout);
    auto const& cached_comp_info = compinfo_map.at(
      stream_id_info{info.id.stripe_idx, info.id.level, info.id.orc_col_idx, info.id.kind});
    //    printf("line %d\n", __LINE__);
    //    fflush(stdout);
    // auto const& cached_comp_info =
    //   compinfo_map[stream_id_info{info.id.stripe_idx, info.id.level, info.id.orc_cold_idx,
    //   info.id.kind}];
    auto& stream_comp_info                   = compinfo.back();
    stream_comp_info.num_compressed_blocks   = cached_comp_info.num_compressed_blocks;
    stream_comp_info.num_uncompressed_blocks = cached_comp_info.num_uncompressed_blocks;
    stream_comp_info.max_uncompressed_size   = cached_comp_info.total_decomp_size;

    num_compressed_blocks += cached_comp_info.num_compressed_blocks;
    num_uncompressed_blocks += cached_comp_info.num_uncompressed_blocks;
    total_decomp_size += cached_comp_info.total_decomp_size;
  }

  CUDF_EXPECTS(
    not((num_uncompressed_blocks + num_compressed_blocks > 0) and (total_decomp_size == 0)),
    "Inconsistent info on compression blocks");

#ifdef XXX
  std::size_t old_num_compressed_blocks   = num_compressed_blocks;
  std::size_t old_num_uncompressed_blocks = num_uncompressed_blocks;
  std::size_t old_total_decomp_size       = total_decomp_size;

  num_compressed_blocks   = 0;
  num_uncompressed_blocks = 0;
  total_decomp_size       = 0;
  for (std::size_t i = 0; i < compinfo.size(); ++i) {
    num_compressed_blocks += compinfo[i].num_compressed_blocks;
    num_uncompressed_blocks += compinfo[i].num_uncompressed_blocks;
    total_decomp_size += compinfo[i].max_uncompressed_size;

    auto const& info = stream_info[i];
    printf("compute info [%d, %d, %d, %d]:  %lu | %lu | %lu\n",
           (int)info.id.stripe_idx,
           (int)info.id.level,
           (int)info.id.orc_cold_idx,
           (int)info.id.kind,
           (size_t)compinfo[i].num_compressed_blocks,
           (size_t)compinfo[i].num_uncompressed_blocks,
           compinfo[i].max_uncompressed_size);
    fflush(stdout);
  }

  if (old_num_compressed_blocks != num_compressed_blocks ||
      old_num_uncompressed_blocks != num_uncompressed_blocks ||
      old_total_decomp_size != total_decomp_size) {
    printf("invalid: %d - %d, %d - %d, %d - %d\n",
           (int)old_num_compressed_blocks,
           (int)num_compressed_blocks,
           (int)old_num_uncompressed_blocks,
           (int)num_uncompressed_blocks,
           (int)old_total_decomp_size,
           (int)total_decomp_size

    );
  }
#endif

  // Buffer needs to be padded.
  // Required by `gpuDecodeOrcColumnData`.
  rmm::device_buffer decomp_data(
    cudf::util::round_up_safe(total_decomp_size, BUFFER_PADDING_MULTIPLE), stream);
  if (decomp_data.is_empty()) { return decomp_data; }

  rmm::device_uvector<device_span<uint8_t const>> inflate_in(
    num_compressed_blocks + num_uncompressed_blocks, stream);
  rmm::device_uvector<device_span<uint8_t>> inflate_out(
    num_compressed_blocks + num_uncompressed_blocks, stream);
  rmm::device_uvector<compression_result> inflate_res(num_compressed_blocks, stream);
  thrust::fill(rmm::exec_policy(stream),
               inflate_res.begin(),
               inflate_res.end(),
               compression_result{0, compression_status::FAILURE});

  // Parse again to populate the decompression input/output buffers
  std::size_t decomp_offset      = 0;
  uint32_t max_uncomp_block_size = 0;
  uint32_t start_pos             = 0;
  auto start_pos_uncomp          = (uint32_t)num_compressed_blocks;
  for (std::size_t i = 0; i < compinfo.size(); ++i) {
    auto dst_base                 = static_cast<uint8_t*>(decomp_data.data());
    compinfo[i].uncompressed_data = dst_base + decomp_offset;
    compinfo[i].dec_in_ctl        = inflate_in.data() + start_pos;
    compinfo[i].dec_out_ctl       = inflate_out.data() + start_pos;
    compinfo[i].dec_res      = {inflate_res.data() + start_pos, compinfo[i].num_compressed_blocks};
    compinfo[i].copy_in_ctl  = inflate_in.data() + start_pos_uncomp;
    compinfo[i].copy_out_ctl = inflate_out.data() + start_pos_uncomp;

    //    stream_info[i].dst_pos = decomp_offset;
    decomp_offset += compinfo[i].max_uncompressed_size;
    start_pos += compinfo[i].num_compressed_blocks;
    start_pos_uncomp += compinfo[i].num_uncompressed_blocks;
    max_uncomp_block_size =
      std::max(max_uncomp_block_size, compinfo[i].max_uncompressed_block_size);
  }
  compinfo.host_to_device_async(stream);
  gpu::ParseCompressedStripeData(compinfo.device_ptr(),
                                 compinfo.size(),
                                 decompressor.GetBlockSize(),
                                 decompressor.GetLog2MaxCompressionRatio(),
                                 stream);

  // Value for checking whether we decompress successfully.
  // It doesn't need to be atomic as there is no race condition: we only write `true` if needed.
  cudf::detail::hostdevice_vector<bool> any_block_failure(1, stream);
  any_block_failure[0] = false;
  any_block_failure.host_to_device_async(stream);

  // Dispatch batches of blocks to decompress
  if (num_compressed_blocks > 0) {
    device_span<device_span<uint8_t const>> inflate_in_view{inflate_in.data(),
                                                            num_compressed_blocks};
    device_span<device_span<uint8_t>> inflate_out_view{inflate_out.data(), num_compressed_blocks};
    switch (decompressor.compression()) {
      case compression_type::ZLIB:
        if (nvcomp::is_decompression_disabled(nvcomp::compression_type::DEFLATE)) {
          gpuinflate(
            inflate_in_view, inflate_out_view, inflate_res, gzip_header_included::NO, stream);
        } else {
          nvcomp::batched_decompress(nvcomp::compression_type::DEFLATE,
                                     inflate_in_view,
                                     inflate_out_view,
                                     inflate_res,
                                     max_uncomp_block_size,
                                     total_decomp_size,
                                     stream);
        }
        break;
      case compression_type::SNAPPY:
        if (nvcomp::is_decompression_disabled(nvcomp::compression_type::SNAPPY)) {
          gpu_unsnap(inflate_in_view, inflate_out_view, inflate_res, stream);
        } else {
          nvcomp::batched_decompress(nvcomp::compression_type::SNAPPY,
                                     inflate_in_view,
                                     inflate_out_view,
                                     inflate_res,
                                     max_uncomp_block_size,
                                     total_decomp_size,
                                     stream);
        }
        break;
      case compression_type::ZSTD:
        if (auto const reason = nvcomp::is_decompression_disabled(nvcomp::compression_type::ZSTD);
            reason) {
          CUDF_FAIL("Decompression error: " + reason.value());
        }
        nvcomp::batched_decompress(nvcomp::compression_type::ZSTD,
                                   inflate_in_view,
                                   inflate_out_view,
                                   inflate_res,
                                   max_uncomp_block_size,
                                   total_decomp_size,
                                   stream);
        break;
      case compression_type::LZ4:
        if (auto const reason = nvcomp::is_decompression_disabled(nvcomp::compression_type::LZ4);
            reason) {
          CUDF_FAIL("Decompression error: " + reason.value());
        }
        nvcomp::batched_decompress(nvcomp::compression_type::LZ4,
                                   inflate_in_view,
                                   inflate_out_view,
                                   inflate_res,
                                   max_uncomp_block_size,
                                   total_decomp_size,
                                   stream);
        break;
      default: CUDF_FAIL("Unexpected decompression dispatch"); break;
    }

    // TODO: proclam return type

    // Check if any block has been failed to decompress.
    // Not using `thrust::any` or `thrust::count_if` to defer stream sync.
    thrust::for_each(
      rmm::exec_policy(stream),
      thrust::make_counting_iterator(std::size_t{0}),
      thrust::make_counting_iterator(inflate_res.size()),
      [results           = inflate_res.begin(),
       any_block_failure = any_block_failure.device_ptr()] __device__(auto const idx) {
        if (results[idx].status != compression_status::SUCCESS) { *any_block_failure = true; }
      });
  }

  if (num_uncompressed_blocks > 0) {
    device_span<device_span<uint8_t const>> copy_in_view{inflate_in.data() + num_compressed_blocks,
                                                         num_uncompressed_blocks};
    device_span<device_span<uint8_t>> copy_out_view{inflate_out.data() + num_compressed_blocks,
                                                    num_uncompressed_blocks};
    gpu_copy_uncompressed_blocks(copy_in_view, copy_out_view, stream);
  }

  // Copy without stream sync, thus need to wait for stream sync below to access.
  any_block_failure.device_to_host_async(stream);

  gpu::PostDecompressionReassemble(compinfo.device_ptr(), compinfo.size(), stream);
  compinfo.device_to_host_sync(stream);  // This also sync stream for `any_block_failure`.

  // We can check on host after stream synchronize
  CUDF_EXPECTS(not any_block_failure[0], "Error during decompression");

  auto const num_columns = static_cast<size_type>(chunks.size().second);

  // Update the stream information with the updated uncompressed info
  // TBD: We could update the value from the information we already
  // have in stream_info[], but using the gpu results also updates
  // max_uncompressed_size to the actual uncompressed size, or zero if
  // decompression failed.
  for (size_type i = 0; i < num_stripes; ++i) {
    for (size_type j = 0; j < num_columns; ++j) {
      auto& chunk = chunks[i][j];
      for (int k = 0; k < gpu::CI_NUM_STREAMS; ++k) {
        if (chunk.strm_len[k] > 0 && chunk.strm_id[k] < compinfo.size()) {
          chunk.streams[k]  = compinfo[chunk.strm_id[k]].uncompressed_data;
          chunk.strm_len[k] = compinfo[chunk.strm_id[k]].max_uncompressed_size;
        }
      }
    }
  }

  if (row_groups.size().first) {
    chunks.host_to_device_async(stream);
    row_groups.host_to_device_async(stream);
    gpu::ParseRowGroupIndex(row_groups.base_device_ptr(),
                            compinfo.device_ptr(),
                            chunks.base_device_ptr(),
                            num_columns,
                            num_stripes,
                            row_index_stride,
                            use_base_stride,
                            stream);
  }

  return decomp_data;
}

/**
 * @brief Updates null mask of columns whose parent is a struct column.
 *
 * If struct column has null element, that row would be skipped while writing child column in ORC,
 * so we need to insert the missing null elements in child column. There is another behavior from
 * pyspark, where if the child column doesn't have any null elements, it will not have present
 * stream, so in that case parent null mask need to be copied to child column.
 *
 * @param chunks Vector of list of column chunk descriptors
 * @param out_buffers Output columns' device buffers
 * @param stream CUDA stream used for device memory operations and kernel launches.
 * @param mr Device memory resource to use for device memory allocation
 */
void update_null_mask(cudf::detail::hostdevice_2dvector<gpu::ColumnDesc>& chunks,
                      host_span<column_buffer> out_buffers,
                      rmm::cuda_stream_view stream,
                      rmm::mr::device_memory_resource* mr)
{
  auto const num_stripes = chunks.size().first;
  auto const num_columns = chunks.size().second;
  bool is_mask_updated   = false;

  for (std::size_t col_idx = 0; col_idx < num_columns; ++col_idx) {
    if (chunks[0][col_idx].parent_validity_info.valid_map_base != nullptr) {
      if (not is_mask_updated) {
        chunks.device_to_host_sync(stream);
        is_mask_updated = true;
      }

      auto parent_valid_map_base = chunks[0][col_idx].parent_validity_info.valid_map_base;
      auto child_valid_map_base  = out_buffers[col_idx].null_mask();
      auto child_mask_len =
        chunks[0][col_idx].column_num_rows - chunks[0][col_idx].parent_validity_info.null_count;
      auto parent_mask_len = chunks[0][col_idx].column_num_rows;

      if (child_valid_map_base != nullptr) {
        rmm::device_uvector<uint32_t> dst_idx(child_mask_len, stream);
        // Copy indexes at which the parent has valid value.
        thrust::copy_if(rmm::exec_policy(stream),
                        thrust::make_counting_iterator(0),
                        thrust::make_counting_iterator(0) + parent_mask_len,
                        dst_idx.begin(),
                        [parent_valid_map_base] __device__(auto idx) {
                          return bit_is_set(parent_valid_map_base, idx);
                        });

        auto merged_null_mask = cudf::detail::create_null_mask(
          parent_mask_len, mask_state::ALL_NULL, rmm::cuda_stream_view(stream), mr);
        auto merged_mask      = static_cast<bitmask_type*>(merged_null_mask.data());
        uint32_t* dst_idx_ptr = dst_idx.data();
        // Copy child valid bits from child column to valid indexes, this will merge both child
        // and parent null masks
        thrust::for_each(rmm::exec_policy(stream),
                         thrust::make_counting_iterator(0),
                         thrust::make_counting_iterator(0) + dst_idx.size(),
                         [child_valid_map_base, dst_idx_ptr, merged_mask] __device__(auto idx) {
                           if (bit_is_set(child_valid_map_base, idx)) {
                             cudf::set_bit(merged_mask, dst_idx_ptr[idx]);
                           };
                         });

        out_buffers[col_idx].set_null_mask(std::move(merged_null_mask));

      } else {
        // Since child column doesn't have a mask, copy parent null mask
        auto mask_size = bitmask_allocation_size_bytes(parent_mask_len);
        out_buffers[col_idx].set_null_mask(
          rmm::device_buffer(static_cast<void*>(parent_valid_map_base), mask_size, stream, mr));
      }
    }
  }

  if (is_mask_updated) {
    // Update chunks with pointers to column data which might have been changed.
    for (std::size_t stripe_idx = 0; stripe_idx < num_stripes; ++stripe_idx) {
      for (std::size_t col_idx = 0; col_idx < num_columns; ++col_idx) {
        auto& chunk          = chunks[stripe_idx][col_idx];
        chunk.valid_map_base = out_buffers[col_idx].null_mask();
      }
    }
    chunks.host_to_device_sync(stream);
  }
}

/**
 * @brief Converts the stripe column data and outputs to columns.
 *
 * @param num_dicts Number of dictionary entries required
 * @param skip_rows Number of rows to offset from start
 * @param row_index_stride Distance between each row index
 * @param level Current nesting level being processed
 * @param tz_table Local time to UTC conversion table
 * @param chunks Vector of list of column chunk descriptors
 * @param row_groups Vector of list of row index descriptors
 * @param out_buffers Output columns' device buffers
 * @param stream CUDA stream used for device memory operations and kernel launches
 * @param mr Device memory resource to use for device memory allocation
 */
void decode_stream_data(std::size_t num_dicts,
                        int64_t skip_rows,
                        size_type row_index_stride,
                        std::size_t level,
                        table_view const& tz_table,
                        cudf::detail::hostdevice_2dvector<gpu::ColumnDesc>& chunks,
                        cudf::detail::device_2dspan<gpu::RowGroup> row_groups,
                        std::vector<column_buffer>& out_buffers,
                        rmm::cuda_stream_view stream,
                        rmm::mr::device_memory_resource* mr)
{
  auto const num_stripes = chunks.size().first;
  auto const num_columns = chunks.size().second;
  printf("decode %d stripess \n", (int)num_stripes);

  thrust::counting_iterator<int> col_idx_it(0);
  thrust::counting_iterator<int> stripe_idx_it(0);

  // Update chunks with pointers to column data
  std::for_each(stripe_idx_it, stripe_idx_it + num_stripes, [&](auto stripe_idx) {
    std::for_each(col_idx_it, col_idx_it + num_columns, [&](auto col_idx) {
      auto& chunk            = chunks[stripe_idx][col_idx];
      chunk.column_data_base = out_buffers[col_idx].data();
      chunk.valid_map_base   = out_buffers[col_idx].null_mask();
    });
  });

  // Allocate global dictionary for deserializing
  rmm::device_uvector<gpu::DictionaryEntry> global_dict(num_dicts, stream);

  chunks.host_to_device_sync(stream);
  gpu::DecodeNullsAndStringDictionaries(
    chunks.base_device_ptr(), global_dict.data(), num_columns, num_stripes, skip_rows, stream);

  if (level > 0) {
    printf("update_null_mask\n");
    // Update nullmasks for children if parent was a struct and had null mask
    update_null_mask(chunks, out_buffers, stream, mr);
  }

  auto const tz_table_dptr = table_device_view::create(tz_table, stream);
  rmm::device_scalar<size_type> error_count(0, stream);
  // Update the null map for child columns

  // printf(
  //   "num col: %d, num stripe: %d, skip row: %d, row_groups size: %d, row index stride: %d, "
  //   "level: "
  //   "%d\n",
  //   (int)num_columns,
  //   (int)num_stripes,
  //   (int)skip_rows,
  //   (int)row_groups.size().first,
  //   (int)row_index_stride,
  //   (int)level
  // );

  gpu::DecodeOrcColumnData(chunks.base_device_ptr(),
                           global_dict.data(),
                           row_groups,
                           num_columns,
                           num_stripes,
                           skip_rows,
                           *tz_table_dptr,
                           row_groups.size().first,
                           row_index_stride,
                           level,
                           error_count.data(),
                           stream);
  chunks.device_to_host_async(stream);
  // `value` synchronizes
  auto const num_errors = error_count.value(stream);
  CUDF_EXPECTS(num_errors == 0, "ORC data decode failed");

  std::for_each(col_idx_it + 0, col_idx_it + num_columns, [&](auto col_idx) {
    out_buffers[col_idx].null_count() =
      std::accumulate(stripe_idx_it + 0,
                      stripe_idx_it + num_stripes,
                      0,
                      [&](auto null_count, auto const stripe_idx) {
                        // printf(
                        //   "null count: %d => %d\n", (int)stripe_idx,
                        //   (int)chunks[stripe_idx][col_idx].null_count);
                        // printf("num child rows: %d \n",
                        // (int)chunks[stripe_idx][col_idx].num_child_rows);

                        return null_count + chunks[stripe_idx][col_idx].null_count;
                      });
  });
}

/**
 * @brief Compute the per-stripe prefix sum of null count, for each struct column in the current
 * layer.
 */
void scan_null_counts(cudf::detail::hostdevice_2dvector<gpu::ColumnDesc> const& chunks,
                      cudf::host_span<rmm::device_uvector<uint32_t>> prefix_sums,
                      rmm::cuda_stream_view stream)
{
  auto const num_stripes = chunks.size().first;
  if (num_stripes == 0) return;

  auto const num_columns = chunks.size().second;
  std::vector<thrust::pair<size_type, cudf::device_span<uint32_t>>> prefix_sums_to_update;
  for (auto col_idx = 0ul; col_idx < num_columns; ++col_idx) {
    // Null counts sums are only needed for children of struct columns
    if (chunks[0][col_idx].type_kind == STRUCT) {
      prefix_sums_to_update.emplace_back(col_idx, prefix_sums[col_idx]);
    }
  }
  auto const d_prefix_sums_to_update = cudf::detail::make_device_uvector_async(
    prefix_sums_to_update, stream, rmm::mr::get_current_device_resource());

  thrust::for_each(rmm::exec_policy(stream),
                   d_prefix_sums_to_update.begin(),
                   d_prefix_sums_to_update.end(),
                   [chunks = cudf::detail::device_2dspan<gpu::ColumnDesc const>{chunks}] __device__(
                     auto const& idx_psums) {
                     auto const col_idx = idx_psums.first;
                     auto const psums   = idx_psums.second;

                     thrust::transform(
                       thrust::seq,
                       thrust::make_counting_iterator(0),
                       thrust::make_counting_iterator(0) + psums.size(),
                       psums.begin(),
                       [&](auto stripe_idx) { return chunks[stripe_idx][col_idx].null_count; });

                     thrust::inclusive_scan(thrust::seq, psums.begin(), psums.end(), psums.begin());
                   });
  // `prefix_sums_to_update` goes out of scope, copy has to be done before we return
  stream.synchronize();
}

// TODO: this is called for each chunk of stripes.
/**
 * @brief Aggregate child metadata from parent column chunks.
 */
void aggregate_child_meta(std::size_t stripe_start,
                          std::size_t level,
                          cudf::io::orc::detail::column_hierarchy const& selected_columns,
                          cudf::detail::host_2dspan<gpu::ColumnDesc> chunks,
                          cudf::detail::host_2dspan<gpu::RowGroup> row_groups,
                          host_span<orc_column_meta const> nested_cols,
                          host_span<column_buffer> out_buffers,
                          reader_column_meta& col_meta)
{
  auto const num_of_stripes         = chunks.size().first;
  auto const num_of_rowgroups       = row_groups.size().first;
  auto const num_child_cols         = selected_columns.levels[level + 1].size();
  auto const number_of_child_chunks = num_child_cols * num_of_stripes;
  auto& num_child_rows              = col_meta.num_child_rows;
  auto& parent_column_data          = col_meta.parent_column_data;

  // Reset the meta to store child column details.
  num_child_rows.resize(selected_columns.levels[level + 1].size());
  std::fill(num_child_rows.begin(), num_child_rows.end(), 0);
  parent_column_data.resize(number_of_child_chunks);
  col_meta.parent_column_index.resize(number_of_child_chunks);
  col_meta.child_start_row.resize(number_of_child_chunks);
  col_meta.num_child_rows_per_stripe.resize(number_of_child_chunks);
  col_meta.rwgrp_meta.resize(num_of_rowgroups * num_child_cols);

  auto child_start_row = cudf::detail::host_2dspan<int64_t>(
    col_meta.child_start_row.data(), num_of_stripes, num_child_cols);
  auto num_child_rows_per_stripe = cudf::detail::host_2dspan<int64_t>(
    col_meta.num_child_rows_per_stripe.data(), num_of_stripes, num_child_cols);
  auto rwgrp_meta = cudf::detail::host_2dspan<reader_column_meta::row_group_meta>(
    col_meta.rwgrp_meta.data(), num_of_rowgroups, num_child_cols);

  int index = 0;  // number of child column processed

  printf("\n\n");
  // For each parent column, update its child column meta for each stripe.
  std::for_each(nested_cols.begin(), nested_cols.end(), [&](auto const p_col) {
    // printf("p_col.id: %d\n", (int)p_col.id);

    auto const parent_col_idx = col_meta.orc_col_map[level][p_col.id];
    // printf("   level: %d, parent_col_idx: %d\n", (int)level, (int)parent_col_idx);

    int64_t start_row         = 0;
    auto processed_row_groups = 0;

    for (std::size_t stripe_id = 0; stripe_id < num_of_stripes; stripe_id++) {
      // Aggregate num_rows and start_row from processed parent columns per row groups
      if (num_of_rowgroups) {
        // printf("   num_of_rowgroups: %d\n", (int)num_of_rowgroups);

        auto stripe_num_row_groups = chunks[stripe_id][parent_col_idx].num_rowgroups;
        auto processed_child_rows  = 0;

        for (std::size_t rowgroup_id = 0; rowgroup_id < stripe_num_row_groups;
             rowgroup_id++, processed_row_groups++) {
          auto const child_rows = row_groups[processed_row_groups][parent_col_idx].num_child_rows;
          for (size_type id = 0; id < p_col.num_children; id++) {
            auto const child_col_idx                                  = index + id;
            rwgrp_meta[processed_row_groups][child_col_idx].start_row = processed_child_rows;
            rwgrp_meta[processed_row_groups][child_col_idx].num_rows  = child_rows;
          }
          processed_child_rows += child_rows;
        }
      }

      // Aggregate start row, number of rows per chunk and total number of rows in a column
      auto const child_rows = chunks[stripe_id][parent_col_idx].num_child_rows;
      // printf("     stripe_id: %d: child_rows: %d\n", (int)stripe_id, (int)child_rows);
      // printf("      p_col.num_children: %d\n", (int)p_col.num_children);

      for (size_type id = 0; id < p_col.num_children; id++) {
        auto const child_col_idx = index + id;

        // TODO: Check for overflow here.
        num_child_rows[child_col_idx] += child_rows;
        num_child_rows_per_stripe[stripe_id][child_col_idx] = child_rows;
        // start row could be different for each column when there is nesting at each stripe level
        child_start_row[stripe_id][child_col_idx] = (stripe_id == 0) ? 0 : start_row;
        // printf("update child_start_row (%d, %d): %d\n",
        //        (int)stripe_id,
        //        (int)child_col_idx,
        //        (int)start_row);
      }
      start_row += child_rows;
      // printf("        start_row: %d\n", (int)start_row);
    }

    // Parent column null mask and null count would be required for child column
    // to adjust its nullmask.
    auto type              = out_buffers[parent_col_idx].type.id();
    auto parent_null_count = static_cast<uint32_t>(out_buffers[parent_col_idx].null_count());
    auto parent_valid_map  = out_buffers[parent_col_idx].null_mask();
    auto num_rows          = out_buffers[parent_col_idx].size;

    for (size_type id = 0; id < p_col.num_children; id++) {
      auto const child_col_idx                    = index + id;
      col_meta.parent_column_index[child_col_idx] = parent_col_idx;
      if (type == type_id::STRUCT) {
        parent_column_data[child_col_idx] = {parent_valid_map, parent_null_count};
        // Number of rows in child will remain same as parent in case of struct column
        num_child_rows[child_col_idx] = num_rows;
      } else {
        parent_column_data[child_col_idx] = {nullptr, 0};
      }
    }
    index += p_col.num_children;
  });
}

/**
 * @brief struct to store buffer data and size of list buffer
 */
struct list_buffer_data {
  size_type* data;
  size_type size;
};

// Generates offsets for list buffer from number of elements in a row.
void generate_offsets_for_list(host_span<list_buffer_data> buff_data, rmm::cuda_stream_view stream)
{
  for (auto& list_data : buff_data) {
    thrust::exclusive_scan(rmm::exec_policy_nosync(stream),
                           list_data.data,
                           list_data.data + list_data.size,
                           list_data.data);
  }
}

/**
 * @brief TODO
 * @param input
 * @param size_limit
 * @param stream
 * @return
 */
std::vector<chunk> find_table_splits(table_view const& input,
                                     size_type segment_length,
                                     std::size_t size_limit,
                                     rmm::cuda_stream_view stream)
{
  printf("find table split, seg length = %d, limit = %d \n", segment_length, (int)size_limit);

  // If segment_length is zero: we don't have any limit on granularity.
  // As such, set segment length to the number of rows.
  if (segment_length == 0) { segment_length = input.num_rows(); }

  // If we have small number of rows, need to adjust segment_length before calling to
  // `segmented_row_bit_count`.
  segment_length = std::min(segment_length, input.num_rows());

  // Default 10k rows.
  auto const d_segmented_sizes = cudf::detail::segmented_row_bit_count(
    input, segment_length, stream, rmm::mr::get_current_device_resource());

  auto segmented_sizes =
    cudf::detail::hostdevice_vector<cumulative_size>(d_segmented_sizes->size(), stream);

  // TODO: exec_policy_nosync
  thrust::transform(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator(0),
    thrust::make_counting_iterator(d_segmented_sizes->size()),
    segmented_sizes.d_begin(),
    [segment_length,
     num_rows = input.num_rows(),
     d_sizes  = d_segmented_sizes->view().begin<size_type>()] __device__(auto const segment_idx) {
      // Since the number of rows may not divisible by segment_length,
      // the last segment may be shorter than the others.
      auto const current_length =
        cuda::std::min(segment_length, num_rows - segment_length * segment_idx);
      auto const size = d_sizes[segment_idx];
      return cumulative_size{current_length, static_cast<std::size_t>(size)};
    });

  // TODO: remove:
  segmented_sizes.device_to_host_sync(stream);
  printf("total row sizes by segment = %d:\n", (int)segment_length);
  for (auto& size : segmented_sizes) {
    printf("size: %ld, %zu\n", size.count, size.size_bytes / CHAR_BIT);
  }

  // TODO: exec_policy_nosync
  thrust::inclusive_scan(rmm::exec_policy(stream),
                         segmented_sizes.d_begin(),
                         segmented_sizes.d_end(),
                         segmented_sizes.d_begin(),
                         cumulative_size_sum{});
  segmented_sizes.device_to_host_sync(stream);

  // Since the segment sizes are in bits, we need to multiply CHAR_BIT with the output limit.
  return find_splits(segmented_sizes, input.num_rows(), size_limit * CHAR_BIT);
}

}  // namespace

// TODO: this should be called per chunk of stripes.
void reader::impl::decompress_and_decode()
{
  if (_file_itm_data.has_no_data()) { return; }

  auto const stripe_chunk =
    _chunk_read_data.decode_stripe_chunks[_chunk_read_data.curr_decode_stripe_chunk++];
  auto const stripe_start = stripe_chunk.start_idx;
  auto const stripe_end   = stripe_chunk.start_idx + stripe_chunk.count;

  auto const load_stripe_start =
    _chunk_read_data.load_stripe_chunks[_chunk_read_data.curr_load_stripe_chunk - 1].start_idx;

  printf("\ndecoding data from stripe %d -> %d\n", (int)stripe_start, (int)stripe_end);

  auto const rows_to_skip = _file_itm_data.rows_to_skip;
  // auto const rows_to_read      = _file_itm_data.rows_to_read;
  auto const& selected_stripes = _file_itm_data.selected_stripes;

  // auto const rows_to_skip = 0;
  auto rows_to_read = 0;
  for (auto stripe_idx = stripe_start; stripe_idx < stripe_end; ++stripe_idx) {
    auto const& stripe     = selected_stripes[stripe_idx];
    auto const stripe_info = stripe.stripe_info;
    // TODO: check overflow
    // CUDF_EXPECTS(per_file_metadata[src_file_idx].ff.stripes[stripe_idx].numberOfRows <
    //                static_cast<uint64_t>(std::numeric_limits<size_type>::max()),
    //              "TODO");
    rows_to_read += static_cast<size_type>(stripe_info->numberOfRows);

    if (_file_itm_data.rows_to_skip > 0) {
      CUDF_EXPECTS(_file_itm_data.rows_to_skip < static_cast<int64_t>(stripe_info->numberOfRows),
                   "TODO");
    }
  }
  rows_to_read = std::min<int64_t>(rows_to_read - rows_to_skip, _file_itm_data.rows_to_read);
  _file_itm_data.rows_to_skip = 0;

  // Set up table for converting timestamp columns from local to UTC time
  auto const tz_table = [&, &selected_stripes = selected_stripes] {
    auto const has_timestamp_column = std::any_of(
      _selected_columns.levels.cbegin(), _selected_columns.levels.cend(), [&](auto const& col_lvl) {
        return std::any_of(col_lvl.cbegin(), col_lvl.cend(), [&](auto const& col_meta) {
          return _metadata.get_col_type(col_meta.id).kind == TypeKind::TIMESTAMP;
        });
      });

    return has_timestamp_column ? cudf::detail::make_timezone_transition_table(
                                    {}, selected_stripes[0].stripe_footer->writerTimezone, _stream)
                                : std::make_unique<cudf::table>();
  }();

  auto& lvl_stripe_data        = _file_itm_data.lvl_stripe_data;
  auto& null_count_prefix_sums = _file_itm_data.null_count_prefix_sums;
  auto& lvl_chunks             = _file_itm_data.lvl_data_chunks;

  null_count_prefix_sums.clear();

  // TODO: move this to global step
  lvl_chunks.resize(_selected_columns.num_levels());
  _out_buffers.clear();
  _out_buffers.resize(_selected_columns.num_levels());

  //
  //
  //
  // TODO: move this to reader_impl.cu, decomp and decode step
  //  std::size_t num_stripes = selected_stripes.size();
  std::size_t num_stripes = stripe_chunk.count;

  // Iterates through levels of nested columns, child column will be one level down
  // compared to parent column.
  auto& col_meta = *_col_meta;

#if 0
  printf("num_child_rows: (size %d)\n", (int)_col_meta->num_child_rows.size());
  if (_col_meta->num_child_rows.size()) {
    for (auto x : _col_meta->num_child_rows) {
      printf("%d, ", (int)x);
    }
    printf("\n");

    _col_meta->num_child_rows.clear();
  }

  printf("parent_column_data null count: (size %d)\n", (int)_col_meta->parent_column_data.size());
  if (_col_meta->parent_column_data.size()) {
    for (auto x : _col_meta->parent_column_data) {
      printf("%d, ", (int)x.null_count);
    }
    printf("\n");
    _col_meta->parent_column_data.clear();
  }

  printf("parent_column_index: (size %d)\n", (int)_col_meta->parent_column_index.size());
  if (_col_meta->parent_column_index.size()) {
    for (auto x : _col_meta->parent_column_index) {
      printf("%d, ", (int)x);
    }
    printf("\n");
    _col_meta->parent_column_index.clear();
  }

  printf("child_start_row: (size %d)\n", (int)_col_meta->child_start_row.size());
  if (_col_meta->child_start_row.size()) {
    for (auto x : _col_meta->child_start_row) {
      printf("%d, ", (int)x);
    }
    printf("\n");
    _col_meta->child_start_row.clear();
  }

  printf("num_child_rows_per_stripe: (size %d)\n",
         (int)_col_meta->num_child_rows_per_stripe.size());
  if (_col_meta->num_child_rows_per_stripe.size()) {
    for (auto x : _col_meta->num_child_rows_per_stripe) {
      printf("%d, ", (int)x);
    }
    printf("\n");
    _col_meta->num_child_rows_per_stripe.clear();
  }

  printf("rwgrp_meta: (size %d)\n", (int)_col_meta->rwgrp_meta.size());
  if (_col_meta->rwgrp_meta.size()) {
    for (auto x : _col_meta->rwgrp_meta) {
      printf("(%d | %d), ", (int)x.start_row, (int)x.num_rows);
    }
    printf("\n");
  }

#endif

  auto& lvl_stripe_stream_chunks = _file_itm_data.lvl_stripe_stream_chunks;

  for (std::size_t level = 0; level < _selected_columns.num_levels(); ++level) {
    printf("processing level = %d\n", (int)level);

    {
      _stream.synchronize();
      auto peak_mem = mem_stats_logger.peak_memory_usage();
      std::cout << __LINE__ << ", decomp and decode, peak_memory_usage: " << peak_mem << "("
                << (peak_mem * 1.0) / (1024.0 * 1024.0) << " MB)" << std::endl;
    }

    auto const& stripe_stream_chunks      = lvl_stripe_stream_chunks[level];
    auto const [stream_begin, stream_end] = get_range(stripe_stream_chunks, stripe_chunk);

    auto& columns_level = _selected_columns.levels[level];

    // TODO: do it in global step
    // Association between each ORC column and its cudf::column
    std::vector<orc_column_meta> nested_cols;

    // Get a list of column data types
    std::vector<data_type> column_types;
    for (auto& col : columns_level) {
      auto col_type =
        to_cudf_type(_metadata.get_col_type(col.id).kind,
                     _config.use_np_dtypes,
                     _config.timestamp_type.id(),
                     to_cudf_decimal_type(_config.decimal128_columns, _metadata, col.id));
      CUDF_EXPECTS(col_type != type_id::EMPTY, "Unknown type");
      if (col_type == type_id::DECIMAL32 or col_type == type_id::DECIMAL64 or
          col_type == type_id::DECIMAL128) {
        // sign of the scale is changed since cuDF follows c++ libraries like CNL
        // which uses negative scaling, but liborc and other libraries
        // follow positive scaling.
        auto const scale =
          -static_cast<size_type>(_metadata.get_col_type(col.id).scale.value_or(0));
        column_types.emplace_back(col_type, scale);
      } else {
        column_types.emplace_back(col_type);
      }

      // Map each ORC column to its column
      if (col_type == type_id::LIST or col_type == type_id::STRUCT) {
        nested_cols.emplace_back(col);
      }
    }

    auto const num_columns = columns_level.size();
    auto& chunks           = lvl_chunks[level];
    chunks = cudf::detail::hostdevice_2dvector<gpu::ColumnDesc>(num_stripes, num_columns, _stream);
    memset(chunks.base_host_ptr(), 0, chunks.size_bytes());

    {
      _stream.synchronize();
      auto peak_mem = mem_stats_logger.peak_memory_usage();
      std::cout << __LINE__ << ", decomp and decode, peak_memory_usage: " << peak_mem << "("
                << (peak_mem * 1.0) / (1024.0 * 1024.0) << " MB)" << std::endl;
    }

    const bool use_index =
      _config.use_index &&
      // Do stripes have row group index
      _metadata.is_row_grp_idx_present() &&
      // Only use if we don't have much work with complete columns & stripes
      // TODO: Consider nrows, gpu, and tune the threshold
      (rows_to_read > _metadata.get_row_index_stride() && !(_metadata.get_row_index_stride() & 7) &&
       _metadata.get_row_index_stride() != 0 && num_columns * num_stripes < 8 * 128) &&
      // Only use if first row is aligned to a stripe boundary
      // TODO: Fix logic to handle unaligned rows
      (rows_to_skip == 0);

    printf(" use_index: %d\n", (int)use_index);

    // Logically view streams as columns
    auto const& stream_info = _file_itm_data.lvl_stream_info[level];

    null_count_prefix_sums.emplace_back();
    null_count_prefix_sums.back().reserve(_selected_columns.levels[level].size());
    std::generate_n(std::back_inserter(null_count_prefix_sums.back()),
                    _selected_columns.levels[level].size(),
                    [&]() {
                      return cudf::detail::make_zeroed_device_uvector_async<uint32_t>(
                        num_stripes, _stream, rmm::mr::get_current_device_resource());
                    });

    // Tracker for eventually deallocating compressed and uncompressed data
    auto& stripe_data = lvl_stripe_data[level];

    int64_t stripe_start_row = 0;
    int64_t num_dict_entries = 0;
    int64_t num_rowgroups    = 0;

    // TODO: Stripe and stream idx must be by chunk.
    //    std::size_t stripe_idx = 0;
    std::size_t stream_idx = 0;

    for (auto stripe_idx = stripe_start; stripe_idx < stripe_end; ++stripe_idx) {
      //    for (auto const& stripe : selected_stripes) {

      printf("processing stripe_idx = %d\n", (int)stripe_idx);
      auto const& stripe       = selected_stripes[stripe_idx];
      auto const stripe_info   = stripe.stripe_info;
      auto const stripe_footer = stripe.stripe_footer;

      // printf("stripeinfo->indexLength: %d, data: %d\n",
      //        (int)stripe_info->indexLength,
      //        (int)stripe_info->dataLength);

      auto const total_data_size = gather_stream_info_and_column_desc(stripe_idx - stripe_start,
                                                                      level,
                                                                      stripe_info,
                                                                      stripe_footer,
                                                                      col_meta.orc_col_map[level],
                                                                      _metadata.get_types(),
                                                                      use_index,
                                                                      level == 0,
                                                                      &num_dict_entries,
                                                                      &stream_idx,
                                                                      std::nullopt,  // stream_info
                                                                      &chunks);

      auto const is_stripe_data_empty = total_data_size == 0;
      printf("is_stripe_data_empty: %d\n", (int)is_stripe_data_empty);

      CUDF_EXPECTS(not is_stripe_data_empty or stripe_info->indexLength == 0,
                   "Invalid index rowgroup stream data");

      // TODO: Wrong?
      // stripe load_stripe_start?
      auto dst_base = static_cast<uint8_t*>(stripe_data[stripe_idx - load_stripe_start].data());

      // printf("line %d\n", __LINE__);
      // fflush(stdout);

      auto const num_rows_per_stripe = static_cast<int64_t>(stripe_info->numberOfRows);
      printf(" num_rows_per_stripe : %d\n", (int)num_rows_per_stripe);

      auto const rowgroup_id    = num_rowgroups;
      auto stripe_num_rowgroups = 0;
      if (use_index) {
        stripe_num_rowgroups = (num_rows_per_stripe + _metadata.get_row_index_stride() - 1) /
                               _metadata.get_row_index_stride();
      }

      // printf("line %d\n", __LINE__);
      // fflush(stdout);

      // Update chunks to reference streams pointers
      for (std::size_t col_idx = 0; col_idx < num_columns; col_idx++) {
        auto& chunk = chunks[stripe_idx - stripe_start][col_idx];
        // start row, number of rows in a each stripe and total number of rows
        // may change in lower levels of nesting
        chunk.start_row =
          (level == 0)
            ? stripe_start_row
            : col_meta.child_start_row[(stripe_idx - stripe_start) * num_columns + col_idx];
        chunk.num_rows =
          (level == 0)
            ? static_cast<int64_t>(stripe_info->numberOfRows)
            : col_meta
                .num_child_rows_per_stripe[(stripe_idx - stripe_start) * num_columns + col_idx];
        printf("col idx: %d, start_row: %d, num rows: %d\n",
               (int)col_idx,
               (int)chunk.start_row,
               (int)chunk.num_rows);

        chunk.column_num_rows = (level == 0) ? rows_to_read : col_meta.num_child_rows[col_idx];
        chunk.parent_validity_info =
          (level == 0) ? column_validity_info{} : col_meta.parent_column_data[col_idx];
        chunk.parent_null_count_prefix_sums =
          (level == 0)
            ? nullptr
            : null_count_prefix_sums[level - 1][col_meta.parent_column_index[col_idx]].data();
        chunk.encoding_kind = stripe_footer->columns[columns_level[col_idx].id].kind;
        chunk.type_kind =
          _metadata.per_file_metadata[stripe.source_idx].ff.types[columns_level[col_idx].id].kind;

        printf("type: %d\n", (int)chunk.type_kind);

        // num_child_rows for a struct column will be same, for other nested types it will be
        // calculated.
        chunk.num_child_rows = (chunk.type_kind != orc::STRUCT) ? 0 : chunk.num_rows;
        chunk.dtype_id       = column_types[col_idx].id();
        chunk.decimal_scale  = _metadata.per_file_metadata[stripe.source_idx]
                                .ff.types[columns_level[col_idx].id]
                                .scale.value_or(0);

        chunk.rowgroup_id   = rowgroup_id;
        chunk.dtype_len     = (column_types[col_idx].id() == type_id::STRING)
                                ? sizeof(string_index_pair)
                              : ((column_types[col_idx].id() == type_id::LIST) or
                             (column_types[col_idx].id() == type_id::STRUCT))
                                ? sizeof(size_type)
                                : cudf::size_of(column_types[col_idx]);
        chunk.num_rowgroups = stripe_num_rowgroups;
        // printf("stripe_num_rowgroups: %d\n", (int)stripe_num_rowgroups);

        if (chunk.type_kind == orc::TIMESTAMP) {
          chunk.timestamp_type_id = _config.timestamp_type.id();
        }
        if (not is_stripe_data_empty) {
          for (int k = 0; k < gpu::CI_NUM_STREAMS; k++) {
            chunk.streams[k] = dst_base + stream_info[chunk.strm_id[k] + stream_begin].dst_pos;
            // printf("chunk.streams[%d] of chunk.strm_id[%d], stripe %d | %d, collect from %d\n",
            //        (int)k,
            //        (int)chunk.strm_id[k],
            //        (int)stripe_idx,
            //        (int)stripe_start,
            //        (int)(chunk.strm_id[k] + stream_begin));
          }
        }
      }

      // printf("line %d\n", __LINE__);
      // fflush(stdout);

      stripe_start_row += num_rows_per_stripe;
      num_rowgroups += stripe_num_rowgroups;

      //      stripe_idx++;
    }  // for (stripe : selected_stripes)

    // printf("line %d\n", __LINE__);
    // fflush(stdout);

    if (stripe_data.empty()) { continue; }

    // Process dataset chunk pages into output columns
    auto row_groups =
      cudf::detail::hostdevice_2dvector<gpu::RowGroup>(num_rowgroups, num_columns, _stream);
    if (level > 0 and row_groups.size().first) {
      cudf::host_span<gpu::RowGroup> row_groups_span(row_groups.base_host_ptr(),
                                                     num_rowgroups * num_columns);
      auto& rw_grp_meta = col_meta.rwgrp_meta;

      // Update start row and num rows per row group
      std::transform(rw_grp_meta.begin(),
                     rw_grp_meta.end(),
                     row_groups_span.begin(),
                     rw_grp_meta.begin(),
                     [&](auto meta, auto& row_grp) {
                       row_grp.num_rows  = meta.num_rows;
                       row_grp.start_row = meta.start_row;
                       return meta;
                     });
    }

    // printf("line %d\n", __LINE__);
    // fflush(stdout);

    // Setup row group descriptors if using indexes
    if (_metadata.per_file_metadata[0].ps.compression != orc::NONE) {
      // printf("decompress----------------------\n");
      // printf("line %d\n", __LINE__);
      // fflush(stdout);
      CUDF_EXPECTS(_chunk_read_data.curr_load_stripe_chunk > 0, "ERRRRR");

      {
        _stream.synchronize();
        auto peak_mem = mem_stats_logger.peak_memory_usage();
        std::cout << __LINE__ << ", decomp and decode, peak_memory_usage: " << peak_mem << "("
                  << (peak_mem * 1.0) / (1024.0 * 1024.0) << " MB)" << std::endl;
      }

      auto decomp_data = decompress_stripe_data(
        _chunk_read_data.load_stripe_chunks[_chunk_read_data.curr_load_stripe_chunk - 1],
        stripe_chunk,
        _file_itm_data.compinfo_map,
        *_metadata.per_file_metadata[0].decompressor,
        stripe_data,
        stream_info,
        chunks,
        row_groups,
        num_stripes,
        _metadata.get_row_index_stride(),
        level == 0,
        _stream);
      // stripe_data.clear();
      // stripe_data.push_back(std::move(decomp_data));

      // TODO: only reset each one if the new size/type are different.
      stripe_data[stripe_start - load_stripe_start] = std::move(decomp_data);
      for (int64_t i = 1; i < stripe_chunk.count; ++i) {
        stripe_data[i + stripe_start - load_stripe_start] = {};
      }

      {
        _stream.synchronize();
        auto peak_mem = mem_stats_logger.peak_memory_usage();
        std::cout << __LINE__ << ", decomp and decode, peak_memory_usage: " << peak_mem << "("
                  << (peak_mem * 1.0) / (1024.0 * 1024.0) << " MB)" << std::endl;
      }

      // printf("line %d\n", __LINE__);
      // fflush(stdout);

    } else {
      // printf("no decompression----------------------\n");

      if (row_groups.size().first) {
        // printf("line %d\n", __LINE__);
        // fflush(stdout);
        chunks.host_to_device_async(_stream);
        row_groups.host_to_device_async(_stream);
        row_groups.host_to_device_async(_stream);
        gpu::ParseRowGroupIndex(row_groups.base_device_ptr(),
                                nullptr,
                                chunks.base_device_ptr(),
                                num_columns,
                                num_stripes,
                                _metadata.get_row_index_stride(),
                                level == 0,
                                _stream);
      }
    }

    // printf("line %d\n", __LINE__);
    // fflush(stdout);

    {
      _stream.synchronize();
      auto peak_mem = mem_stats_logger.peak_memory_usage();
      std::cout << __LINE__ << ", decomp and decode, peak_memory_usage: " << peak_mem << "("
                << (peak_mem * 1.0) / (1024.0 * 1024.0) << " MB)" << std::endl;
    }

    // TODO: do not clear but reset each one.
    // and only reset if the new size/type are different.
    _out_buffers[level].clear();

    {
      _stream.synchronize();
      auto peak_mem = mem_stats_logger.peak_memory_usage();
      std::cout << __LINE__ << ", decomp and decode, peak_memory_usage: " << peak_mem << "("
                << (peak_mem * 1.0) / (1024.0 * 1024.0) << " MB)" << std::endl;
    }

    for (std::size_t i = 0; i < column_types.size(); ++i) {
      bool is_nullable = false;
      for (std::size_t j = 0; j < num_stripes; ++j) {
        if (chunks[j][i].strm_len[gpu::CI_PRESENT] != 0) {
          printf("   is nullable\n");
          is_nullable = true;
          break;
        }
      }
      auto is_list_type = (column_types[i].id() == type_id::LIST);
      auto n_rows       = (level == 0) ? rows_to_read : col_meta.num_child_rows[i];

      // printf("  create col, num rows: %d\n", (int)n_rows);

      {
        _stream.synchronize();
        auto peak_mem = mem_stats_logger.peak_memory_usage();
        std::cout << __LINE__ << ", decomp and decode, peak_memory_usage: " << peak_mem << "("
                  << (peak_mem * 1.0) / (1024.0 * 1024.0) << " MB)" << std::endl;
      }

      // For list column, offset column will be always size + 1
      if (is_list_type) n_rows++;
      _out_buffers[level].emplace_back(column_types[i], n_rows, is_nullable, _stream, _mr);

      {
        _stream.synchronize();
        auto peak_mem = mem_stats_logger.peak_memory_usage();
        std::cout << __LINE__ << ", buffer size: " << n_rows
                  << ", decomp and decode, peak_memory_usage: " << peak_mem << "("
                  << (peak_mem * 1.0) / (1024.0 * 1024.0) << " MB)" << std::endl;
      }
    }

    // printf("line %d\n", __LINE__);
    // fflush(stdout);

    {
      _stream.synchronize();
      auto peak_mem = mem_stats_logger.peak_memory_usage();
      std::cout << __LINE__ << ", decomp and decode, peak_memory_usage: " << peak_mem << "("
                << (peak_mem * 1.0) / (1024.0 * 1024.0) << " MB)" << std::endl;
    }

    decode_stream_data(num_dict_entries,
                       rows_to_skip,
                       _metadata.get_row_index_stride(),
                       level,
                       tz_table->view(),
                       chunks,
                       row_groups,
                       _out_buffers[level],
                       _stream,
                       _mr);

    {
      _stream.synchronize();
      auto peak_mem = mem_stats_logger.peak_memory_usage();
      std::cout << __LINE__ << ", decomp and decode, peak_memory_usage: " << peak_mem << "("
                << (peak_mem * 1.0) / (1024.0 * 1024.0) << " MB)" << std::endl;
    }

    // printf("line %d\n", __LINE__);
    // fflush(stdout);

    if (nested_cols.size()) {
      printf("have nested col\n");

      // Extract information to process nested child columns
      scan_null_counts(chunks, null_count_prefix_sums[level], _stream);

      row_groups.device_to_host_sync(_stream);
      aggregate_child_meta(stripe_start,
                           level,
                           _selected_columns,
                           chunks,
                           row_groups,
                           nested_cols,
                           _out_buffers[level],
                           col_meta);

      // ORC stores number of elements at each row, so we need to generate offsets from that
      std::vector<list_buffer_data> buff_data;
      std::for_each(
        _out_buffers[level].begin(), _out_buffers[level].end(), [&buff_data](auto& out_buffer) {
          if (out_buffer.type.id() == type_id::LIST) {
            auto data = static_cast<size_type*>(out_buffer.data());
            buff_data.emplace_back(list_buffer_data{data, out_buffer.size});
          }
        });

      if (not buff_data.empty()) { generate_offsets_for_list(buff_data, _stream); }
    }

    // printf("line %d\n", __LINE__);
    // fflush(stdout);
  }  // end loop level

  {
    _stream.synchronize();
    auto peak_mem = mem_stats_logger.peak_memory_usage();
    std::cout << __LINE__ << ", decomp and decode, peak_memory_usage: " << peak_mem << "("
              << (peak_mem * 1.0) / (1024.0 * 1024.0) << " MB)" << std::endl;
  }

  std::vector<std::unique_ptr<column>> out_columns;
  _out_metadata = get_meta_with_user_data();
  std::transform(
    _selected_columns.levels[0].begin(),
    _selected_columns.levels[0].end(),
    std::back_inserter(out_columns),
    [&](auto const& orc_col_meta) {
      _out_metadata.schema_info.emplace_back("");
      auto col_buffer = assemble_buffer(
        orc_col_meta.id, 0, *_col_meta, _metadata, _selected_columns, _out_buffers, _stream, _mr);
      return make_column(col_buffer, &_out_metadata.schema_info.back(), std::nullopt, _stream);
    });
  _chunk_read_data.decoded_table = std::make_unique<table>(std::move(out_columns));

  // TODO: do not clear but reset each one.
  // and only reset if the new size/type are different.
  // This clear is just to check if there is memory leak.
  for (std::size_t level = 0; level < _selected_columns.num_levels(); ++level) {
    _out_buffers[level].clear();

    auto& stripe_data = lvl_stripe_data[level];

    if (_metadata.per_file_metadata[0].ps.compression != orc::NONE) {
      stripe_data[stripe_start - load_stripe_start] = {};
    } else {
      for (int64_t i = 0; i < stripe_chunk.count; ++i) {
        stripe_data[i + stripe_start - load_stripe_start] = {};
      }
    }
  }

  {
    _stream.synchronize();
    auto peak_mem = mem_stats_logger.peak_memory_usage();
    std::cout << __LINE__ << ", decomp and decode, peak_memory_usage: " << peak_mem << "("
              << (peak_mem * 1.0) / (1024.0 * 1024.0) << " MB)" << std::endl;
  }

  // printf("col: \n");
  // cudf::test::print(_chunk_read_data.decoded_table->get_column(0).view());

  // DEBUG only
  // _chunk_read_data.output_size_limit = _chunk_read_data.data_read_limit / 3;

  _chunk_read_data.curr_output_table_chunk = 0;
  _chunk_read_data.output_table_chunks =
    _chunk_read_data.output_size_limit == 0
      ? std::vector<chunk>{chunk{0, _chunk_read_data.decoded_table->num_rows()}}
      : find_table_splits(_chunk_read_data.decoded_table->view(),
                          _chunk_read_data.output_row_granularity,
                          _chunk_read_data.output_size_limit,
                          _stream);

  auto& splits = _chunk_read_data.output_table_chunks;
  printf("------------\nSplits decoded table (/total num rows = %d): \n",
         (int)_chunk_read_data.decoded_table->num_rows());
  for (size_t idx = 0; idx < splits.size(); idx++) {
    printf("{%ld, %ld}\n", splits[idx].start_idx, splits[idx].count);
  }
  fflush(stdout);

  {
    _stream.synchronize();
    auto peak_mem = mem_stats_logger.peak_memory_usage();
    std::cout << "decomp and decode, peak_memory_usage: " << peak_mem << "("
              << (peak_mem * 1.0) / (1024.0 * 1024.0) << " MB)" << std::endl;
  }
}

void reader::impl::prepare_data(int64_t skip_rows,
                                std::optional<size_type> const& num_rows_opt,
                                std::vector<std::vector<size_type>> const& stripes)
{
  // Selected columns at different levels of nesting are stored in different elements
  // of `selected_columns`; thus, size == 1 means no nested columns
  CUDF_EXPECTS(skip_rows == 0 or _selected_columns.num_levels() == 1,
               "skip_rows is not supported by nested columns");

  // There are no columns in the table.
  if (_selected_columns.num_levels() == 0) { return; }

  std::cout << "call global, skip = " << skip_rows << std::endl;

  global_preprocess(skip_rows, num_rows_opt, stripes);

  if (!_chunk_read_data.more_table_chunk_to_output()) {
    if (!_chunk_read_data.more_stripe_to_decode() && _chunk_read_data.more_stripe_to_load()) {
      printf("load more data\n\n");
      load_data();
    }

    if (_chunk_read_data.more_stripe_to_decode()) {
      printf("decode more data\n\n");
      decompress_and_decode();
    }
  }

  printf("done load and decode data\n\n");

  // decompress_and_decode();
  // while (_chunk_read_data.more_stripe_to_decode()) {
  //   decompress_and_decode();
  //   _file_itm_data.out_buffers.push_back(std::move(_out_buffers));
  // }
}

table_with_metadata reader::impl::make_output_chunk()
{
  {
    _stream.synchronize();
    auto peak_mem = mem_stats_logger.peak_memory_usage();
    std::cout << "start to make out, peak_memory_usage: " << peak_mem << "("
              << (peak_mem * 1.0) / (1024.0 * 1024.0) << " MB)" << std::endl;
  }

  // There is no columns in the table.
  if (_selected_columns.num_levels() == 0) { return {std::make_unique<table>(), table_metadata{}}; }

  // If no rows or stripes to read, return empty columns
  if (_file_itm_data.has_no_data() || !_chunk_read_data.more_table_chunk_to_output()) {
    printf("has no next\n");
    std::vector<std::unique_ptr<column>> out_columns;
    auto out_metadata = get_meta_with_user_data();
    std::transform(_selected_columns.levels[0].begin(),
                   _selected_columns.levels[0].end(),
                   std::back_inserter(out_columns),
                   [&](auto const& col_meta) {
                     out_metadata.schema_info.emplace_back("");
                     return create_empty_column(col_meta.id,
                                                _metadata,
                                                _config.decimal128_columns,
                                                _config.use_np_dtypes,
                                                _config.timestamp_type,
                                                out_metadata.schema_info.back(),
                                                _stream);
                   });
    return {std::make_unique<table>(std::move(out_columns)), std::move(out_metadata)};
  }

#if 1
  auto out_table = [&] {
    if (_chunk_read_data.output_table_chunks.size() == 1) {
      _chunk_read_data.curr_output_table_chunk++;
      printf("one chunk, no more table---------------------------------\n");
      return std::move(_chunk_read_data.decoded_table);
    }

    {
      _stream.synchronize();
      auto peak_mem = mem_stats_logger.peak_memory_usage();
      std::cout << "prepare to make out, peak_memory_usage: " << peak_mem << "("
                << (peak_mem * 1.0) / (1024.0 * 1024.0) << " MB)" << std::endl;
    }

    auto const out_chunk =
      _chunk_read_data.output_table_chunks[_chunk_read_data.curr_output_table_chunk++];
    auto const out_tview =
      cudf::detail::slice(_chunk_read_data.decoded_table->view(),
                          {static_cast<size_type>(out_chunk.start_idx),
                           static_cast<size_type>(out_chunk.start_idx + out_chunk.count)},
                          _stream)[0];
    {
      _stream.synchronize();
      auto peak_mem = mem_stats_logger.peak_memory_usage();
      std::cout << "done make out, peak_memory_usage: " << peak_mem << "("
                << (peak_mem * 1.0) / (1024.0 * 1024.0) << " MB)" << std::endl;
    }

    return std::make_unique<table>(out_tview, _stream, _mr);
  }();

#endif

  if (!_chunk_read_data.has_next()) {
    static int count{0};
    count++;
    _stream.synchronize();
    auto peak_mem = mem_stats_logger.peak_memory_usage();
    std::cout << "complete, " << count << ", peak_memory_usage: " << peak_mem
              << " , MB = " << (peak_mem * 1.0) / (1024.0 * 1024.0) << std::endl;
  } else {
    _stream.synchronize();
    auto peak_mem = mem_stats_logger.peak_memory_usage();
    std::cout << "done, partial, peak_memory_usage: " << peak_mem
              << " , MB = " << (peak_mem * 1.0) / (1024.0 * 1024.0) << std::endl;
  }

  return {std::move(out_table), _out_metadata};
}

table_metadata reader::impl::get_meta_with_user_data()
{
  if (_meta_with_user_data) { return table_metadata{*_meta_with_user_data}; }

  // Copy user data to the output metadata.
  table_metadata out_metadata;
  out_metadata.per_file_user_data.reserve(_metadata.per_file_metadata.size());
  std::transform(_metadata.per_file_metadata.cbegin(),
                 _metadata.per_file_metadata.cend(),
                 std::back_inserter(out_metadata.per_file_user_data),
                 [](auto const& meta) {
                   std::unordered_map<std::string, std::string> kv_map;
                   std::transform(meta.ff.metadata.cbegin(),
                                  meta.ff.metadata.cend(),
                                  std::inserter(kv_map, kv_map.end()),
                                  [](auto const& kv) {
                                    return std::pair{kv.name, kv.value};
                                  });
                   return kv_map;
                 });
  out_metadata.user_data = {out_metadata.per_file_user_data[0].begin(),
                            out_metadata.per_file_user_data[0].end()};

  // Save the output table metadata into `_meta_with_user_data` for reuse next time.
  _meta_with_user_data = std::make_unique<table_metadata>(out_metadata);

  return out_metadata;
}

reader::impl::impl(std::vector<std::unique_ptr<datasource>>&& sources,
                   orc_reader_options const& options,
                   rmm::cuda_stream_view stream,
                   rmm::mr::device_memory_resource* mr)
  : reader::impl::impl(0UL, 0UL, std::move(sources), options, stream, mr)
{
}

reader::impl::impl(std::size_t output_size_limit,
                   std::size_t data_read_limit,
                   std::vector<std::unique_ptr<datasource>>&& sources,
                   orc_reader_options const& options,
                   rmm::cuda_stream_view stream,
                   rmm::mr::device_memory_resource* mr)
  : reader::impl::impl(output_size_limit,
                       data_read_limit,
                       DEFAULT_OUTPUT_ROW_GRANULARITY,
                       std::move(sources),
                       options,
                       stream,
                       mr)
{
}

reader::impl::impl(std::size_t output_size_limit,
                   std::size_t data_read_limit,
                   size_type output_row_granularity,
                   std::vector<std::unique_ptr<datasource>>&& sources,
                   orc_reader_options const& options,
                   rmm::cuda_stream_view stream,
                   rmm::mr::device_memory_resource* mr)
  : _stream(stream),
    _mr(mr),
    _config{options.get_timestamp_type(),
            options.is_enabled_use_index(),
            options.is_enabled_use_np_dtypes(),
            options.get_decimal128_columns(),
            options.get_skip_rows(),
            options.get_num_rows(),
            options.get_stripes()},
    _col_meta{std::make_unique<reader_column_meta>()},
    _sources(std::move(sources)),
    _metadata{_sources, stream},
    _selected_columns{_metadata.select_columns(options.get_columns())},
    _chunk_read_data{
      output_size_limit,
      data_read_limit,
      output_row_granularity > 0 ? output_row_granularity : DEFAULT_OUTPUT_ROW_GRANULARITY},
    mem_stats_logger(mr)
{
  printf("construct reader , limit = %d, %d, gradunarity %d \n",

         (int)output_size_limit,
         (int)data_read_limit,
         (int)output_row_granularity

  );
}

table_with_metadata reader::impl::read(int64_t skip_rows,
                                       std::optional<size_type> const& num_rows_opt,
                                       std::vector<std::vector<size_type>> const& stripes)
{
  prepare_data(skip_rows, num_rows_opt, stripes);
  return make_output_chunk();
}

bool reader::impl::has_next()
{
  printf("==================query has next \n");
  prepare_data(_config.skip_rows, _config.num_read_rows, _config.selected_stripes);

  printf("has next: %d\n", (int)_chunk_read_data.has_next());
  return _chunk_read_data.has_next();
}

table_with_metadata reader::impl::read_chunk()
{
  printf("==================call read chunk\n");
  {
    _stream.synchronize();
    auto peak_mem = mem_stats_logger.peak_memory_usage();
    std::cout << "\n\n\n------------start read chunk, peak_memory_usage: " << peak_mem << "("
              << (peak_mem * 1.0) / (1024.0 * 1024.0) << " MB)" << std::endl;
  }

  {
    static int count{0};
    ++count;

#if 0
    if (count == 3) {
      _file_itm_data.lvl_stripe_data.clear();
      {
        _stream.synchronize();
        auto peak_mem = mem_stats_logger.peak_memory_usage();
        std::cout << "clear all, peak_memory_usage: " << peak_mem << "("
                  << (peak_mem * 1.0) / (1024.0 * 1024.0) << " MB)" << std::endl;
      }
      exit(0);
    }
#endif
  }

  prepare_data(_config.skip_rows, _config.num_read_rows, _config.selected_stripes);

  {
    _stream.synchronize();
    auto peak_mem = mem_stats_logger.peak_memory_usage();
    std::cout << "done prepare data, peak_memory_usage: " << peak_mem << "("
              << (peak_mem * 1.0) / (1024.0 * 1024.0) << " MB)" << std::endl;
  }

  return make_output_chunk();
}

}  // namespace cudf::io::orc::detail
