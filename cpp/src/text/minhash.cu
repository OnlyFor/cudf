#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/copy.hpp>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/sequence.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/hashing/detail/hashing.hpp>
#include <cudf/hashing/detail/murmurhash3_x64_128.cuh>
#include <cudf/hashing/detail/murmurhash3_x86_32.cuh>
#include <cudf/lists/list_device_view.cuh>
#include <cudf/lists/lists_column_device_view.cuh>
#include <cudf/strings/string_view.cuh>
#include <cudf/utilities/default_stream.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/memory_resource.hpp>

#include <nvtext/minhash.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

#include <cuda/atomic>
#include <thrust/execution_policy.h>
#include <thrust/fill.h>

#include <limits>

namespace nvtext {
namespace detail {
namespace {

constexpr cudf::thread_index_type block_size = 256;
// for tuning independently from block_size
constexpr cudf::thread_index_type tile_size = block_size;

/**
 * @brief Compute the minhash of each string for each seed
 *
 * This is a block-per-string algorithm where parallel threads within a block
 * work on a single string row.
 *
 * @tparam HashFunction hash function to use on each substring
 *
 * @param d_strings Strings column to process
 * @param seeds Seeds for hashing each string
 * @param width Substring window size in characters
 * @param working_memory Memory used to hold intermediate hash values
 * @param d_hashes Minhash output values for each string
 */
template <
  typename HashFunction,
  typename hash_value_type = std::
    conditional_t<std::is_same_v<typename HashFunction::result_type, uint32_t>, uint32_t, uint64_t>>
CUDF_KERNEL void minhash_kernel(cudf::column_device_view const d_strings,
                                cudf::device_span<hash_value_type const> seeds,
                                cudf::size_type width,
                                hash_value_type* working_memory,
                                hash_value_type* d_hashes)
{
  auto const idx     = cudf::detail::grid_1d::global_thread_id();
  auto const str_idx = idx / tile_size;
  if (str_idx >= d_strings.size()) { return; }
  if (d_strings.is_null(str_idx)) { return; }

  auto const d_str    = d_strings.element<cudf::string_view>(str_idx);
  auto const init     = d_str.empty() ? 0 : std::numeric_limits<hash_value_type>::max();
  auto const lane_idx = idx % tile_size;

  auto tile_hashes = working_memory + (str_idx * tile_size * seeds.size());

  // initialize working memory
  for (std::size_t seed_idx = lane_idx; seed_idx < seeds.size(); seed_idx += tile_size) {
    auto begin = tile_hashes + (seed_idx * tile_size);
    thrust::uninitialized_fill(thrust::seq, begin, begin + tile_size, init);
  }
  __syncthreads();

  auto const d_output = d_hashes + (str_idx * seeds.size());

  auto const begin = d_str.data() + lane_idx;
  auto const end   = d_str.data() + d_str.size_bytes();

  // each lane hashes 'width' substrings of d_str
  for (auto itr = begin; itr < end; itr += tile_size) {
    if (cudf::strings::detail::is_utf8_continuation_char(*itr)) { continue; }
    auto const check_str =  // used for counting 'width' characters
      cudf::string_view(itr, static_cast<cudf::size_type>(thrust::distance(itr, end)));
    auto const [bytes, left] = cudf::strings::detail::bytes_to_character_position(check_str, width);
    if ((itr != d_str.data()) && (left > 0)) { continue; }  // true if past the end of the string

    auto const hash_str = cudf::string_view(itr, bytes);
    for (std::size_t seed_idx = 0; seed_idx < seeds.size(); ++seed_idx) {
      auto const hasher = HashFunction(seeds[seed_idx]);
      hash_value_type hv;
      if constexpr (std::is_same_v<hash_value_type, uint32_t>) {
        hv = hasher(hash_str);
      } else {
        hv = thrust::get<0>(hasher(hash_str));
      }
      tile_hashes[(seed_idx * tile_size) + lane_idx] =
        cuda::std::min(hv, tile_hashes[(seed_idx * tile_size) + lane_idx]);
    }
  }
  __syncthreads();

  // compute final result
  for (std::size_t seed_idx = lane_idx; seed_idx < seeds.size(); seed_idx += tile_size) {
    auto begin = tile_hashes + (seed_idx * tile_size);
    auto hv    = thrust::reduce(thrust::seq, begin, begin + tile_size, init, thrust::minimum{});
    d_output[seed_idx] = hv;
  }
}

template <
  typename HashFunction,
  typename hash_value_type = std::
    conditional_t<std::is_same_v<typename HashFunction::result_type, uint32_t>, uint32_t, uint64_t>>
CUDF_KERNEL void minhash_permuted_kernel(cudf::column_device_view const d_strings,
                                         cudf::device_span<hash_value_type const> parmA,
                                         cudf::device_span<hash_value_type const> parmB,
                                         cudf::size_type width,
                                         hash_value_type* d_hashes)
{
  auto const idx     = cudf::detail::grid_1d::global_thread_id();
  auto const str_idx = idx / tile_size;
  if (str_idx >= d_strings.size()) { return; }
  if (d_strings.is_null(str_idx)) { return; }

  auto const d_str    = d_strings.element<cudf::string_view>(str_idx);
  auto const init     = d_str.empty() ? 0 : std::numeric_limits<hash_value_type>::max();
  auto const lane_idx = idx % tile_size;

  auto const d_output = d_hashes + (str_idx * parmA.size());

  auto const begin = d_str.data() + (lane_idx);
  auto const end   = d_str.data() + d_str.size_bytes();

  constexpr std::size_t seed_chunk   = 16;  // based on block-size==256
  constexpr uint64_t mersenne_prime  = (1UL << 61) - 1;
  constexpr hash_value_type hash_max = std::numeric_limits<hash_value_type>::max();

  extern __shared__ char shmem[];
  auto const block_hashes = reinterpret_cast<hash_value_type*>(shmem);

  for (std::size_t i = 0; i < parmA.size(); i += seed_chunk) {
    // initialize working memory
    auto const tile_hashes = block_hashes + (lane_idx * seed_chunk);
    thrust::uninitialized_fill(thrust::seq, tile_hashes, tile_hashes + seed_chunk, init);
    __syncthreads();

    auto const seed_count = cuda::std::min(seed_chunk, parmA.size() - i);

    // each lane hashes 'width' substrings of d_str
    for (auto itr = begin; itr < end; itr += tile_size) {
      if (cudf::strings::detail::is_utf8_continuation_char(*itr)) { continue; }
      auto const check_str =  // used for counting 'width' characters
        cudf::string_view(itr, static_cast<cudf::size_type>(thrust::distance(itr, end)));
      auto const [bytes, left] =
        cudf::strings::detail::bytes_to_character_position(check_str, width);
      if ((itr != d_str.data()) && (left > 0)) { continue; }  // true if past the end of the string

      auto const hash_str = cudf::string_view(itr, bytes);
      auto const hasher   = HashFunction(parmA[0]);
      hash_value_type hv1;
      if constexpr (std::is_same_v<hash_value_type, uint32_t>) {
        hv1 = hasher(hash_str);
      } else {
        hv1 = thrust::get<0>(hasher(hash_str));
      }

      for (std::size_t seed_idx = i; seed_idx < (i + seed_count); ++seed_idx) {
        hash_value_type const hv =
          seed_idx == 0 ? hv1
                        : ((hv1 * parmA[seed_idx] + parmB[seed_idx]) % mersenne_prime) & hash_max;
        auto const block_idx    = ((seed_idx % seed_chunk) * tile_size) + lane_idx;
        block_hashes[block_idx] = cuda::std::min(hv, block_hashes[block_idx]);
      }
    }
    __syncthreads();

    if (lane_idx < seed_count) {
      auto const hvs = block_hashes + (lane_idx * tile_size);
      auto const hv  = thrust::reduce(thrust::seq, hvs, hvs + tile_size, init, thrust::minimum{});
      d_output[lane_idx + i] = hv;
    }
    __syncthreads();
  }
}

template <
  typename HashFunction,
  typename hash_value_type = std::
    conditional_t<std::is_same_v<typename HashFunction::result_type, uint32_t>, uint32_t, uint64_t>>
std::unique_ptr<cudf::column> minhash_fn(cudf::strings_column_view const& input,
                                         cudf::device_span<hash_value_type const> seeds,
                                         cudf::size_type width,
                                         rmm::cuda_stream_view stream,
                                         rmm::device_async_resource_ref mr)
{
  CUDF_EXPECTS(!seeds.empty(), "Parameter seeds cannot be empty", std::invalid_argument);
  CUDF_EXPECTS(width >= 2,
               "Parameter width should be an integer value of 2 or greater",
               std::invalid_argument);
  CUDF_EXPECTS((static_cast<std::size_t>(input.size()) * seeds.size()) <
                 static_cast<std::size_t>(std::numeric_limits<cudf::size_type>::max()),
               "The number of seeds times the number of input rows exceeds the column size limit",
               std::overflow_error);

  auto const output_type = cudf::data_type{cudf::type_to_id<hash_value_type>()};
  if (input.is_empty()) { return cudf::make_empty_column(output_type); }

  auto const d_strings = cudf::column_device_view::create(input.parent(), stream);

  auto hashes   = cudf::make_numeric_column(output_type,
                                          input.size() * static_cast<cudf::size_type>(seeds.size()),
                                          cudf::mask_state::UNALLOCATED,
                                          stream,
                                          mr);
  auto d_hashes = hashes->mutable_view().data<hash_value_type>();

  auto const wm_size  = cudf::util::round_up_safe(seeds.size() * tile_size * input.size(),
                                                 static_cast<std::size_t>(block_size));
  auto working_memory = rmm::device_uvector<hash_value_type>(wm_size, stream);

  cudf::detail::grid_1d grid{static_cast<cudf::thread_index_type>(input.size()) * tile_size,
                             block_size};
  minhash_kernel<HashFunction><<<grid.num_blocks, grid.num_threads_per_block, 0, stream.value()>>>(
    *d_strings, seeds, width, working_memory.data(), d_hashes);

  return hashes;
}

template <
  typename HashFunction,
  typename hash_value_type = std::
    conditional_t<std::is_same_v<typename HashFunction::result_type, uint32_t>, uint32_t, uint64_t>>
std::unique_ptr<cudf::column> minhash_fn(cudf::strings_column_view const& input,
                                         cudf::device_span<hash_value_type const> parmA,
                                         cudf::device_span<hash_value_type const> parmB,
                                         cudf::size_type width,
                                         rmm::cuda_stream_view stream,
                                         rmm::device_async_resource_ref mr)
{
  CUDF_EXPECTS(!parmA.empty(), "Parameters A and B cannot be empty", std::invalid_argument);
  CUDF_EXPECTS(width >= 2,
               "Parameter width should be an integer value of 2 or greater",
               std::invalid_argument);
  CUDF_EXPECTS((static_cast<std::size_t>(input.size()) * parmA.size()) <
                 static_cast<std::size_t>(std::numeric_limits<cudf::size_type>::max()),
               "The number of seeds times the number of input rows exceeds the column size limit",
               std::overflow_error);
  CUDF_EXPECTS(parmA.size() == parmB.size(),
               "Parameters A and B should have the same number of elements",
               std::invalid_argument);

  auto const output_type = cudf::data_type{cudf::type_to_id<hash_value_type>()};
  if (input.is_empty()) { return cudf::make_empty_column(output_type); }

  auto const d_strings = cudf::column_device_view::create(input.parent(), stream);

  auto hashes   = cudf::make_numeric_column(output_type,
                                          input.size() * static_cast<cudf::size_type>(parmA.size()),
                                          cudf::mask_state::UNALLOCATED,
                                          stream,
                                          mr);
  auto d_hashes = hashes->mutable_view().data<hash_value_type>();

  // 16 seeds can be held in shared-memory: 32K/block_size(256)/sizeof(hash_value_type) = ~16
  auto const shmem_size = block_size * 16 * sizeof(hash_value_type);

  cudf::detail::grid_1d grid{static_cast<cudf::thread_index_type>(input.size()) * tile_size,
                             block_size};
  minhash_permuted_kernel<HashFunction>
    <<<grid.num_blocks, grid.num_threads_per_block, shmem_size, stream.value()>>>(
      *d_strings, parmA, parmB, width, d_hashes);

  return hashes;
}

/**
 * @brief Compute the minhash of each list row of strings for each seed
 *
 * This is a warp-per-row algorithm where parallel threads within a warp
 * work on strings in a single list row.
 *
 * @tparam HashFunction hash function to use on each string
 *
 * @param d_input List of strings to process
 * @param seeds Seeds for hashing each string
 * @param d_hashes Minhash output values (one per row)
 */
template <
  typename HashFunction,
  typename hash_value_type = std::
    conditional_t<std::is_same_v<typename HashFunction::result_type, uint32_t>, uint32_t, uint64_t>>
CUDF_KERNEL void minhash_word_kernel(cudf::detail::lists_column_device_view const d_input,
                                     cudf::device_span<hash_value_type const> seeds,
                                     hash_value_type* d_hashes)
{
  auto const idx     = cudf::detail::grid_1d::global_thread_id();
  auto const row_idx = idx / cudf::detail::warp_size;

  if (row_idx >= d_input.size()) { return; }
  if (d_input.is_null(row_idx)) { return; }

  auto const d_row    = cudf::list_device_view(d_input, row_idx);
  auto const d_output = d_hashes + (row_idx * seeds.size());

  // initialize hashes output for this row
  auto const lane_idx = static_cast<cudf::size_type>(idx % cudf::detail::warp_size);
  if (lane_idx == 0) {
    auto const init = d_row.size() == 0 ? 0 : std::numeric_limits<hash_value_type>::max();
    thrust::fill(thrust::seq, d_output, d_output + seeds.size(), init);
  }
  __syncwarp();

  // each lane hashes a string from the input row
  for (auto str_idx = lane_idx; str_idx < d_row.size(); str_idx += cudf::detail::warp_size) {
    auto const hash_str =
      d_row.is_null(str_idx) ? cudf::string_view{} : d_row.element<cudf::string_view>(str_idx);
    for (std::size_t seed_idx = 0; seed_idx < seeds.size(); ++seed_idx) {
      auto const hasher = HashFunction(seeds[seed_idx]);
      // hash string and store the min value
      hash_value_type hv;
      if constexpr (std::is_same_v<hash_value_type, uint32_t>) {
        hv = hasher(hash_str);
      } else {
        // This code path assumes the use of MurmurHash3_x64_128 which produces 2 uint64 values
        // but only uses the first uint64 value as requested by the LLM team.
        hv = thrust::get<0>(hasher(hash_str));
      }
      cuda::atomic_ref<hash_value_type, cuda::thread_scope_block> ref{*(d_output + seed_idx)};
      ref.fetch_min(hv, cuda::std::memory_order_relaxed);
    }
  }
}

template <
  typename HashFunction,
  typename hash_value_type = std::
    conditional_t<std::is_same_v<typename HashFunction::result_type, uint32_t>, uint32_t, uint64_t>>
std::unique_ptr<cudf::column> word_minhash_fn(cudf::lists_column_view const& input,
                                              cudf::device_span<hash_value_type const> seeds,
                                              rmm::cuda_stream_view stream,
                                              rmm::device_async_resource_ref mr)
{
  CUDF_EXPECTS(!seeds.empty(), "Parameter seeds cannot be empty", std::invalid_argument);
  CUDF_EXPECTS((static_cast<std::size_t>(input.size()) * seeds.size()) <
                 static_cast<std::size_t>(std::numeric_limits<cudf::size_type>::max()),
               "The number of seeds times the number of input rows exceeds the column size limit",
               std::overflow_error);

  auto const output_type = cudf::data_type{cudf::type_to_id<hash_value_type>()};
  if (input.is_empty()) { return cudf::make_empty_column(output_type); }

  auto const d_input = cudf::column_device_view::create(input.parent(), stream);

  auto hashes   = cudf::make_numeric_column(output_type,
                                          input.size() * static_cast<cudf::size_type>(seeds.size()),
                                          cudf::mask_state::UNALLOCATED,
                                          stream,
                                          mr);
  auto d_hashes = hashes->mutable_view().data<hash_value_type>();
  auto lcdv     = cudf::detail::lists_column_device_view(*d_input);

  constexpr cudf::thread_index_type block_size = 256;
  cudf::detail::grid_1d grid{
    static_cast<cudf::thread_index_type>(input.size()) * cudf::detail::warp_size, block_size};
  minhash_word_kernel<HashFunction>
    <<<grid.num_blocks, grid.num_threads_per_block, 0, stream.value()>>>(lcdv, seeds, d_hashes);

  return hashes;
}

std::unique_ptr<cudf::column> build_list_result(cudf::column_view const& input,
                                                std::unique_ptr<cudf::column>&& hashes,
                                                cudf::size_type seeds_size,
                                                rmm::cuda_stream_view stream,
                                                rmm::device_async_resource_ref mr)
{
  // build the offsets for the output lists column
  auto const zero = cudf::numeric_scalar<cudf::size_type>(0, true, stream);
  auto const size = cudf::numeric_scalar<cudf::size_type>(seeds_size, true, stream);
  auto offsets    = cudf::detail::sequence(input.size() + 1, zero, size, stream, mr);
  hashes->set_null_mask(rmm::device_buffer{}, 0);  // children have no nulls

  // build the lists column from the offsets and the hashes
  auto result = make_lists_column(input.size(),
                                  std::move(offsets),
                                  std::move(hashes),
                                  input.null_count(),
                                  cudf::detail::copy_bitmask(input, stream, mr),
                                  stream,
                                  mr);
  // expect this condition to be very rare
  if (input.null_count() > 0) {
    result = cudf::detail::purge_nonempty_nulls(result->view(), stream, mr);
  }
  return result;
}
}  // namespace

std::unique_ptr<cudf::column> minhash(cudf::strings_column_view const& input,
                                      cudf::numeric_scalar<uint32_t> const& seed,
                                      cudf::size_type width,
                                      rmm::cuda_stream_view stream,
                                      rmm::device_async_resource_ref mr)
{
  using HashFunction = cudf::hashing::detail::MurmurHash3_x86_32<cudf::string_view>;
  auto const seeds   = cudf::device_span<uint32_t const>{seed.data(), 1};
  auto hashes        = detail::minhash_fn<HashFunction>(input, seeds, width, stream, mr);
  hashes->set_null_mask(cudf::detail::copy_bitmask(input.parent(), stream, mr), input.null_count());
  return hashes;
}

std::unique_ptr<cudf::column> minhash(cudf::strings_column_view const& input,
                                      cudf::device_span<uint32_t const> seeds,
                                      cudf::size_type width,
                                      rmm::cuda_stream_view stream,
                                      rmm::device_async_resource_ref mr)
{
  using HashFunction = cudf::hashing::detail::MurmurHash3_x86_32<cudf::string_view>;
  auto hashes        = detail::minhash_fn<HashFunction>(input, seeds, width, stream, mr);
  return build_list_result(input.parent(), std::move(hashes), seeds.size(), stream, mr);
}

std::unique_ptr<cudf::column> minhash(cudf::strings_column_view const& input,
                                      cudf::device_span<uint32_t const> parmA,
                                      cudf::device_span<uint32_t const> parmB,
                                      cudf::size_type width,
                                      rmm::cuda_stream_view stream,
                                      rmm::device_async_resource_ref mr)
{
  using HashFunction = cudf::hashing::detail::MurmurHash3_x86_32<cudf::string_view>;
  auto hashes        = detail::minhash_fn<HashFunction>(input, parmA, parmB, width, stream, mr);
  return build_list_result(input.parent(), std::move(hashes), parmA.size(), stream, mr);
}

std::unique_ptr<cudf::column> minhash64(cudf::strings_column_view const& input,
                                        cudf::numeric_scalar<uint64_t> const& seed,
                                        cudf::size_type width,
                                        rmm::cuda_stream_view stream,
                                        rmm::device_async_resource_ref mr)
{
  using HashFunction = cudf::hashing::detail::MurmurHash3_x64_128<cudf::string_view>;
  auto const seeds   = cudf::device_span<uint64_t const>{seed.data(), 1};
  auto hashes        = detail::minhash_fn<HashFunction>(input, seeds, width, stream, mr);
  hashes->set_null_mask(cudf::detail::copy_bitmask(input.parent(), stream, mr), input.null_count());
  return hashes;
}

std::unique_ptr<cudf::column> minhash64(cudf::strings_column_view const& input,
                                        cudf::device_span<uint64_t const> seeds,
                                        cudf::size_type width,
                                        rmm::cuda_stream_view stream,
                                        rmm::device_async_resource_ref mr)
{
  using HashFunction = cudf::hashing::detail::MurmurHash3_x64_128<cudf::string_view>;
  auto hashes        = detail::minhash_fn<HashFunction>(input, seeds, width, stream, mr);
  return build_list_result(input.parent(), std::move(hashes), seeds.size(), stream, mr);
}

std::unique_ptr<cudf::column> minhash64(cudf::strings_column_view const& input,
                                        cudf::device_span<uint64_t const> parmA,
                                        cudf::device_span<uint64_t const> parmB,
                                        cudf::size_type width,
                                        rmm::cuda_stream_view stream,
                                        rmm::device_async_resource_ref mr)
{
  using HashFunction = cudf::hashing::detail::MurmurHash3_x64_128<cudf::string_view>;
  auto hashes        = detail::minhash_fn<HashFunction>(input, parmA, parmB, width, stream, mr);
  return build_list_result(input.parent(), std::move(hashes), parmA.size(), stream, mr);
}

std::unique_ptr<cudf::column> word_minhash(cudf::lists_column_view const& input,
                                           cudf::device_span<uint32_t const> seeds,
                                           rmm::cuda_stream_view stream,
                                           rmm::device_async_resource_ref mr)
{
  using HashFunction = cudf::hashing::detail::MurmurHash3_x86_32<cudf::string_view>;
  auto hashes        = detail::word_minhash_fn<HashFunction>(input, seeds, stream, mr);
  return build_list_result(input.parent(), std::move(hashes), seeds.size(), stream, mr);
}

std::unique_ptr<cudf::column> word_minhash64(cudf::lists_column_view const& input,
                                             cudf::device_span<uint64_t const> seeds,
                                             rmm::cuda_stream_view stream,
                                             rmm::device_async_resource_ref mr)
{
  using HashFunction = cudf::hashing::detail::MurmurHash3_x64_128<cudf::string_view>;
  auto hashes        = detail::word_minhash_fn<HashFunction>(input, seeds, stream, mr);
  return build_list_result(input.parent(), std::move(hashes), seeds.size(), stream, mr);
}
}  // namespace detail

std::unique_ptr<cudf::column> minhash(cudf::strings_column_view const& input,
                                      cudf::numeric_scalar<uint32_t> seed,
                                      cudf::size_type width,
                                      rmm::cuda_stream_view stream,
                                      rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::minhash(input, seed, width, stream, mr);
}

std::unique_ptr<cudf::column> minhash(cudf::strings_column_view const& input,
                                      cudf::device_span<uint32_t const> seeds,
                                      cudf::size_type width,
                                      rmm::cuda_stream_view stream,
                                      rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::minhash(input, seeds, width, stream, mr);
}

std::unique_ptr<cudf::column> minhash_permuted(cudf::strings_column_view const& input,
                                               cudf::device_span<uint32_t const> parmA,
                                               cudf::device_span<uint32_t const> parmB,
                                               cudf::size_type width,
                                               rmm::cuda_stream_view stream,
                                               rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::minhash(input, parmA, parmB, width, stream, mr);
}

std::unique_ptr<cudf::column> minhash64(cudf::strings_column_view const& input,
                                        cudf::numeric_scalar<uint64_t> seed,
                                        cudf::size_type width,
                                        rmm::cuda_stream_view stream,
                                        rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::minhash64(input, seed, width, stream, mr);
}

std::unique_ptr<cudf::column> minhash64(cudf::strings_column_view const& input,
                                        cudf::device_span<uint64_t const> seeds,
                                        cudf::size_type width,
                                        rmm::cuda_stream_view stream,
                                        rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::minhash64(input, seeds, width, stream, mr);
}

std::unique_ptr<cudf::column> minhash64_permuted(cudf::strings_column_view const& input,
                                                 cudf::device_span<uint64_t const> parmA,
                                                 cudf::device_span<uint64_t const> parmB,
                                                 cudf::size_type width,
                                                 rmm::cuda_stream_view stream,
                                                 rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::minhash64(input, parmA, parmB, width, stream, mr);
}

std::unique_ptr<cudf::column> word_minhash(cudf::lists_column_view const& input,
                                           cudf::device_span<uint32_t const> seeds,
                                           rmm::cuda_stream_view stream,
                                           rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::word_minhash(input, seeds, stream, mr);
}

std::unique_ptr<cudf::column> word_minhash64(cudf::lists_column_view const& input,
                                             cudf::device_span<uint64_t const> seeds,
                                             rmm::cuda_stream_view stream,
                                             rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::word_minhash64(input, seeds, stream, mr);
}
}  // namespace nvtext
